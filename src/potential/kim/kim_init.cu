#include "hip/hip_runtime.h"
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <onika/math/basic_types_stream.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/domain.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include "kim_init.h"

namespace exaStamp
{

  using namespace exanb;

  class KIMInitOperator : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( std::string, model     , INPUT , REQUIRED );

  public:
    // Operator execution
    inline void execute () override final
    {

      std::cout << "KIM Initialization function" << std::endl;
    }

  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(kim_init)
  {  
    OperatorNodeFactory::instance()->register_factory( "kim_init" , make_simple_operator< KIMInitOperator > );
  }

}
