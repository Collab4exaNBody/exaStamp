#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <onika/math/basic_types_stream.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/domain.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include "kim.h"
#define MY_ERROR(message)                                                \
  {                                                                      \
    std::cout << "* Error : \"" << message << "\" : " << __LINE__ << ":" \
              << __FILE__ << std::endl;                                  \
    exit(1);                                                             \
  }

namespace exaStamp
{

  using namespace exanb;
  using namespace KIM;
  
  class KIMInitOperator : public OperatorNode
  {
    // ========= I/O slots =======================
    //    ADD_SLOT( KIMParams,   parameters    , INPUT , REQUIRED );
    ADD_SLOT( std::string,   kim_model_name , INPUT_OUTPUT , REQUIRED );
    ADD_SLOT( double,        rcut_max       , INPUT_OUTPUT );    
    ADD_SLOT( KIMContext,    kim_ctx        , OUTPUT );
    
  public:
    // Operator execution
    inline void execute () override final
    {
      std::cout << "KIM Initialization function" << std::endl;

      KIM::Model * kim_local_model;
      int requestedUnitsAccepted;
      int error = KIM::Model::Create(KIM::NUMBERING::zeroBased,
                                     KIM::LENGTH_UNIT::A,
                                     KIM::ENERGY_UNIT::eV,
                                     KIM::CHARGE_UNIT::e,
                                     KIM::TEMPERATURE_UNIT::K,
                                     KIM::TIME_UNIT::ps,
                                     *kim_model_name,
                                     &requestedUnitsAccepted,
                                     &kim_local_model);
      if (error) { MY_ERROR("KIM::Model::Create()"); }
      
      // Check for compatibility with the model
      if (!requestedUnitsAccepted) { MY_ERROR("Must Adapt to model units"); }

      // Check that we know about all required routines
      int numberOfModelRoutineNames;
      KIM::MODEL_ROUTINE_NAME::GetNumberOfModelRoutineNames(&numberOfModelRoutineNames);
      
      for (int i = 0; i < numberOfModelRoutineNames; ++i)
        {
          KIM::ModelRoutineName modelRoutineName;
          int error
            = KIM::MODEL_ROUTINE_NAME::GetModelRoutineName(i, &modelRoutineName);
          if (error) { MY_ERROR("Unable to get ModelRoutineName."); }
          int present;
          int required;
          error = kim_local_model->IsRoutinePresent(modelRoutineName, &present, &required);
          if (error) { MY_ERROR("Unable to get routine present/required."); }

          std::cout << "Model routine name \"" << modelRoutineName.ToString()
                    << "\" has present = " << present
                    << " and required = " << required << "." << std::endl;

          if ((present == true) && (required == true))
            {
              using namespace KIM::MODEL_ROUTINE_NAME;
              if (!((modelRoutineName == Create)
                    || (modelRoutineName == ComputeArgumentsCreate)
                    || (modelRoutineName == Compute) || (modelRoutineName == Refresh)
                    || (modelRoutineName == ComputeArgumentsDestroy)
                    || (modelRoutineName == Destroy)))
                {
                  MY_ERROR("Unknown Routine \"" + modelRoutineName.ToString()
                           + "\" is required by model.");
                }
            }
        }

      // print model units
      KIM::LengthUnit lengthUnit;
      KIM::EnergyUnit energyUnit;
      KIM::ChargeUnit chargeUnit;
      KIM::TemperatureUnit temperatureUnit;
      KIM::TimeUnit timeUnit;

      kim_local_model->GetUnits(&lengthUnit, &energyUnit, &chargeUnit, &temperatureUnit, &timeUnit);

      std::cout << "LengthUnit is \"" << lengthUnit.ToString() << "\"" << std::endl
                << "EnergyUnit is \"" << energyUnit.ToString() << "\"" << std::endl
                << "ChargeUnit is \"" << chargeUnit.ToString() << "\"" << std::endl
                << "TemperatureUnit is \"" << temperatureUnit.ToString() << "\""
                << std::endl
                << "TimeUnit is \"" << timeUnit.ToString() << "\"" << std::endl;

      // check species
      int speciesIsSupported;
      int modelTaCode;
      error = kim_local_model->GetSpeciesSupportAndCode(KIM::SPECIES_NAME::Ta, &speciesIsSupported, &modelTaCode);
      if ((error) || (!speciesIsSupported))
        {
          MY_ERROR("Species Ta not supported");
        }

      KIM::ComputeArguments * computeArguments;
      error = kim_local_model->ComputeArgumentsCreate(&computeArguments);
      if (error) { MY_ERROR("Unable to create a ComputeArguments object."); }

      const double* cutoffs;
      double influencedistance;
      
      const int* modelWillNotRequestNeighborsOfNoncontributingParticles;
      int numberOfNeighborLists;
      kim_local_model->GetNeighborListPointers(&numberOfNeighborLists,
                                               &cutoffs,
                                               &modelWillNotRequestNeighborsOfNoncontributingParticles);
      double rcutmax = 0.;
      for (int i=0; i<numberOfNeighborLists;i++) {
        rcutmax = std::max(rcutmax, cutoffs[i]);
      }
      *rcut_max = rcutmax;
      std::cout << "RCUT MAX = " << *rcut_max << std::endl;
      kim_local_model->GetInfluenceDistance( &influencedistance);
      std::cout << "INF DIST = " << influencedistance << std::endl;
      
    }
    
    //    (*parameters).rcut = *rcut;    
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(kim_init)
  {  
    OperatorNodeFactory::instance()->register_factory( "kim_init" , make_simple_operator< KIMInitOperator > );
  }

}
