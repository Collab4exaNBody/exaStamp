#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/file_utils.h>

#include "kim.h"

#include <onika/physics/constants.h>
#include <onika/cuda/cuda_context.h>

#include <vector>
#include <memory>
#include <iostream>
#define MY_ERROR(message)                                                \
  {                                                                      \
    std::cout << "* Error : \"" << message << "\" : " << __LINE__ << ":" \
              << __FILE__ << std::endl;                                  \
    exit(1);                                                             \
  }

namespace exaStamp
{
  using onika::memory::DEFAULT_ALIGNMENT;


  /* Define neighborlist structure */
  typedef struct
  {
    double cutoff;
    int numberOfParticles;
    int * NNeighbors;
    int * neighborList;
  } NeighList;
  
  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_id >
    >
  class KIMComputeForce : public OperatorNode
  {
    // ========= I/O slots =======================
    //    ADD_SLOT( KIMParams             , parameters        , INPUT        , REQUIRED );
    ADD_SLOT( std::string           , kim_model_name    , INPUT );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0      );
    ADD_SLOT( ParticleSpecies       , species           , INPUT        , REQUIRED );
    ADD_SLOT( int64_t               , timestep          , INPUT        , REQUIRED );
    ADD_SLOT( GridChunkNeighbors    , chunk_neighbors   , INPUT        , GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                  , ghost             , INPUT        , false    );
    ADD_SLOT( GridT                 , grid              , INPUT_OUTPUT            );
    ADD_SLOT( Domain                , domain            , INPUT        , REQUIRED );
    ADD_SLOT( KIMContext            , kim_ctx           , INPUT );

    // shortcut to the Compute buffer used (and passed to functor) by compute_cell_particle_pairs
    using ComputeBuffer = ComputePairBuffer2<false,false>;
    using CellParticles = typename GridT::CellParticles;
    //    using ParticleLock  = decltype( ComputePairOptionalLocks<false>{}[0][0] );

    // compile time constant indicating if grid has virial field
    static constexpr bool has_virial_field = GridHasField<GridT,field::_virial>::value;

    // attributes processed during computation
    using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_id >;
    using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_id, field::_virial >;
    using ComputeFields              = std::conditional_t< has_virial_field , ComputeFieldsWithVirial , ComputeFieldsWithoutVirial >;
    static constexpr ComputeFields compute_force_field_set{};
    
  public:
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );
      size_t nt = omp_get_max_threads();
      if (nt > kim_ctx->m_thread_ctx.size()) {
        size_t old_nt = kim_ctx->m_thread_ctx.size();
        std::cout << "resizing thread context " << std::endl;
        std::cout << "\told size = " << old_nt << ", new size = " << nt << std::endl;
        kim_ctx->m_thread_ctx.resize( nt );
        int requestedUnitsAccepted;
        int error;
        for(size_t j=old_nt;j<nt;j++)
          {
            assert( kim_ctx->m_thread_ctx[j].kim_model == nullptr );
            error = KIM::Model::Create(KIM::NUMBERING::zeroBased,
                                       KIM::LENGTH_UNIT::A,
                                       KIM::ENERGY_UNIT::eV,
                                       KIM::CHARGE_UNIT::e,
                                       KIM::TEMPERATURE_UNIT::K,
                                       KIM::TIME_UNIT::ps,
                                       *kim_model_name,
                                       &requestedUnitsAccepted,
                                       &kim_ctx->m_thread_ctx[j].kim_model);
            if (error) { MY_ERROR("KIM::Model::Create()"); }
            // Check for compatibility with the model
            if (!requestedUnitsAccepted) { MY_ERROR("Must Adapt to model units"); }


            // Check that we know about all required routines
            int numberOfModelRoutineNames;
            KIM::MODEL_ROUTINE_NAME::GetNumberOfModelRoutineNames(&numberOfModelRoutineNames);
      
            for (int i = 0; i < numberOfModelRoutineNames; ++i)
              {
                KIM::ModelRoutineName modelRoutineName;
                int error = KIM::MODEL_ROUTINE_NAME::GetModelRoutineName(i, &modelRoutineName);
                if (error) { MY_ERROR("Unable to get ModelRoutineName."); }
                int present;
                int required;
                error = kim_ctx->m_thread_ctx[j].kim_model->IsRoutinePresent(modelRoutineName, &present, &required);
                if (error) { MY_ERROR("Unable to get routine present/required."); }

                std::cout << "Model routine name \"" << modelRoutineName.ToString()
                          << "\" has present = " << present
                          << " and required = " << required << "." << std::endl;

                if ((present == true) && (required == true))
                  {
                    using namespace KIM::MODEL_ROUTINE_NAME;
                    if (!((modelRoutineName == Create)
                          || (modelRoutineName == ComputeArgumentsCreate)
                          || (modelRoutineName == Compute) || (modelRoutineName == Refresh)
                          || (modelRoutineName == ComputeArgumentsDestroy)
                          || (modelRoutineName == Destroy)))
                      {
                        MY_ERROR("Unknown Routine \"" + modelRoutineName.ToString()
                                 + "\" is required by model.");
                      }
                  }
              }

            // print model units
            KIM::LengthUnit lengthUnit;
            KIM::EnergyUnit energyUnit;
            KIM::ChargeUnit chargeUnit;
            KIM::TemperatureUnit temperatureUnit;
            KIM::TimeUnit timeUnit;

            kim_ctx->m_thread_ctx[j].kim_model->GetUnits(&lengthUnit, &energyUnit, &chargeUnit, &temperatureUnit, &timeUnit);

            std::cout << "LengthUnit is \"" << lengthUnit.ToString() << "\"" << std::endl
                      << "EnergyUnit is \"" << energyUnit.ToString() << "\"" << std::endl
                      << "ChargeUnit is \"" << chargeUnit.ToString() << "\"" << std::endl
                      << "TemperatureUnit is \"" << temperatureUnit.ToString() << "\""
                      << std::endl
                      << "TimeUnit is \"" << timeUnit.ToString() << "\"" << std::endl;

            // check species
            int speciesIsSupported;
            int modelTaCode;
            error = kim_ctx->m_thread_ctx[j].kim_model->GetSpeciesSupportAndCode(KIM::SPECIES_NAME::Ta, &speciesIsSupported, &modelTaCode);
            if ((error) || (!speciesIsSupported))
              {
                MY_ERROR("Species Ta not supported");
              }

            KIM::ComputeArguments * computeArguments;
            error = kim_ctx->m_thread_ctx[j].kim_model->ComputeArgumentsCreate(&computeArguments);
            if (error) { MY_ERROR("Unable to create a ComputeArguments object."); }
          }
      }
          
      //      if (nt > pace
      size_t n_cells = grid->number_of_cells();
      if( n_cells == 0 )
      {
        return ;
      }
		
      ForceOp force_op { *rcut_max , kim_ctx->m_thread_ctx.data() };
      ComputePairNullWeightIterator          cp_weight{};
      ComputePairOptionalLocks<false>        cp_locks {};
      GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();
      ComputePairTrivialCellFiltering cpu_cell_filter = {};

      if( domain->xform_is_identity() )
        {
          NullXForm cp_xform;
          auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks, cpu_cell_filter );
          compute_cell_particle_pairs( *grid, *rcut_max, *ghost, optional, force_buf, force_op , compute_force_field_set, parallel_execution_context() );
        }
      else
        {
          LinearXForm cp_xform { domain->xform() };
          auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks, cpu_cell_filter );
          compute_cell_particle_pairs( *grid, *rcut_max, *ghost, optional, force_buf, force_op , compute_force_field_set, parallel_execution_context());
        }
    }
    
    private:
    
    struct alignas(DEFAULT_ALIGNMENT) ForceOp 
    {
      const double m_rcut;

      KIMThreadContext* m_thread_ctx = nullptr;
      
      inline void operator ()
        (
        size_t n,
        ComputeBuffer& buf,
        double& en,
        double& fx,
        double& fy,
        double& fz,
        unsigned int type, // On a besoin du type de l'atome courant
        unsigned int id, // idem pour l'identifiant de l'atome courant
        CellParticles* unused
        ) const
      {
        Mat3d virial;
        this->operator () ( n,buf,en,fx,fy,fz,type,id,virial, unused );
      }

      inline void operator ()
        (
        size_t n,
        ComputeBuffer& buf,
        double& en,
        double& fx,
        double& fy,
        double& fz,
        unsigned int type,
        unsigned int id,
        Mat3d& virial ,
        CellParticles*
        ) const
      {
        
        size_t tid = omp_get_thread_num();
        assert(tid < m_thread_ctx.size());
        KIMThreadContext & kim_ctx = m_thread_ctx[tid];
        auto kimptr = kim_ctx.kim_model;
        
        // energy and force contributions to the particle
        double _fx = 0.;	
        double _fy = 0.;
        double _fz = 0.;
        int numberOfParticles_cluster = n;
        double energy_cluster_model;
        double forces_cluster[n * 3];

        /* Setup local neighborhood coordinates */
        double coords_cluster[n][3];
        for (int i = 0; i < n; ++i) {
          coords_cluster[i][0] = buf.drx[i];
          coords_cluster[i][1] = buf.dry[i];
          coords_cluster[i][2] = buf.drz[i];
        }
        
        // Flag to decide whether neighboring particle contributes to central particle's
        int particleContributing_cluster_model[n];
        for (int i = 0; i < n; ++i)
          particleContributing_cluster_model[i] = 1; /* every particle contributes */

        /* setup particleSpecies */
        int particleSpecies_cluster_model[n];
        int isSpeciesSupported;
        int error = kimptr->GetSpeciesSupportAndCode(KIM::SPECIES_NAME::Ta,
                                                     &isSpeciesSupported,
                                                     &(particleSpecies_cluster_model[0]));
        if (error) MY_ERROR("get_species_code");
        for (int i = 1; i < n; ++i)
          particleSpecies_cluster_model[i] = particleSpecies_cluster_model[0];
        std::vector<Vec3d> centerParticleCoordinates;
        centerParticleCoordinates.resize(n);

        //        MatrixXd localForces(subconfigOfParticle.numberOfParticles, DIM);
        // localForces.setZero();
        KIM::ComputeArguments * computeArguments;
        error = kimptr->ComputeArgumentsCreate(&computeArguments);

        error = kimptr->Compute(computeArguments);

        int numberOfComputeArgumentNames;
        KIM::COMPUTE_ARGUMENT_NAME::GetNumberOfComputeArgumentNames(&numberOfComputeArgumentNames);
        for (int i = 0; i < numberOfComputeArgumentNames; ++i)
          {
            KIM::ComputeArgumentName computeArgumentName;
            KIM::SupportStatus supportStatus;
            KIM::COMPUTE_ARGUMENT_NAME::GetComputeArgumentName(i, &computeArgumentName);
            KIM::DataType dataType;
            KIM::COMPUTE_ARGUMENT_NAME::GetComputeArgumentDataType(computeArgumentName,
                                                                   &dataType);
            error = computeArguments->GetArgumentSupportStatus(computeArgumentName,
                                                               &supportStatus);
            if (error) MY_ERROR("unable to get ComputeArgument SupportStatus");

            std::cout << "ComputeArgument Name \"" << computeArgumentName.ToString()
                      << "\""
                      << " is of type \"" << dataType.ToString() << "\""
                      << " and has supportStatus \"" << supportStatus.ToString() << "\""
                      << std::endl;
          }
        
        int numberOfParameters;
        kimptr->GetNumberOfParameters(&numberOfParameters);
        for (int i = 0; i < numberOfParameters; ++i)
          {
            KIM::DataType dataType;
            std::string const * strName;
            std::string const * strDesc;
            int extent;
            kimptr->GetParameterMetadata(i, &dataType, &extent, &strName, &strDesc);
            std::cout << "Parameter No. " << i << " has" << std::endl
                      << " data type   : \"" << dataType.ToString() << "\"" << std::endl
                      << " extent      : " << extent << std::endl
                      << " name        : " << *strName << std::endl
                      << " description : " << *strDesc << std::endl;
          }

        // Check supported extensions, if any
        int present;
        error = kimptr->IsRoutinePresent(KIM::MODEL_ROUTINE_NAME::Extension, &present, NULL);
        if (error) { MY_ERROR("Unable to get Extension present/required."); }
        if (present)
          {
            KIM::SupportedExtensions supportedExtensions;
            error = kimptr->Extension(KIM_SUPPORTED_EXTENSIONS_ID,&supportedExtensions);
            if (error) { MY_ERROR("Error returned from KIM::Model::Extension()."); }
            std::cout << "Model Supports "
                      << supportedExtensions.numberOfSupportedExtensions
                      << " Extensions:" << std::endl;
            for (int i = 0; i < supportedExtensions.numberOfSupportedExtensions; ++i)
              {
                std::cout << " spportedExtensionID[" << std::setw(2) << i << "] = \""
                          << supportedExtensions.supportedExtensionID[i] << "\" "
                          << "which has required = "
                          << supportedExtensions.supportedExtensionRequired[i] << "."
                          << std::endl;
              }
          }

        error = computeArguments->SetArgumentPointer(KIM::COMPUTE_ARGUMENT_NAME::numberOfParticles,
                                                     (int *) &numberOfParticles_cluster)
          || computeArguments->SetArgumentPointer(KIM::COMPUTE_ARGUMENT_NAME::particleSpeciesCodes,
                                                  particleSpecies_cluster_model)
          || computeArguments->SetArgumentPointer(KIM::COMPUTE_ARGUMENT_NAME::particleContributing,
                                                  particleContributing_cluster_model)
          || computeArguments->SetArgumentPointer(KIM::COMPUTE_ARGUMENT_NAME::coordinates,
                                                  (double *) coords_cluster)
          || computeArguments->SetArgumentPointer(KIM::COMPUTE_ARGUMENT_NAME::partialEnergy,
                                                  &energy_cluster_model)
          || computeArguments->SetArgumentPointer(KIM::COMPUTE_ARGUMENT_NAME::partialForces,
                                                  (double *) forces_cluster);

        // if (error) MY_ERROR("KIM_API_set_data");
        // error = computeArguments->SetCallbackPointer(KIM::COMPUTE_CALLBACK_NAME::GetNeighborList,
        //                                              KIM::LANGUAGE_NAME::cpp,
        //                                              (KIM::Function *) &get_cluster_neigh,
        //                                              &nl_cluster_model);
        // if (error) MY_ERROR("set_call_back");
        
        error = kimptr->Compute(computeArguments);
        if (error) MY_ERROR("compute");
        
        error = kimptr->ComputeArgumentsDestroy(&computeArguments);
        
                // // broadcast to model
                // p_kimLocal->broadcastToModel(&subconfigOfParticle,
                //                      subconfigOfParticle.particleContributing,
                //                      &localForces,
                //                      nlOfParticle,
                //                      (KIM::Function *) &nbl_get_neigh,
                //                      nullptr,
                //                      nullptr);
                // // compute partial forces
                // p_kimLocal->compute();
        
      }
    };

  };

  template<class GridT> using KIMComputeForceTmpl = KIMComputeForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(kim)
  {
    OperatorNodeFactory::instance()->register_factory( "kim_force" ,make_grid_variant_operator< KIMComputeForceTmpl > );
  }

}


