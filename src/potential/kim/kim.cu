#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/file_utils.h>

#include "kim.h"
#include "KIM_Log.hpp"
#include "KIM_LogVerbosity.hpp"
//#include "Eigen/Eigen/Dense"
#include <onika/physics/units.h>

#include <onika/physics/constants.h>
#include <onika/cuda/cuda_context.h>

#include <vector>
#include <memory>
#include <iostream>
#define MY_ERROR(message)                                                \
  {                                                                      \
    std::cout << "* Error : \"" << message << "\" : " << __LINE__ << ":" \
              << __FILE__ << std::endl;                                  \
    exit(1);                                                             \
  }

namespace exaStamp
{
  using onika::memory::DEFAULT_ALIGNMENT;


  /* Define neighborlist structure */
  typedef struct
  {
    double cutoff;
    int numberOfParticles;
    int * NNeighbors;
    int * neighborList;
  } NeighList;
  
  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_id >
    >
  class KIMComputeForce : public OperatorNode
  {
    // ========= I/O slots =======================
    //    ADD_SLOT( KIMParams             , parameters        , INPUT        , REQUIRED );
    ADD_SLOT( std::string           , kim_model_name    , INPUT );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0      );
    ADD_SLOT( ParticleSpecies       , species           , INPUT        , REQUIRED );
    ADD_SLOT( int64_t               , timestep          , INPUT        , REQUIRED );
    ADD_SLOT( GridChunkNeighbors    , chunk_neighbors   , INPUT        , GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                  , ghost             , INPUT        , false    );
    ADD_SLOT( GridT                 , grid              , INPUT_OUTPUT            );
    ADD_SLOT( Domain                , domain            , INPUT        , REQUIRED );
    ADD_SLOT( KIMContext            , kim_ctx           , INPUT );

    // shortcut to the Compute buffer used (and passed to functor) by compute_cell_particle_pairs
    using ComputeBuffer = ComputePairBuffer2<false,false>;
    using CellParticles = typename GridT::CellParticles;
    //    using ParticleLock  = decltype( ComputePairOptionalLocks<false>{}[0][0] );

    // compile time constant indicating if grid has virial field
    static constexpr bool has_virial_field = GridHasField<GridT,field::_virial>::value;

    // attributes processed during computation
    using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_id >;
    using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_id, field::_virial >;
    using ComputeFields              = std::conditional_t< has_virial_field , ComputeFieldsWithVirial , ComputeFieldsWithoutVirial >;
    static constexpr ComputeFields compute_force_field_set{};
    
  public:
    // Operator execution
    inline void execute () override final
    {
      KIM::Log::PushDefaultVerbosity(KIM::LOG_VERBOSITY::silent);
      
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );
      size_t nt = omp_get_max_threads();
      if (nt > kim_ctx->m_thread_ctx.size()) {
        size_t old_nt = kim_ctx->m_thread_ctx.size();
        kim_ctx->m_thread_ctx.resize( nt );
        int requestedUnitsAccepted;
        int error;
        for(size_t j=old_nt;j<nt;j++)
          {
            assert( kim_ctx->m_thread_ctx[j].kim_model == nullptr );
            error = KIM::Model::Create(KIM::NUMBERING::zeroBased,
                                       KIM::LENGTH_UNIT::A,
                                       KIM::ENERGY_UNIT::eV,
                                       KIM::CHARGE_UNIT::e,
                                       KIM::TEMPERATURE_UNIT::K,
                                       KIM::TIME_UNIT::ps,
                                       *kim_model_name,
                                       &requestedUnitsAccepted,
                                       &kim_ctx->m_thread_ctx[j].kim_model);
            if (error) { MY_ERROR("KIM::Model::Create()"); }
            // Check for compatibility with the model
            if (!requestedUnitsAccepted) { MY_ERROR("Must Adapt to model units"); }


            // Check that we know about all required routines
            int numberOfModelRoutineNames;
            KIM::MODEL_ROUTINE_NAME::GetNumberOfModelRoutineNames(&numberOfModelRoutineNames);
      
            for (int i = 0; i < numberOfModelRoutineNames; ++i)
              {
                KIM::ModelRoutineName modelRoutineName;
                int error = KIM::MODEL_ROUTINE_NAME::GetModelRoutineName(i, &modelRoutineName);
                if (error) { MY_ERROR("Unable to get ModelRoutineName."); }
                int present;
                int required;
                error = kim_ctx->m_thread_ctx[j].kim_model->IsRoutinePresent(modelRoutineName, &present, &required);
                if (error) { MY_ERROR("Unable to get routine present/required."); }

                ldbg << "Model routine name \"" << modelRoutineName.ToString()
                     << "\" has present = " << present
                     << " and required = " << required << "." << std::endl;

                if ((present == true) && (required == true))
                  {
                    using namespace KIM::MODEL_ROUTINE_NAME;
                    if (!((modelRoutineName == Create)
                          || (modelRoutineName == ComputeArgumentsCreate)
                          || (modelRoutineName == Compute) || (modelRoutineName == Refresh)
                          || (modelRoutineName == ComputeArgumentsDestroy)
                          || (modelRoutineName == Destroy)))
                      {
                        MY_ERROR("Unknown Routine \"" + modelRoutineName.ToString()
                                 + "\" is required by model.");
                      }
                  }
              }

            // print model units
            KIM::LengthUnit lengthUnit;
            KIM::EnergyUnit energyUnit;
            KIM::ChargeUnit chargeUnit;
            KIM::TemperatureUnit temperatureUnit;
            KIM::TimeUnit timeUnit;

            kim_ctx->m_thread_ctx[j].kim_model->GetUnits(&lengthUnit, &energyUnit, &chargeUnit, &temperatureUnit, &timeUnit);

            ldbg << "LengthUnit is \"" << lengthUnit.ToString() << "\"" << std::endl
                 << "EnergyUnit is \"" << energyUnit.ToString() << "\"" << std::endl
                 << "ChargeUnit is \"" << chargeUnit.ToString() << "\"" << std::endl
                 << "TemperatureUnit is \"" << temperatureUnit.ToString() << "\""
                 << std::endl
                 << "TimeUnit is \"" << timeUnit.ToString() << "\"" << std::endl;

            // check species
            int speciesIsSupported;
            int modelTaCode;
            error = kim_ctx->m_thread_ctx[j].kim_model->GetSpeciesSupportAndCode(KIM::SPECIES_NAME::Ta, &speciesIsSupported, &modelTaCode);
            if ((error) || (!speciesIsSupported))
              {
                MY_ERROR("Species Ta not supported");
              }

            KIM::ComputeArguments * computeArguments;
            error = kim_ctx->m_thread_ctx[j].kim_model->ComputeArgumentsCreate(&computeArguments);
            if (error) { MY_ERROR("Unable to create a ComputeArguments object."); }
          }
      }
          
      size_t n_cells = grid->number_of_cells();
      if( n_cells == 0 )
      {
        return ;
      }
		
      ForceOp force_op { *rcut_max , kim_ctx->m_thread_ctx.data() };
      ComputePairNullWeightIterator          cp_weight{};
      ComputePairOptionalLocks<false>        cp_locks {};
      GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      ComputePairTrivialCellFiltering        cpu_cell_filter = {};
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();

      if( domain->xform_is_identity() )
        {
          NullXForm cp_xform;
          auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks, cpu_cell_filter );
          compute_cell_particle_pairs( *grid, *rcut_max, *ghost, optional, force_buf, force_op , compute_force_field_set, parallel_execution_context() );
        }
      else
        {
          LinearXForm cp_xform { domain->xform() };
          auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks, cpu_cell_filter );
          compute_cell_particle_pairs( *grid, *rcut_max, *ghost, optional, force_buf, force_op , compute_force_field_set, parallel_execution_context());
        }
      KIM::Log::PopDefaultVerbosity(); 
    }
    
    private:
    
    struct alignas(DEFAULT_ALIGNMENT) ForceOp 
    {
      const double m_rcut;
      KIMThreadContext* m_thread_ctx = nullptr;
      
      inline void operator ()
        (
        size_t n,
        ComputeBuffer& buf,
        double& en,
        double& fx,
        double& fy,
        double& fz,
        unsigned int type,
        unsigned int id,
        CellParticles* unused
        ) const
      {
        Mat3d virial;
        this->operator () ( n,buf,en,fx,fy,fz,type,id,virial, unused);
      }

      inline void operator ()
        (
        size_t n,
        ComputeBuffer& buf,
        double& en,
        double& fx,
        double& fy,
        double& fz,
        unsigned int type,
        unsigned int id,
        Mat3d& virial ,
        CellParticles*
        ) const
      {
        
        size_t tid = omp_get_thread_num();
        //        assert(tid < (*m_thread_ctx).size());
        KIMThreadContext & kim_ctx = m_thread_ctx[tid];
        auto kimptr = kim_ctx.kim_model;
        
        // number of particles in this local cluster: 1 (center) + n neighbors
        const int np = static_cast<int>(n) + 1;
        
        // put central at origin; neighbors are already r_ij = (drx, dry, drz)
        std::vector<double> coords(3 * static_cast<size_t>(np), 0.0);
        for (int i = 0; i < static_cast<int>(n); ++i) {
          coords[3 * ( i + 1 ) + 0] = buf.drx[i];
          coords[3 * ( i + 1 ) + 1] = buf.dry[i];
          coords[3 * ( i + 1 ) + 2] = buf.drz[i];
          //          std::cout << "cx,cy,cz = " << coords[3 * i + 0] << ","<< coords[3 * i + 1] << ","<< coords[3 * i + 2] << std::endl;
        }
        
        // contributing: only central particle is a contributing particle. Other particles just serve to compute energy and force on central particle.
        std::vector<int> contributing(np, 0);
        contributing[0] = 1;
        
        // species: reuse the code you already queried into particleSpecies_cluster_model[0]
        int isSpeciesSupported;
        std::vector<int> species_codes(np, 0);        
        int error = kimptr->GetSpeciesSupportAndCode(KIM::SPECIES_NAME::Ta,
                                                     &isSpeciesSupported,
                                                     &(species_codes[0]));
        if (error) MY_ERROR("get_species_code");

        // Defining outputs
        double localEnergy = 0.0;
        std::vector<double> energies(static_cast<size_t>(np), 0.0);
        std::vector<double> forces(3 * static_cast<size_t>(np), 0.0);
        std::vector<double> virials(6 * static_cast<size_t>(np), 0.0);

        // lightweight neighbor-list payload: ONLY central has neighbors
        struct CentralOnlyNL {
          int np;
          int n;                              // number of neighbors of central
          std::vector<int> neighbors_indices; // length n, values 1..n
        } nl;
        
        nl.np = np;
        nl.n  = static_cast<int>(n);
        nl.neighbors_indices.resize(nl.n);
        for (int j = 0; j < nl.n; ++j) nl.neighbors_indices[j] = j + 1; // neighbors of central

        // prepare ComputeArguments
        KIM::ComputeArguments* computeArguments = nullptr;
        {
          int err = kimptr->ComputeArgumentsCreate(&computeArguments);
          if (err) { MY_ERROR("KIM::ComputeArgumentsCreate() failed."); }
        }

        // wire required argument pointers
        {
          int np_local = np;
          int err =
            computeArguments->SetArgumentPointer(
                                                 KIM::COMPUTE_ARGUMENT_NAME::numberOfParticles, &np_local) ||
            computeArguments->SetArgumentPointer(
                                                 KIM::COMPUTE_ARGUMENT_NAME::particleSpeciesCodes, species_codes.data()) ||
            computeArguments->SetArgumentPointer(
                                                 KIM::COMPUTE_ARGUMENT_NAME::particleContributing, contributing.data()) ||
            computeArguments->SetArgumentPointer(
                                                 KIM::COMPUTE_ARGUMENT_NAME::coordinates, coords.data()) ||
            computeArguments->SetArgumentPointer(
                                                 KIM::COMPUTE_ARGUMENT_NAME::partialEnergy, &localEnergy) ||
            computeArguments->SetArgumentPointer(
                                                 KIM::COMPUTE_ARGUMENT_NAME::partialParticleEnergy, energies.data());
            computeArguments->SetArgumentPointer(
                                                 KIM::COMPUTE_ARGUMENT_NAME::partialForces, forces.data());
            computeArguments->SetArgumentPointer(
                                                 KIM::COMPUTE_ARGUMENT_NAME::partialVirial, virials.data());          
          if (err) { kimptr->ComputeArgumentsDestroy(&computeArguments); MY_ERROR("Error in SetArgumentPointer."); }
        }

        // install a GetNeighborList callback (non-capturing lambda decays to function ptr)
        using GetNeighSig = int(void*, int, double const*, int, int, int*, int const**);
        GetNeighSig* get_neigh_cb = +[](void* dataObject,
                                        int numberOfNeighborLists,
                                        double const* /*cutoffs*/,
                                        int neighborListIndex,
                                        int particleNumber,
                                        int* numberOfNeighbors,
                                        int const** neighborsOfParticle) -> int
        {
          //          auto* nl = static_cast<LocalNeighList*>(dataObject);
          auto* nl = static_cast<CentralOnlyNL*>(dataObject);          
          if (numberOfNeighborLists != 1) return 1;
          if (neighborListIndex != 0)     return 1;
          
          // if (particleNumber < 0 || particleNumber >= nl->numberOfParticles) return 1;

          // *numberOfNeighbors   = nl->NNeighbors[particleNumber];
          // *neighborsOfParticle = &(nl->neighborList[particleNumber * nl->numberOfParticles]);

          if (particleNumber == 0) {
            *numberOfNeighbors   = nl->n;
            *neighborsOfParticle = nl->neighbors_indices.data();
          } else {
            // No neighbor list for non-central particles
            *numberOfNeighbors   = 0;
            *neighborsOfParticle = nullptr;
          }
          return 0;
        };

        {
          int err = computeArguments->SetCallbackPointer(
                                                         KIM::COMPUTE_CALLBACK_NAME::GetNeighborList,
                                                         KIM::LANGUAGE_NAME::cpp,
                                                         reinterpret_cast<KIM::Function*>(get_neigh_cb),
                                                         &nl);
          if (err) { kimptr->ComputeArgumentsDestroy(&computeArguments); MY_ERROR("Error in SetCallbackPointer(GetNeighborList)."); }
        }

        // call the model (use the 1-argument overload as in the official example)
        {
          int err = kimptr->Compute(computeArguments);
          if (err) { kimptr->ComputeArgumentsDestroy(&computeArguments); MY_ERROR("KIM::Model::Compute() failed."); }
        }

        //std::cout << "###################################" << std::endl;
        //std::cout << "list of forces = " << std::endl;
        //std::cout << "fx,fy,fz central atom = " << Vec3d{forces[0],forces[1],forces[2]} << std::endl;
        // Vec3d sumforces {0.,0.,0.};
        // for (int i = 1; i < static_cast<int>(n+1); ++i) {
        //   sumforces += Vec3d{forces[3*i+0],forces[3*i+1],forces[3*i+2] };
        //   //          //std::cout << "fx,fy,fz for atom " << i << " = " << Vec3d{forces[3*i+0],forces[3*i+1],forces[3*i+2] } << std::endl;
        // }
        //std::cout << "sumforces on neihgors = " << sumforces << std::endl;
        //        ldbg << "###################################" << std::endl;

        //        std::cout << "###################################" << std::endl;
        //        std::cout << "list of energies = " << std::endl;
        //        std::cout << "fx,fy,fz central atom = " << Vec3d{forces[0],forces[1],forces[2]} << std::endl;
        //        std::cout << "energy central atom = " << energies[0] << std::endl;        
        // for (int i = 1; i < static_cast<int>(n+1); ++i) {
        //   std::cout << "energy for atom " << i << " = " << energies[i] << std::endl;
        // }
        //std::cout << "sumforces on neihgors = " << sumforces << std::endl;
        //        ldbg << "###################################" << std::endl;                
        double conv_energy_factor = ONIKA_CONST_QUANTITY( 1. * eV ).convert();
        Vec3d localForce = Vec3d{forces[0],forces[1],forces[2]} * conv_energy_factor * 2.0;
        fx += localForce.x;
        fy += localForce.y;
        fz += localForce.z;
        //        en = localEnenergies[0] * conv_energy_factor;
        en = (localEnergy * conv_energy_factor);

        // cleanup
        {
          int err = kimptr->ComputeArgumentsDestroy(&computeArguments);
          if (err) { MY_ERROR("KIM::ComputeArgumentsDestroy() failed."); }
        }
        // -----------------------------------------------------------------------------

        //        KIM::Log::PopDefaultVerbosity();
        //
        
      }
    };

  };

  template<class GridT> using KIMComputeForceTmpl = KIMComputeForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(kim)
  {
    OperatorNodeFactory::instance()->register_factory( "kim_force" ,make_grid_variant_operator< KIMComputeForceTmpl > );
  }

}


