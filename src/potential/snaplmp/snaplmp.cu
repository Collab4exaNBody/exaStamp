#include "hip/hip_runtime.h"
//#/* */pragma xstamp



#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <exanb/core/cpp_utils.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/file_utils.h>

#include <exaStamp/potential/snap/snap_params.h>
#include <exaStamp/potential/snap/snap_read_lammps.h>
#include <exaStamp/potential/snap/snap_config.h>
#include <exaStamp/potential/snap/snap_check_bispectrum.h>

#include <exanb/particle_neighbors/chunk_neighbors.h>

#include <vector>
#include <memory>
#include <iostream>

#include <mpi.h>

#include "sna.h"
#include "memory.h"

#include "snap_context.h"
#include "snap_force_op.h"
#include "snap_bispectrum_op.h"

namespace exaStamp
{

  using namespace exanb;
  using onika::memory::DEFAULT_ALIGNMENT;
//  using namespace SnapExt;


  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz >
    >
  class SnapLMPForce : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( MPI_Comm              , mpi               , INPUT , REQUIRED);
    ADD_SLOT( SnapParms             , parameters        , INPUT , REQUIRED );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( exanb::GridChunkNeighbors , chunk_neighbors   , INPUT , exanb::GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                  , ghost             , INPUT , false );
    ADD_SLOT( bool                  , conv_coef_units   , INPUT , false );
    ADD_SLOT( bool                  , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( GridT                 , grid              , INPUT_OUTPUT );
    ADD_SLOT( Domain                , domain            , INPUT , REQUIRED );
    ADD_SLOT( GridParticleLocks     , particle_locks    , INPUT , OPTIONAL , DocString{"particle spin locks"} );

    ADD_SLOT( long                  , timestep          , INPUT , REQUIRED , DocString{"Iteration number"} );
    ADD_SLOT( std::string           , bispectrumchkfile , INPUT , OPTIONAL , DocString{"file with reference values to check bispectrum correctness"} );

    ADD_SLOT( SnapLMPContext        , snap_ctx          , PRIVATE );

    // shortcut to the Compute buffer used (and passed to functor) by compute_cell_particle_pairs
    static constexpr bool UseWeights = false;
    static constexpr bool UseNeighbors = true;
    //static constexpr bool UseLocks = true;
    //    using ComputeBuffer = ComputePairBuffer2<UseWeights,UseNeighbors>;
    using ComputeBuffer = ComputePairBuffer2<UseWeights,UseNeighbors,SnapComputeBuffer,CopyParticleType>;

    using CellParticles = typename GridT::CellParticles;

    // compile time constant indicating if grid has virial field
    static constexpr bool has_virial_field = GridHasField<GridT,field::_virial>::value;

    // attributes processed during computation
    // using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz >;
    // using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_virial>;
    using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type >;
    using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_virial >;
    using ComputeFields = std::conditional_t< has_virial_field , ComputeFieldsWithVirial , ComputeFieldsWithoutVirial >;
    static constexpr ComputeFields compute_force_field_set{};
    static constexpr FieldSet< field::_type> compute_bispectrum_field_set{};
        
  public:
    
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );

      //ldbg << "rcut="<<snap_ctx->m_rcut <<std::endl << std::flush;
      if( snap_ctx->m_rcut == 0.0 )
      {
        std::string lammps_param = onika::data_file_path( parameters->lammps_param );
        std::string lammps_coef = onika::data_file_path( parameters->lammps_coef ); 
        ldbg << "Snap: read lammps files "<<lammps_param<<" and "<<lammps_coef<<std::endl << std::flush;
        SnapExt::snap_read_lammps(lammps_param, lammps_coef, snap_ctx->m_config, *conv_coef_units );
        ldbg <<"rfac0="<<snap_ctx->m_config.rfac0() <<", rmin0="<<snap_ctx->m_config.rmin0() <<", rcutfac="<<snap_ctx->m_config.rcutfac() 
             <<", twojmax="<<snap_ctx->m_config.twojmax()<<", bzeroflag="<<snap_ctx->m_config.bzeroflag()<<", nmat="<<snap_ctx->m_config.materials().size() <<std::endl;
        snap_ctx->m_rcut = snap_ctx->m_config.rcutfac(); // because LAMMPS uses angstrom while exastamp uses nm
      }

      *rcut_max = std::max( *rcut_max , snap_ctx->m_rcut );
      
      size_t n_cells = grid->number_of_cells();
      if( n_cells==0 )
      {
        return ;
      }

      if( ! particle_locks.has_value() )
      {
        fatal_error() << "No particle locks" << std::endl;
      }

      if( snap_ctx->m_coefs.empty() )
      {
        //snap_ctx->m_cg_nt = parameters->nt;
        
        for( const auto& mat : snap_ctx->m_config.materials() )
        {
          ldbg << '\t' << mat.name() << ": radelem="<<mat.radelem()<<", weight="<<mat.weight()<<", ncoefs="<<mat.number_of_coefficients()<<std::endl;
          for(size_t i=0;i<mat.number_of_coefficients();i++)
          {
            ldbg << "\t\t" << mat.coefficient(i) << std::endl;
          }
        }
        
        //double jmax = snap_ctx->m_config.twojmax()*0.5;
        int nmat = snap_ctx->m_config.materials().size();
	
        // if( nmat != 1 )
        // {
        //   lerr << "Snap: ERROR: only 1 material is allowed" << std::endl;
	//   //          std::abort();
        // }
      
        // temporay, enable mutiple species if they all have weight=1. modifications needed for true multimaterial
        snap_ctx->m_factor.assign( nmat, 1.0 );
        snap_ctx->m_radelem.assign( nmat, 0.0 );

	int cnt=0;
	for ( const auto& mat : snap_ctx->m_config.materials() )
	  {
	    snap_ctx->m_factor[cnt] = mat.weight();	    
	    snap_ctx->m_radelem[cnt] = mat.radelem();
	    cnt+=1;
	  }

        // snap_ctx->m_factor.assign( MAX_PARTICLE_SPECIES, 1.0 );
        // snap_ctx->m_factor[0] = mat.weight();
        // snap_ctx->m_radelem.assign( MAX_PARTICLE_SPECIES, 0.0 );
        // snap_ctx->m_radelem[0] = mat.radelem();
	size_t ncoefs_per_specy = snap_ctx->m_config.materials()[0].number_of_coefficients();
        snap_ctx->m_coefs.resize( nmat * ncoefs_per_specy );
        for(int j=0;j<nmat;j++)
	  {
	    const auto& mat = snap_ctx->m_config.materials()[j];
	    for(size_t i=0;i<ncoefs_per_specy;i++)
	      {
		snap_ctx->m_coefs[ j * ncoefs_per_specy + i ] = mat.coefficient(i);
	      }
	  }

      }
      
      if( snap_ctx->ptr == nullptr )
      {
        snap_ctx->ptr = new LAMMPS_NS::LAMMPS;
        snap_ctx->ptr->error = new LAMMPS_NS::ErrorLogWrapper;
        snap_ctx->ptr->comm = new LAMMPS_NS::CommunicatorInfo;
        snap_ctx->ptr->memory = new LAMMPS_NS::Memory(snap_ctx->ptr);
      }
          
      size_t nt = omp_get_max_threads();
      if( nt > snap_ctx->m_thread_ctx.size() )
      {
        size_t old_nt = snap_ctx->m_thread_ctx.size();
        snap_ctx->m_thread_ctx.resize( nt );
        for(size_t i=old_nt;i<nt;i++)
        {
          assert( snap_ctx->m_thread_ctx[i].sna == nullptr );
          snap_ctx->m_thread_ctx[i].sna =
            new LAMMPS_NS::SNA( snap_ctx->ptr
                              , snap_ctx->m_config.rfac0() 
                              , snap_ctx->m_config.twojmax() 
                              , snap_ctx->m_config.rmin0()
                              , snap_ctx->m_config.switchflag()
                              , snap_ctx->m_config.bzeroflag()
                              , snap_ctx->m_config.chemflag()
                              , snap_ctx->m_config.bnormflag()
                              , snap_ctx->m_config.wselfallflag()
                              , snap_ctx->m_config.nelements()
                              , snap_ctx->m_config.switchinnerflag()
                              );
          snap_ctx->m_thread_ctx[i].sna->init();
          snap_ctx->m_thread_ctx[i].sna->grow_rij(1024);
        }
      }

      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
      {
        ldbg << "trigger_thermo_state = " << *trigger_thermo_state << std::endl;
        log_energy = *trigger_thermo_state ;
      }
      else
      {
        ldbg << "trigger_thermo_state missing " << std::endl;
      }

      const double cutsq = snap_ctx->m_rcut * snap_ctx->m_rcut;
      const bool eflag = log_energy || bispectrumchkfile.has_value();
      const bool quadraticflag = snap_ctx->m_config.quadraticflag();
      const bool switchinnerflag = snap_ctx->m_config.switchinnerflag();
      const bool chemflag = snap_ctx->m_config.chemflag();

      // exanb objects to perform computations with neighbors      
      ComputePairNullWeightIterator cp_weight{};
      exanb::GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();      
      LinearXForm cp_xform { domain->xform() };

      // constants to resize bispectrum and beta intermediate terms
      const size_t total_particles = grid->number_of_particles();
      size_t ncoefs_per_specy = snap_ctx->m_config.materials()[0].number_of_coefficients();
      int ncoeffall = ncoefs_per_specy; //_per_specysnap_ctx->m_coefs.size() ;
      int ncoeff = -1;
      
      if (!quadraticflag)
        ncoeff = ncoeffall - 1;
      else {
        ncoeff = sqrt(2*ncoeffall)-1;
        int ncoeffq = (ncoeff*(ncoeff+1))/2;
        int ntmp = 1+ncoeff+ncoeffq;
        if (ntmp != ncoeffall) {
          lerr << "Incorrect SNAP coeff file" << std::endl;
          std::abort();
        }
      }

      ldbg << "snaplmp: quadratic="<<quadraticflag<<", eflag="<<eflag<<", ncoeff="<<ncoeff<<", ncoeffall="<<ncoeffall<<std::endl;

      if (quadraticflag || eflag)
      {
        // ************ compute_bispectrum(); ****************
        snap_ctx->m_bispectrum.clear();
        snap_ctx->m_bispectrum.resize( total_particles * ncoeff );

        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, ComputePairOptionalLocks<false>{} );
        BispectrumOp bispectrum_op {
                           snap_ctx->m_thread_ctx.data(), snap_ctx->m_thread_ctx.size(),
                           grid->cell_particle_offset_data(), snap_ctx->m_beta.data(), snap_ctx->m_bispectrum.data(),
                           snap_ctx->m_coefs.data(), ncoeff,
                           snap_ctx->m_factor.data(), snap_ctx->m_radelem.data(),
                           nullptr, nullptr,
                           snap_ctx->m_rcut, cutsq,
                           eflag, quadraticflag,
                           switchinnerflag, chemflag
                           };
        compute_cell_particle_pairs( *grid, snap_ctx->m_rcut, *ghost, optional, force_buf, bispectrum_op , compute_bispectrum_field_set , parallel_execution_context() );
        // *********************************************
        if( bispectrumchkfile.has_value() )
        {
          std::ostringstream oss; oss << *bispectrumchkfile << "." << *timestep;
          std::string file_name = onika::data_file_path( oss.str() );
          ldbg << "bispectrumchkfile is set, check bispectrum from file "<< file_name << std::endl;
          snap_check_bispectrum(*mpi, *grid, file_name, ncoeff, snap_ctx->m_bispectrum.data() );
        }
      }

      // // ************ compute_beta(); ****************
      // {        
      //   snap_ctx->m_beta.clear();
      //   snap_ctx->m_beta.resize( total_particles * ncoeff );
      //   for(size_t ii=0;ii<total_particles;ii++)
      //   {
      //     for(int icoeff=0;icoeff<ncoeff;icoeff++)
      //     {
      // 	    snap_ctx->m_beta[ ii * ncoeff + icoeff ] = snap_ctx->m_coefs[icoeff+1];

      // 	    // Here we need to know the particle type to get the proper coefficients (ex: ntypes 0 or 1
      // 	    // We might need to shift the compute_beta into the snap force operator since it is thread dependent.
      // 	    // const int iitype = type[ii];
      // 	    // snap_ctx->m_beta[ ii * ncoeff + icoeff ] = snap_ctx->m_coefs[typeii * (ncoeff+1) + icoeff+1];

      //     }
      //     if (quadraticflag)
      //     {
      //       const double * const coeffi = snap_ctx->m_coefs.data();

      //       int k = ncoeff+1;
      //       for (int icoeff = 0; icoeff < ncoeff; icoeff++) {
      //         double bveci = snap_ctx->m_bispectrum[ ii * ncoeff + icoeff ]; // bispectrum[ii][icoeff];
      //         snap_ctx->m_beta[ ii * ncoeff + icoeff ] /*beta[ii][icoeff]*/ += coeffi[k]*bveci;
      //         k++;
      //         for (int jcoeff = icoeff+1; jcoeff < ncoeff; jcoeff++) {
      //           double bvecj = snap_ctx->m_bispectrum[ ii * ncoeff + jcoeff ]; //bispectrum[ii][jcoeff];
      //           snap_ctx->m_beta[ ii * ncoeff + icoeff ] /*beta[ii][icoeff]*/ += coeffi[k]*bvecj;
      //           snap_ctx->m_beta[ ii * ncoeff + jcoeff ] /*beta[ii][jcoeff]*/ += coeffi[k]*bveci;
      //           k++;
      //         }
      //       }
      //     }
      //     //printf("SNAPDBG: beta[%d]:",int(ii));
      //     //for(int icoeff=0;icoeff<ncoeff;icoeff++) printf(" % .3e",snap_ctx->m_beta[ ii * ncoeff + icoeff ]);
      //     //printf("\n");
      //   }
      // }
      // // *********************************************

      auto compute_opt_locks = [&](auto cp_locks)
      {
        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks );
        ForceOp force_op { snap_ctx->m_thread_ctx.data(), snap_ctx->m_thread_ctx.size(),
                           grid->cell_particle_offset_data(), snap_ctx->m_beta.data(), snap_ctx->m_bispectrum.data(),
                           snap_ctx->m_coefs.data(), static_cast<long>(snap_ctx->m_coefs.size()), ncoeff,
                           snap_ctx->m_factor.data(), snap_ctx->m_radelem.data(),
                           nullptr, nullptr,
                           snap_ctx->m_rcut, cutsq,
                           eflag, quadraticflag,
                           switchinnerflag, chemflag,
                           ! (*conv_coef_units) // if coefficients were not converted, then output energy/force must be converted
                           };      
        compute_cell_particle_pairs( *grid, snap_ctx->m_rcut, *ghost, optional, force_buf, force_op , compute_force_field_set , parallel_execution_context() );
      };
      ldbg << "snaplmp: nthreads="<< omp_get_max_threads() <<std::endl;
      
      if( omp_get_max_threads() > 1 ) compute_opt_locks( ComputePairOptionalLocks<true>{ particle_locks->data() } );
      else                            compute_opt_locks( ComputePairOptionalLocks<false>{} );

      ldbg << "snaplmp: done"<<std::endl;
    }

  };

  template<class GridT> using SnapLMPForceTmpl = SnapLMPForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(snaplmp)
  {
    OperatorNodeFactory::instance()->register_factory( "snaplmp_force" ,make_grid_variant_operator< SnapLMPForceTmpl > );
  }

}


