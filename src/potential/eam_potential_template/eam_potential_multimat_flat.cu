#include "hip/hip_runtime.h"
//  // DO NOT REMOVE THIS LINE

//  // DO NOT REMOVE THIS LINE

#include <exanb/core/grid.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <exanb/core/particle_type_pair.h>
#include <exanb/core/domain.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <exanb/compute/compute_pair_optional_args.h>
#include <onika/thread.h>

#include <exaStamp/potential/eam/eam_buffer.h>
#include <exaStamp/potential/eam/eam_yaml.h>
#include "potential.h"

#include <exanb/particle_neighbors/flat_neighbor_lists.h>
#include <onika/parallel/parallel_for.h>
#include <onika/parallel/memset.h>

#ifdef USTAMP_POTENTIAL_EAM_MM // operator compiled only if potential is multimaterial

#include "eam_force_op_multimat_flat.h"

namespace exaStamp
{
  using namespace exanb;

  template< class GridT >
  class EamPotentialFlatName : public OperatorNode
  {  
    using CellParticles = typename GridT::CellParticles;

    using EamScratch = EamMultimatPotentialScratch< USTAMP_POTENTIAL_PARMS >;
    using StringVector = std::vector< std::string >;

    template<bool NewtonSym, class XFormT> using FlatSymRhoOp = PRIV_NAMESPACE_NAME::FlatSymRhoOp<NewtonSym,XFormT>;
    template<bool EnergyFlag> using FlatRho2EmbOp = PRIV_NAMESPACE_NAME::FlatRho2EmbOp<EnergyFlag>;
    template<bool NewtonSym, class XFormT> using FlatSymForceOp = PRIV_NAMESPACE_NAME::FlatSymForceOp<NewtonSym,XFormT>;

    // ========= I/O slots =======================
    ADD_SLOT( ParticleSpecies       , species          , INPUT , REQUIRED );
    ADD_SLOT( USTAMP_POTENTIAL_PARMS, parameters       , INPUT_OUTPUT , REQUIRED );
    ADD_SLOT( StringVector          , types            , INPUT , StringVector{} , DocString{"Empty list means all types are used, otherwise list types handled by this potential"} );
    ADD_SLOT( double                , rcut             , INPUT );
    ADD_SLOT( double                , rcut_max         , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( double                , ghost_dist_max   , INPUT_OUTPUT , 0.0 );   
    ADD_SLOT( FlatPartNbhList       , flat_nbh_list    , INPUT_OUTPUT );
    ADD_SLOT( GridT                 , grid             , INPUT_OUTPUT );
    ADD_SLOT( Domain                , domain           , INPUT , REQUIRED );

    ADD_SLOT( bool                  , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( bool                  , compute_virial   , INPUT , false );

    ADD_SLOT( bool                  , eam_rho      , INPUT , true );
    ADD_SLOT( bool                  , eam_rho2emb  , INPUT , true );
    ADD_SLOT( bool                  , eam_ghost    , INPUT , true );
    ADD_SLOT( bool                  , eam_force    , INPUT , true );
    ADD_SLOT( bool                  , eam_symmetry , INPUT , false );

    ADD_SLOT( spin_mutex_array      , flat_particle_locks       , INPUT_OUTPUT , OPTIONAL , DocString{"particle spin locks"} );

    ADD_SLOT( EamScratch            , eam_scratch          , PRIVATE );
    
  public:

    // Operator execution
    inline void execute () override final
    {
      //MeamPotential meam( *rcut, *parameters );
      *rcut_max = std::max( *rcut , *rcut_max );
      if( ( *eam_rho || *eam_force ) && *eam_ghost )
      {
        *ghost_dist_max = std::max( *ghost_dist_max , (*rcut) * 2.0 );
      }

      const size_t n_particles = grid->number_of_particles();
      if( n_particles == 0 ) { return ; } // short cut to avoid errors in pre-initialization step

      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
      {
        log_energy = *trigger_thermo_state ;
      }
      else
      {
        ldbg << "trigger_thermo_state missing " << std::endl;
      }

      const bool need_particle_locks = ( omp_get_max_threads() > 1 ) && ( *eam_symmetry ) ;
      const bool need_virial = log_energy && *compute_virial;

      ldbg << "EAM Multimat Flat:"
           <<  " rho="<<std::boolalpha<< *eam_rho
           <<" , rho2emb="<< *eam_rho 
           <<" , rho2emb="<< *eam_rho2emb 
           <<" , force="<< *eam_force
           <<" , ghost="<< *eam_ghost 
           <<" , sym="<< *eam_symmetry
           <<" , eflag="<< log_energy
           <<" , virflag="<<need_virial
           <<" , need_locks="<< need_particle_locks << std::endl;
      
      if( need_particle_locks )
      {
        if( ! flat_particle_locks.has_value() )
        {
          fatal_error()<<"flat_particle_locks is needed, but corresponding slot has no value" << std::endl;
        }
        if( flat_particle_locks->size() != grid->number_of_particles() )
        {
          fatal_error()<<"flat_particle_locks has wrong size : "<<flat_particle_locks->size()<<" <> "<< grid->number_of_particles() << std::endl;
        }
      }

      const size_t n_species = species->size();
      const size_t n_type_pairs = unique_pair_count( n_species );
      const bool initialize_scratch = eam_scratch->m_pair_enabled.empty();
      
      if( initialize_scratch )
      {
        eam_scratch->m_pair_enabled.assign( n_type_pairs , false );
        for(size_t i=0;i<n_type_pairs;i++)
        {
          unsigned int a=0, b=0;
          pair_id_to_type_pair(i,a,b);
          const bool a_enabled = types->empty() || ( std::find( types->begin() , types->end() , species->at(a).name() ) != types->end() );
          const bool b_enabled = types->empty() || ( std::find( types->begin() , types->end() , species->at(b).name() ) != types->end() );
          eam_scratch->m_pair_enabled[i] = ( a_enabled && b_enabled );
        }
      }
      USTAMP_POTENTIAL_EAM_MM_INIT_TYPES( *parameters , n_species , eam_scratch->m_pair_enabled.data() );

      auto rho_emb = grid->field_accessor( field::rho_dEmb );
      auto energy = grid->field_accessor( field::flat_ep );
      auto fx = grid->field_accessor( field::flat_fx );
      auto fy = grid->field_accessor( field::flat_fy );
      auto fz = grid->field_accessor( field::flat_fz );
      auto rx = grid->field_accessor( field::flat_rx );
      auto ry = grid->field_accessor( field::flat_ry );
      auto rz = grid->field_accessor( field::flat_rz );
      auto types = grid->field_accessor( field::flat_type );

      const double rc = *rcut;
      const double rc2 = rc*rc;

      // execute the 2 passes
      auto compute_eam_force = [&]( const auto& cp_xform , auto newtonSym )
      {
        using XFormT = std::remove_reference_t<decltype(cp_xform)>;
        
        if( *eam_rho )
        {          
          onika::parallel::parallel_memset( rho_emb.m_flat_array_ptr , n_particles , 0.0 , parallel_execution_context() );

          FlatSymRhoOp< newtonSym.value , XFormT > rho_op =
            { *parameters, rc2
            , flat_nbh_list->m_neighbor_offset.data(), flat_nbh_list->m_neighbor_list.data(), flat_nbh_list->m_half_count.data()
            , (*eam_ghost) ? nullptr : grid->particle_ghost_flag_data() , eam_scratch->m_pair_enabled.data()
            , rho_emb.m_flat_array_ptr, types.m_flat_array_ptr, rx.m_flat_array_ptr, ry.m_flat_array_ptr, rz.m_flat_array_ptr
            , cp_xform , flat_particle_locks->data() };

          parallel_for( n_particles , rho_op , parallel_execution_context() );
        }
        
        if( *eam_rho2emb )
        {
          if( log_energy )
          {
            std::cout << "Log energy true" << std::endl;
            FlatRho2EmbOp<true> rho2emb_op{ *parameters, (*eam_ghost) ? nullptr : grid->particle_ghost_flag_data(), eam_scratch->m_pair_enabled.data()
                                          , types.m_flat_array_ptr, rho_emb.m_flat_array_ptr, energy.m_flat_array_ptr };
            parallel_for( n_particles , rho2emb_op , parallel_execution_context() );
          }
          else
          {
            FlatRho2EmbOp<false> rho2emb_op{ *parameters, (*eam_ghost) ? nullptr : grid->particle_ghost_flag_data(), eam_scratch->m_pair_enabled.data()
                                          , types.m_flat_array_ptr, rho_emb.m_flat_array_ptr };
            parallel_for( n_particles , rho2emb_op , parallel_execution_context() );
          }
        }

        if( *eam_force )
        {
          FlatSymForceOp< newtonSym.value, XFormT > force_op =
            { *parameters, rc2
            , flat_nbh_list->m_neighbor_offset.data(), flat_nbh_list->m_neighbor_list.data(), flat_nbh_list->m_half_count.data()
            , (*eam_ghost) ? nullptr : grid->particle_ghost_flag_data() , eam_scratch->m_pair_enabled.data()
            , rho_emb.m_flat_array_ptr, types.m_flat_array_ptr
            , rx.m_flat_array_ptr, ry.m_flat_array_ptr, rz.m_flat_array_ptr
            , fx.m_flat_array_ptr, fy.m_flat_array_ptr, fz.m_flat_array_ptr
            , energy.m_flat_array_ptr, cp_xform, flat_particle_locks->data() };
          parallel_for( n_particles , force_op , parallel_execution_context() );            
        }

      };
  
      auto compute_eam_xform = [&]( const auto& cp_xform )
      {
        if( *eam_symmetry ) compute_eam_force( cp_xform , std::true_type{} );
        else                compute_eam_force( cp_xform , std::false_type{} );
      };

      if( domain->xform_is_identity() ) compute_eam_xform( exanb::NullXForm{} );
      else                              compute_eam_xform( exanb::LinearXForm{domain->xform()} );
    }

  };

  namespace tmplhelper
  {
    template<class GridT> using EamPotentialFlatName  = ::exaStamp::EamPotentialFlatName<GridT>;
  }

  // === register factories ===  
  ONIKA_AUTORUN_INIT(eam_potential_multimat_flat)
  {
    OperatorNodeFactory::instance()->register_factory( EamPotentialFlatStr , make_grid_variant_operator< tmplhelper::EamPotentialFlatName > );
  }

}

#endif // only compiled if potential supports multimaterial

