#include "hip/hip_runtime.h"
// _cuda_enable // DO NOT REMOVE THIS LINE

//  // DO NOT REMOVE THIS LINE

#include <exanb/core/grid.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <exanb/core/particle_type_pair.h>
#include <exanb/core/domain.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <exanb/core/cpp_utils.h>

#include <exaStamp/potential/eam/eam_buffer.h>
#include <exaStamp/potential/eam/eam_yaml.h>
#include "potential.h"

#include <exanb/particle_neighbors/chunk_neighbors.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exanb/compute/compute_cell_particles.h>
#include <onika/parallel/memset.h>

#ifdef USTAMP_POTENTIAL_EAM_MM // operator compiled only if potential is multimaterial

#include "eam_force_op_multimat.h"

#ifndef USTAMP_POTENTIAL_EAM_MM_INIT_TYPES
#define USTAMP_POTENTIAL_EAM_MM_INIT_TYPES(p,nt,pe) /**/
#endif

namespace exaStamp
{
  using namespace exanb;

  class EamParameterInitName : public OperatorNode
  {  
    ADD_SLOT( USTAMP_POTENTIAL_PARMS, parameters       , OUTPUT , REQUIRED );
  public:
    inline void execute () override final {}
  };

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep, field::_fx, field::_fy, field::_fz, field::_type >
    >
  class EamPotentialOperatorName : public OperatorNode
  {  
    using CellParticles = typename GridT::CellParticles;

    using EamScratch = EamMultimatPotentialScratch< USTAMP_POTENTIAL_PARMS >;
    using StringVector = std::vector< std::string >;
    template<bool Sym,class CPLocksT> using SymRhoOp = PRIV_NAMESPACE_NAME::SymRhoOp<Sym,CPLocksT>;
    template<bool Sym,class CPLocksT, class VirFieldT> using SymForceOp = PRIV_NAMESPACE_NAME::SymForceOp<Sym,CPLocksT,VirFieldT>;
    using Rho2EmbOp = PRIV_NAMESPACE_NAME::Rho2EmbOp;
    using ForceOpExt = PRIV_NAMESPACE_NAME::ForceOpExtStorage;
    using ForceOpEnergyExt = PRIV_NAMESPACE_NAME::ForceEnergyOpExtStorage;
    using RhoOpExtStorage = PRIV_NAMESPACE_NAME::RhoOpExtStorage;

    // attributes processed during computation
    //static inline constexpr FieldSet< field::_type , field::_virial > cp_force_fields_v{};
    static inline constexpr FieldSet< field::_type > cp_emb_fields_v{};
    static inline constexpr FieldSet< field::_ep , field::_type > cp_emb_fields_energy_v{};

    // ========= I/O slots =======================
    ADD_SLOT( ParticleSpecies       , species          , INPUT , REQUIRED );
    ADD_SLOT( USTAMP_POTENTIAL_PARMS, parameters       , INPUT_OUTPUT , REQUIRED );
    ADD_SLOT( StringVector          , types            , INPUT , StringVector{} , DocString{"Empty list means all types are used, otherwise list types handled by this potential"} );
    ADD_SLOT( double                , rcut             , INPUT );
    ADD_SLOT( double                , rcut_max         , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( double                , ghost_dist_max   , INPUT_OUTPUT , 0.0 );   
    ADD_SLOT( exanb::GridChunkNeighbors , chunk_neighbors  , INPUT , exanb::GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( GridT                 , grid             , INPUT_OUTPUT );
    ADD_SLOT( Domain                , domain           , INPUT , REQUIRED );

    ADD_SLOT( bool                  , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( bool                  , compute_virial   , INPUT , false );

    ADD_SLOT( bool                  , eam_rho      , INPUT , false );
    ADD_SLOT( bool                  , eam_rho2emb  , INPUT , false );
    ADD_SLOT( bool                  , eam_ghost    , INPUT , true );
    ADD_SLOT( bool                  , eam_force    , INPUT , true );
    ADD_SLOT( bool                  , eam_symmetry , INPUT , false );

    ADD_SLOT( GridParticleLocks     , particle_locks      , INPUT_OUTPUT , OPTIONAL , DocString{"particle spin locks"} );

    ADD_SLOT( EamScratch            , eam_scratch      , PRIVATE );
  public:

    // Operator execution
    inline void execute () override final
    {
      //MeamPotential meam( *rcut, *parameters );
      *rcut_max = std::max( *rcut , *rcut_max );
      if( ( *eam_rho || *eam_force ) && *eam_ghost )
      {
        *ghost_dist_max = std::max( *ghost_dist_max , (*rcut) * 2.0 );
      }

      size_t n_cells = grid->number_of_cells();
      if( n_cells == 0 ) { return ; } // short cut to avoid errors in pre-initialization step

      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
      {
        log_energy = *trigger_thermo_state ;
      }
      else
      {
        ldbg << "trigger_thermo_state missing " << std::endl;
      }

      const bool need_particle_locks = ( omp_get_max_threads() > 1 ) && ( *eam_symmetry ) ;
      const bool need_virial = log_energy && *compute_virial;

      ldbg << "EAM Multimat: rho="<<std::boolalpha<< *eam_rho
           <<" , rho2emb="<< *eam_rho 
           <<" , rho2emb="<< *eam_rho2emb 
           <<" , force="<< *eam_force
           <<" , ghost="<< *eam_ghost 
           <<" , sym="<< *eam_symmetry
           <<" , eflag="<< log_energy
           <<" , virflag="<<need_virial
           <<" , need_locks="<< need_particle_locks << std::endl;
      
      // we use eflag also to trigger virial computation
      log_energy = log_energy || need_virial;

      size_t n_species = species->size();
      size_t n_type_pairs = unique_pair_count( n_species );
      
      bool initialize_scratch = eam_scratch->m_pair_enabled.empty();
      if( initialize_scratch )
      {
        eam_scratch->m_pair_enabled.assign( n_type_pairs , false );
        for(size_t i=0;i<n_type_pairs;i++)
        {
          unsigned int a=0, b=0;
          pair_id_to_type_pair(i,a,b);
          const bool a_enabled = types->empty() || ( std::find( types->begin() , types->end() , species->at(a).name() ) != types->end() );
          const bool b_enabled = types->empty() || ( std::find( types->begin() , types->end() , species->at(b).name() ) != types->end() );
          eam_scratch->m_pair_enabled[i] = ( a_enabled && b_enabled );
        }
      }
      USTAMP_POTENTIAL_EAM_MM_INIT_TYPES( *parameters , n_species , eam_scratch->m_pair_enabled.data() );

      auto rho_emb_field = grid->field_accessor( field::rho_dEmb );
      auto * rho_emb_ptr = rho_emb_field.m_flat_array_ptr;

      auto c_rho_emb_field = grid->field_const_accessor( field::rho_dEmb );
      const auto * c_rho_emb_ptr = c_rho_emb_field.m_flat_array_ptr;

      // execute the 2 passes
      auto compute_eam_force = [&]( const auto& cp_xform , const auto& cp_locks , auto cp_emb_fields , auto force_buf )
      {
        using CPLocksT = std::remove_reference_t<decltype(cp_locks)>;
        // common bricks for both compute passes
        exanb::GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
        exanb::GridChunkNeighborsLightWeightIt<true> nbh_it_sym{ *chunk_neighbors };
        ComputePairNullWeightIterator cp_weight {};
        const auto * __restrict__ c_particle_offset = grid->cell_particle_offset_data();

        // 1st pass (new) computes rho then emb, without compute buffer
        if( *eam_rho )
        {
          onika::parallel::parallel_memset( rho_emb_ptr , grid->number_of_particles() , 0.0 , parallel_execution_context() );

          auto rho_op_fields = make_field_tuple_from_field_set( FieldSet< field::_type >{} );
          auto rho_buf_factory = ComputePairBufferFactory< ComputePairBuffer2<false,false,RhoOpExtStorage> > {}; // make_empty_pair_buffer<RhoOpExtStorage>();
          if( *eam_symmetry )
          {
            auto rho_optional = make_compute_pair_optional_args( nbh_it_sym, cp_weight , cp_xform , cp_locks, ComputePairTrivialCellFiltering{}, ComputePairTrivialParticleFiltering{} );
            SymRhoOp<true,CPLocksT> rho_op { *parameters, eam_scratch->m_pair_enabled.data(), c_particle_offset, rho_emb_ptr, cp_locks };
            compute_cell_particle_pairs( *grid, *rcut, *eam_ghost, rho_optional, rho_buf_factory, rho_op, rho_op_fields, parallel_execution_context() );
          }
          else
          {
            auto rho_optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform , cp_locks, ComputePairTrivialCellFiltering{}, ComputePairTrivialParticleFiltering{} );
            SymRhoOp<false,CPLocksT> rho_op { *parameters, eam_scratch->m_pair_enabled.data(), c_particle_offset, rho_emb_ptr, cp_locks };
            compute_cell_particle_pairs( *grid, *rcut, *eam_ghost, rho_optional, rho_buf_factory, rho_op, rho_op_fields, parallel_execution_context() );
          }
        }
        
        if( *eam_rho2emb )
        {
          Rho2EmbOp rho2emb_op { *parameters , eam_scratch->m_pair_enabled.data() };
          auto rho2emb_op_fields = make_field_tuple_from_field_set( cp_emb_fields , rho_emb_field );
          compute_cell_particles( *grid , *eam_ghost , rho2emb_op , rho2emb_op_fields , parallel_execution_context() );
        }

        // 2nd pass parameters: compute final force using the emb term, only for non ghost particles (but reading EMB terms from neighbor particles)
        if( *eam_force )
        {
          auto vir_field = grid->field_accessor(field::virial);
          using VirFieldT = decltype(vir_field);
          auto cp_force_fields_v = onika::make_flat_tuple( grid->field_accessor(field::type) , vir_field );
          
          if( *eam_symmetry )
          {
            auto force_optional = make_compute_pair_optional_args( nbh_it_sym, cp_weight , cp_xform , cp_locks );
            SymForceOp<true,CPLocksT,VirFieldT> force_op { *parameters, eam_scratch->m_pair_enabled.data(), c_particle_offset, c_rho_emb_ptr , cp_locks , vir_field };
            compute_cell_particle_pairs( *grid, *rcut, false, force_optional, force_buf, force_op, cp_force_fields_v, parallel_execution_context() );
          }
          else
          {
            auto force_optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform , cp_locks );
            SymForceOp<false,CPLocksT,VirFieldT> force_op { *parameters, eam_scratch->m_pair_enabled.data(), c_particle_offset, c_rho_emb_ptr, cp_locks , vir_field };
            compute_cell_particle_pairs( *grid, *rcut, false, force_optional, force_buf, force_op, cp_force_fields_v, parallel_execution_context() );
          }          
        }
      };

      if( need_particle_locks && ! particle_locks.has_value() )
      {
        fatal_error()<<"particle_locks is needed, but corresponding slot has no value" << std::endl;
      }
      
      auto compute_eam_xform_locks = [&](const auto& cp_xform, const auto& cp_locks)
      {
        if( log_energy ) compute_eam_force( cp_xform, cp_locks, cp_emb_fields_energy_v , ComputePairBufferFactory< ComputePairBuffer2<false,false,ForceOpEnergyExt> >{} );
        else             compute_eam_force( cp_xform, cp_locks, cp_emb_fields_v        , ComputePairBufferFactory< ComputePairBuffer2<false,false,ForceOpExt> >{} );
      };

      if( domain->xform_is_identity() )
      {
        if( need_particle_locks ) compute_eam_xform_locks( exanb::NullXForm{} , exanb::ComputePairOptionalLocks<true>{particle_locks->data()} );
        else                      compute_eam_xform_locks( exanb::NullXForm{} , exanb::ComputePairOptionalLocks<false>{} );
      }
      else
      {
        if( need_particle_locks ) compute_eam_xform_locks( exanb::LinearXForm{domain->xform()} , exanb::ComputePairOptionalLocks<true>{particle_locks->data()} );
        else                      compute_eam_xform_locks( exanb::LinearXForm{domain->xform()} , exanb::ComputePairOptionalLocks<false>{} );
      }

    }

  };

  namespace tmplhelper
  {
    template<class GridT> using EamPotentialOperatorName  = ::exaStamp::EamPotentialOperatorName<GridT>;
  }

  // === register factories ===  
  ONIKA_AUTORUN_INIT(eam_potential_multimat)
  {
    OperatorNodeFactory::instance()->register_factory( EamPotentialStr , make_grid_variant_operator< tmplhelper::EamPotentialOperatorName > );
    OperatorNodeFactory::instance()->register_factory( EamParameterInitStr , make_simple_operator< EamParameterInitName > );
    
  }

}

#endif // only compiled if potential supports multimaterial

