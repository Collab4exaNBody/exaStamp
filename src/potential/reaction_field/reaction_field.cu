#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

//  // DO NOT REMOVE THIS LINE
//  // DO NOT REMOVE THIS LINE

#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>

#include <exaStamp/potential/reaction_field/reaction_field.h>

#include <exanb/core/config.h> // for MAX_PARTICLE_NEIGHBORS constant
#include <exanb/particle_neighbors/chunk_neighbors.h>
#include <exanb/core/concurent_add_contributions.h>

namespace exaStamp
{
  using namespace exanb;

  using onika::memory::DEFAULT_ALIGNMENT;

  template<bool _ComputeEnergy, bool _ComputeVirial>
  struct ReactionFieldComputeContext;

  template<> struct ReactionFieldComputeContext<false,false>
  {
    static inline constexpr bool ComputeEnergy = false;
    static inline constexpr bool ComputeVirial = false;
    double charge_a = 0.0;
    Vec3d f = {0.,0.,0.};
  };

  template<> struct ReactionFieldComputeContext<true,false>
  {
    static inline constexpr bool ComputeEnergy = true;
    static inline constexpr bool ComputeVirial = false;
    double charge_a = 0.0;
    Vec3d f = {0.,0.,0.};
    double ep = 0.0;
  };

  template<> struct ReactionFieldComputeContext<true,true>
  {
    static inline constexpr bool ComputeEnergy = true;
    static inline constexpr bool ComputeVirial = true;
    double charge_a = 0.0;
    Vec3d f = {0.,0.,0.};
    double ep = 0.0;
    Mat3d virial = {0.,0.,0.,0.,0.,0.,0.,0.,0.};
  };

  // Reaction Field Compute functor
  template<class CPLocksT, class ChargeFieldT, class TypeFieldT , class VirialFieldT, bool _PerAtomCharge=false, bool _UseSymetry=false, bool _ComputeEnergy = false, bool _ComputeVirial = false>
  struct ReactioFieldForceOp
  {
    static inline constexpr bool PerAtomCharge = _PerAtomCharge;
    static inline constexpr bool ComputeEnergy = _ComputeEnergy;
    static inline constexpr bool ComputeVirial = _ComputeVirial;
    static inline constexpr bool UseSymetry = _UseSymetry;

    static_assert( !ComputeVirial || ComputeEnergy );
    
    // poetential parameters
    const ReactionFieldParms m_params;
    const ParticleSpecie * __restrict__ m_species = nullptr;
    CPLocksT & m_locks;
    ChargeFieldT m_charge_field;
    TypeFieldT m_type_field;
    VirialFieldT m_virial_field;

    using ParticleLockT = decltype( m_locks[0][0] );

    template<class ComputeBufferT, class CellParticlesT>
    ONIKA_HOST_DEVICE_FUNC
    inline void operator () (ComputeBufferT& ctx, CellParticlesT cells, size_t cell_a , size_t p_a, exanb::ComputePairParticleContextStart ) const
    {
      ctx.ext.f = Vec3d{0.,0.,0.};
      if constexpr ( ComputeEnergy )
      {
        ctx.ext.ep = 0.0;
        if constexpr ( ComputeVirial ) ctx.ext.virial = Mat3d{0.,0.,0.,0.,0.,0.,0.,0.,0.};
      }
      if constexpr (  PerAtomCharge ) ctx.ext.charge_a = cells[cell_a][m_charge_field][p_a];
      if constexpr ( !PerAtomCharge ) ctx.ext.charge_a = m_species[ cells[cell_a][field::type][p_a] ].m_charge;
    }

    template<class ComputeBufferT, class CellParticlesT>
    ONIKA_HOST_DEVICE_FUNC
    inline void operator () (ComputeBufferT& ctx, CellParticlesT cells, size_t cell_a, size_t p_a, exanb::ComputePairParticleContextStop ) const
    {
      static constexpr bool CPAA = UseSymetry &&   gpu_device_execution();
      static constexpr bool LOCK = UseSymetry && ! gpu_device_execution();

      if constexpr ( ComputeEnergy && ComputeVirial )
      {
        concurent_add_contributions<ParticleLockT,CPAA,LOCK,double,double,double,double,Mat3d> (
            m_locks[cell_a][p_a]
          , cells[cell_a][field::fx][p_a], cells[cell_a][field::fy][p_a], cells[cell_a][field::fz][p_a], cells[cell_a][field::ep][p_a], cells[cell_a][m_virial_field][p_a]
          , ctx.ext.f.x, ctx.ext.f.y, ctx.ext.f.z, ctx.ext.ep, ctx.ext.virial );
      }
      if constexpr ( ComputeEnergy && !ComputeVirial )
      {
        concurent_add_contributions<ParticleLockT,CPAA,LOCK,double,double,double,double> (
            m_locks[cell_a][p_a]
          , cells[cell_a][field::fx][p_a], cells[cell_a][field::fy][p_a], cells[cell_a][field::fz][p_a], cells[cell_a][field::ep][p_a]
          , ctx.ext.f.x, ctx.ext.f.y, ctx.ext.f.z, ctx.ext.ep );
      }
      if constexpr ( !ComputeEnergy && !ComputeVirial )
      {
        concurent_add_contributions<ParticleLockT,CPAA,LOCK,double,double,double> (
            m_locks[cell_a][p_a]
          , cells[cell_a][field::fx][p_a], cells[cell_a][field::fy][p_a], cells[cell_a][field::fz][p_a]
          , ctx.ext.f.x, ctx.ext.f.y, ctx.ext.f.z );
      }
    }

    template<class ComputeBufferT, class CellParticlesT>
    ONIKA_HOST_DEVICE_FUNC
    inline void operator () (
      ComputeBufferT& ctx, Vec3d dr,double d2,
      CellParticlesT cells,size_t cell_b,size_t p_b, double weight ) const
    {
      static constexpr bool CPAA = UseSymetry &&   gpu_device_execution();
      static constexpr bool LOCK = UseSymetry && ! gpu_device_execution();

      double charge_b = 0.0;
      if constexpr (  PerAtomCharge ) charge_b = cells[cell_b][m_charge_field][p_b];
      if constexpr ( !PerAtomCharge ) charge_b = m_species[ cells[cell_b][field::type][p_b] ].m_charge;

      const double r = std::sqrt(d2);
      double e=0.0, de=0.0;
      reaction_field_compute_energy( m_params, ctx.ext.charge_a * charge_b, r, e, de );
      e *= weight; de *= weight; // weighting function
      de /= r;
      const Vec3d dr_fe = de * dr;
      ctx.ext.f += dr_fe;
      [[maybe_unused]] Mat3d virial;
      
      if constexpr ( ComputeEnergy )
      {
        ctx.ext.ep += .5 * e;
        if constexpr ( ComputeVirial )
        {
          virial = tensor( dr_fe, dr ) * -0.5;
          ctx.ext.virial += virial;
        }
      }

      if constexpr ( UseSymetry )
      {
        if constexpr ( ComputeEnergy && ComputeVirial )
        {
          concurent_add_contributions<ParticleLockT,CPAA,LOCK,double,double,double,double,Mat3d> (
              m_locks[cell_b][p_b]
            , cells[cell_b][field::fx][p_b], cells[cell_b][field::fy][p_b], cells[cell_b][field::fz][p_b], cells[cell_b][field::ep][p_b], cells[cell_b][m_virial_field][p_b]
            , -dr_fe.x, -dr_fe.y , -dr_fe.z, .5*e, virial );
        }
        if constexpr ( ComputeEnergy && !ComputeVirial )
        {
          concurent_add_contributions<ParticleLockT,CPAA,LOCK,double,double,double,double> (
              m_locks[cell_b][p_b]
            , cells[cell_b][field::fx][p_b], cells[cell_b][field::fy][p_b], cells[cell_b][field::fz][p_b], cells[cell_b][field::ep][p_b]
            , -dr_fe.x, -dr_fe.y , -dr_fe.z, .5*e );
        }
        if constexpr ( !ComputeEnergy && !ComputeVirial )
        {
          concurent_add_contributions<ParticleLockT,CPAA,LOCK,double,double,double> (
              m_locks[cell_b][p_b]
            , cells[cell_b][field::fx][p_b], cells[cell_b][field::fy][p_b], cells[cell_b][field::fz][p_b]
            , -dr_fe.x, -dr_fe.y , -dr_fe.z );
        }
      }

    }
    
  };

}

namespace exanb
{
  template<class CPLocksT, class ChargeFieldT, class TypeFieldT , class VirialFieldT, bool _PerAtomCharge, bool _UseSymetry, bool _ComputeEnergy, bool _ComputeVirial>
  struct ComputePairTraits< exaStamp::ReactioFieldForceOp<CPLocksT,ChargeFieldT,TypeFieldT,VirialFieldT,_PerAtomCharge,_UseSymetry,_ComputeEnergy,_ComputeVirial> >
  {
    static inline constexpr bool RequiresBlockSynchronousCall = false;
    static inline constexpr bool ComputeBufferCompatible      = false;
    static inline constexpr bool BufferLessCompatible         = true;
    static inline constexpr bool HasParticleContextStart      = true;    
    static inline constexpr bool HasParticleContext           = true;
    static inline constexpr bool HasParticleContextStop       = true;
    static inline constexpr bool CudaCompatible               = true;
  };
}


namespace exaStamp
{

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz >
    >
  class ReactionFieldPC : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( ReactionFieldParms        , parameters          , INPUT , REQUIRED );
    ADD_SLOT( exanb::GridChunkNeighbors , chunk_neighbors     , INPUT , exanb::GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( CompactGridPairWeights    , compact_nbh_weight  , INPUT , OPTIONAL );
    ADD_SLOT( bool                      , enable_pair_weights , INPUT, true );
    ADD_SLOT( bool                      , per_atom_charge     , INPUT, true );
    ADD_SLOT( bool                      , use_symmetry        , INPUT, false );
    ADD_SLOT( bool                      , compute_virial      , INPUT, false );
    ADD_SLOT( bool                      , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( Domain                    , domain              , INPUT , REQUIRED );
    ADD_SLOT( ParticleSpecies           , species             , INPUT , REQUIRED );    

    ADD_SLOT( GridT                     , grid                , INPUT_OUTPUT );
    ADD_SLOT( double                    , rcut_max            , INPUT_OUTPUT , 0.0 );

    ADD_SLOT( GridParticleLocks         , particle_locks      , INPUT_OUTPUT , OPTIONAL , DocString{"particle spin locks"} );

    // ========= Internal types =======================

    // cell particles array type
    using CellParticles = typename GridT::CellParticles;

  public:
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );

      const double rcut = parameters->rc;
      *rcut_max = std::max( *rcut_max , rcut );
      
      size_t n_cells = grid->number_of_cells();

      // in this case, nothing to compute.
      // this is usefull case where compute_force is called at the very first to initialize rcut_max
      if( n_cells==0 ) return ;

      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
      {
        log_energy = *trigger_thermo_state ;
      }
      else
      {
        ldbg << "trigger_thermo_state missing " << std::endl;
      }
       
      const bool need_particle_locks = ( omp_get_max_threads() > 1 ) && ( *use_symmetry ) ;
      const bool need_virial = log_energy ; // && *compute_virial;
      const bool pair_weights = compact_nbh_weight.has_value() && ( *enable_pair_weights );
      const bool need_ghost = *use_symmetry;

      if( need_particle_locks && ! particle_locks.has_value() )
      {
        fatal_error() << "missing particle_locks value"<<std::endl;
      }

      ldbg << std::boolalpha
           <<"Reaction field: rc="<< rcut
           <<" , pair_weights="<< pair_weights
           <<" , log_energy="<< log_energy
           <<" , need_virial="<< need_virial
           <<" , use_symmetry="<< *use_symmetry
           <<" , ghost="<< need_ghost
           <<" , per_atom_charge="<< *per_atom_charge
           <<" , need_locks="<< need_particle_locks << std::endl;


      using ChargeFieldT = decltype( grid->field_accessor( field::charge ) );
      using VirialFieldT = decltype( grid->field_accessor( field::virial ) );
      using TypeFieldT   = decltype( grid->field_accessor( field::type   ) );      

      ChargeFieldT charge_field = {};
      TypeFieldT   type_field   = {};
      VirialFieldT virial_field = {};

      if( *per_atom_charge )
      {
        charge_field = grid->field_accessor( field::charge );
      }
      else
      {
        type_field = grid->field_accessor( field::type );
      }      
      if( need_virial )
      {
        virial_field = grid->field_accessor( field::virial );
      }
            
      auto compute_force_energy = [&](auto & cp_locks, const auto & cp_weight, const auto & force_op, auto && cpbuf_factory )
      {
        exanb::GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
        LinearXForm cp_xform { domain->xform() };
        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks );
//        [[maybe_unused]] static constexpr onika::parallel::AssertFunctorSizeFitIn< alignof(force_op) , 1 , decltype(force_op) > _check_functor_size = {};
        static constexpr std::true_type use_cells_accessor = {};
        compute_cell_particle_pairs2( *grid, rcut, need_ghost, optional, cpbuf_factory, force_op, onika::FlatTuple<>{}, DefaultPositionFields{}, parallel_execution_context(), use_cells_accessor );
      };
      
      auto compute_force_energy_opt_weights = [&](auto & cp_locks, auto && cp_weight)
      {
        // template<class CPLocksT, class ChargeFieldT, class TypeFieldT , class VirialFieldT, bool _PerAtomCharge=false, bool _UseSymetry=false, bool _ComputeEnergy = false, bool _ComputeVirial = false>
        if( log_energy )
        {
          using ForceOp = ReactioFieldForceOp<decltype(cp_locks),ChargeFieldT,TypeFieldT,VirialFieldT,true,false,true,true>;
          using CPBufT = ComputePairBuffer2<false,false, ReactionFieldComputeContext<true,true> >;
          compute_force_energy( cp_locks, cp_weight, ForceOp{*parameters, species->data(), cp_locks, charge_field, type_field, virial_field} , make_compute_pair_buffer<CPBufT>() );
        }
        else
        {
          using ForceOp = ReactioFieldForceOp<decltype(cp_locks),ChargeFieldT,TypeFieldT,VirialFieldT,true,false,false,false>;
          using CPBufT = ComputePairBuffer2<false,false, ReactionFieldComputeContext<false,false> >;
          compute_force_energy( cp_locks, cp_weight, ForceOp{*parameters, species->data(), cp_locks, charge_field, type_field, virial_field} , make_compute_pair_buffer<CPBufT>() );
        }
      };
      
      auto compute_force_energy_opt_locks = [&](auto && cp_locks)
      {
        if( pair_weights ) compute_force_energy_opt_weights( cp_locks, CompactPairWeightIterator{ compact_nbh_weight->m_cell_weights.data() } );
        else               compute_force_energy_opt_weights( cp_locks, ComputePairNullWeightIterator{} );
      };

      if( need_particle_locks ) compute_force_energy_opt_locks( ComputePairOptionalLocks<false>{} );
      else                      compute_force_energy_opt_locks( ComputePairOptionalLocks<true>{ particle_locks->data() } );      
    }

  };

  template<class GridT> using ReactionFieldPCTmpl = ReactionFieldPC<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(reaction_field)
  {  
    OperatorNodeFactory::instance()->register_factory( "reaction_field" , make_grid_variant_operator<ReactionFieldPCTmpl> );
  }

}


