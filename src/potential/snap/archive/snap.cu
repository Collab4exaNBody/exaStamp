#include "hip/hip_runtime.h"




#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/file_utils.h>

#include <exaStamp/potential/snap/snap_params.h>
#include <exaStamp/potential/snap/snap_read_lammps.h>
#include <exaStamp/potential/snap/snap_config.h>

#include "snapCg.h"
#include "snapBs.h"

#ifdef XNB_CUDA_VERSION
#include <onika/cuda/cuda_context.h>
#include "snap_gpu.h"
#endif

#include <exanb/particle_neighbors/chunk_neighbors.h>

#include <vector>
#include <memory>
#include <iostream>

// this allows for parallel compilation of templated operator for each available field set



namespace exaStamp
{

  using namespace exanb;
  using onika::memory::DEFAULT_ALIGNMENT;
  using namespace SnapExt;

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz >
    >
  class SnapComputeForce : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( SnapParms             , parameters        , INPUT , REQUIRED );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( exanb::GridChunkNeighbors    , chunk_neighbors   , INPUT , exanb::GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                  , ghost             , INPUT , false );
    ADD_SLOT( GridT                 , grid              , INPUT_OUTPUT );
    ADD_SLOT( Domain                , domain            , INPUT , REQUIRED );
    ADD_SLOT( GridParticleLocks     , particle_locks    , INPUT        , OPTIONAL , DocString{"particle spin locks"} );

    // shortcut to the Compute buffer used (and passed to functor) by compute_cell_particle_pairs
    static constexpr bool UseWeights = false;
    static constexpr bool UseNeighbors = true;
    using ComputeBuffer = ComputePairBuffer2<UseWeights,UseNeighbors>;
    using CellParticles = typename GridT::CellParticles;
//   using ParticleLock = decltype( ComputePairOptionalLocks<false>{}[0][0] );

    // compile time constant indicating if grid has virial field
    static constexpr bool has_virial_field = GridHasField<GridT,field::_virial>::value;

    // attributes processed during computation
    using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz >;
    using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_virial >;
    using ComputeFields = std::conditional_t< has_virial_field , ComputeFieldsWithVirial , ComputeFieldsWithoutVirial >;
    static constexpr ComputeFields compute_force_field_set{};
        
  public:
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );

      ldbg << "rcut="<<m_rcut <<std::endl << std::flush;
      if( m_rcut == 0.0 )
      {
        std::string lammps_param = onika::data_file_path( parameters->lammps_param );
        std::string lammps_coef = onika::data_file_path( parameters->lammps_coef ); 
        ldbg << "Snap: read lammps files "<<lammps_param<<" and "<<lammps_coef<<std::endl << std::flush;
        snap_read_lammps(lammps_param, lammps_coef, m_config);
        ldbg <<"rfac0="<<m_config.rfac0() <<", rmin0="<<m_config.rmin0() <<", rcutfac="<<m_config.rcutfac() <<", twojmax="<<m_config.twojmax()<<", bzeroflag="<<m_config.bzeroflag()<<", nmat="<<m_config.materials().size()<<std::endl << std::flush;
        m_rcut = m_config.rcutfac(); // because LAMMPS uses angstrom while exastamp uses nm
	      m_rfac0 = m_config.rfac0();
	      m_rmin0 = m_config.rmin0();	
	      m_bzflag = m_config.bzeroflag();	
      }      

      *rcut_max = std::max( *rcut_max , m_rcut );
      
      size_t n_cells = grid->number_of_cells();
      if( n_cells==0 )
      {
        return ;
      }

      if( m_cg == nullptr )
      {
        m_cg_nt = parameters->nt;
        
        for( const auto& mat : m_config.materials() )
        {
          ldbg << '\t' << mat.name() << ": radelem="<<mat.radelem()<<", weight="<<mat.weight()<<", ncoefs="<<mat.number_of_coefficients()<<std::endl;
          for(size_t i=0;i<mat.number_of_coefficients();i++)
          {
            ldbg << "\t\t" << mat.coefficient(i) << std::endl;
          }
        }
        
        double jmax = m_config.twojmax()*0.5;
        int nmat = m_config.materials().size();
	
        if( nmat != 1 )
        {
          lerr << "Snap: ERROR: only 1 material is allowed" << std::endl;
          std::abort();
        }
      
        const SnapMaterial& mat = m_config.materials()[0];

        // temporay, enable mutiple species if they all have weight=1. modifications needed for true multimaterial
        m_factor.assign( MAX_PARTICLE_SPECIES, 1.0 );
        m_factor[0] = mat.weight();

        m_coefs.resize( mat.number_of_coefficients() );
        for(size_t i=0;i<mat.number_of_coefficients();i++)
        {
          m_coefs[i] = mat.coefficient(i);
          //std::cout<<"coef["<<i<<"] = "<<m_coefs[i]<<std::endl;
        }
        
        ldbg<<"compute Cg with jmax="<<jmax<<", NT="<<m_cg_nt<<std::endl;
        m_cg = std::make_shared<snapCg>( m_config.twojmax()*0.5 , m_cg_nt );
        m_cg->compute();        
      }
          
      size_t nt = omp_get_max_threads();
      if( nt > m_thread_ctx.size() )
      {
        size_t old_nt = m_thread_ctx.size();
        m_thread_ctx.resize( nt );
        for(size_t i=old_nt;i<nt;i++)
        {
          assert( m_thread_ctx[i].m_snapbs == nullptr );
          m_thread_ctx[i].m_snapbs = std::make_shared<snapBs>( m_cg->get_jmax(), *m_cg, m_coefs.data(), m_factor[0] );
        }
      }

      ForceOp force_op { m_thread_ctx, m_rcut, m_rfac0, m_rmin0, m_bzflag };

      ComputePairNullWeightIterator cp_weight{};
      
      exanb::GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();

#     ifdef XNB_CUDA_VERSION
      static SnapGPUContext<SnapExt::CUDA_BLOCK_SIZE,3> snap_gpu_jmax3;
      static SnapGPUContext<SnapExt::CUDA_BLOCK_SIZE,4> snap_gpu_jmax4;
      bool go_gpu = false;
      if( global_cuda_ctx() != nullptr ) go_gpu = global_cuda_ctx()->has_devices() && ( m_cg->get_jmax()==3 || m_cg->get_jmax()==4 );
      if( go_gpu )
      {
#       pragma omp critical(cuda_snap_alloc)
        {
          switch( int( m_cg->get_jmax() ) )
          {
            case 3 :
              if( snap_gpu_jmax3.d_bs_fblock == nullptr ) snap_gpu_jmax3.initialize( * (global_cuda_ctx()) , *(m_thread_ctx[0].m_snapbs) );
              break;
            case 4 :
              if( snap_gpu_jmax4.d_bs_fblock == nullptr ) snap_gpu_jmax4.initialize( * (global_cuda_ctx()) , *(m_thread_ctx[0].m_snapbs) );
              break;
            default:
              std::abort();
              break;
          }
        }

        assert( !m_thread_ctx.empty() );
        ldbg<<"going GPU ..."<<std::endl;

        //ProfilingTimer timer;
        //profiling_timer_start(timer);
        
        LinearXForm cp_xform { domain->xform() };
        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, ComputePairOptionalLocks<false>{}  );
       // cuda_snap_force( snap_gpu_context, *(m_thread_ctx[0].m_snapbs), *grid, m_rcut, m_factor[0], m_coefs[0], m_rfac0, m_rmin0, m_bzflag, *ghost, optional , compute_force_field_set );
        switch( int( m_cg->get_jmax() ) )
        {
          case 3 :
            cuda_snap_force( snap_gpu_jmax3, *(m_thread_ctx[0].m_snapbs), *grid, m_rcut, m_factor[0], m_coefs[0], m_rfac0, m_rmin0, m_bzflag, *ghost, optional , compute_force_field_set );
            snap_gpu_jmax3.synchronize();
            break;
          case 4 :
            cuda_snap_force( snap_gpu_jmax4, *(m_thread_ctx[0].m_snapbs), *grid, m_rcut, m_factor[0], m_coefs[0], m_rfac0, m_rmin0, m_bzflag, *ghost, optional , compute_force_field_set );
            snap_gpu_jmax4.synchronize();
            break;
          default: std::abort(); break;
        }        

        //parallel_execution_context()->account_gpu_execution_time ( profiling_timer_elapsed_restart(timer) );
      }
      else
#     endif
      {
        LinearXForm cp_xform { domain->xform() };
        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, ComputePairOptionalLocks<true>{ particle_locks->data() } );
        compute_cell_particle_pairs( *grid, m_rcut, *ghost, optional, force_buf, force_op, compute_force_field_set, parallel_execution_context()  );
      }

    }

    private:

    struct PerThreadContext
    {
      std::shared_ptr<snapBs> m_snapbs = nullptr;
    };

    // Reaction Field Compute functor
    struct alignas(DEFAULT_ALIGNMENT) ForceOp 
    {
      std::vector<PerThreadContext>& m_thread_ctx;
      const double m_rcut;
      const double m_rfac0;
      const double m_rmin0;
      const size_t m_bzflag;

      template<class CellsAccessorT, class GridCellLocksT, class ParticleLockT>
      inline void operator ()
        (
        size_t n,
        ComputeBuffer& buf,
        double& en,
        double& fx,
        double& fy,
        double& fz,
        CellsAccessorT cells,
        GridCellLocksT locks,
        ParticleLockT& lock_a
        ) const
      {
        FakeMat3d virial;
        this->operator () ( n,buf,en,fx,fy,fz,virial, cells,locks,lock_a );
      }

      template<class CellsAccessorT>
      inline void operator ()
        (
        size_t n,
        ComputeBuffer& buf,
        double& en,
        double& fx,
        double& fy,
        double& fz,
        CellsAccessorT cells
        ) const
      {
        FakeMat3d virial;
        ComputePairOptionalLocks<false> locks = {};
        FakeParticleLock lock_a = {};
        this->operator () ( n,buf,en,fx,fy,fz,virial, cells, locks , lock_a );
      }

      template<class CellsAccessorT>
      inline void operator ()
        (
        size_t n,
        ComputeBuffer& buf,
        double& en,
        double& fx,
        double& fy,
        double& fz,
        Mat3d& virial ,
        CellsAccessorT cells
        ) const
      {
        ComputePairOptionalLocks<false> locks = {};
        FakeParticleLock lock_a = {};
        this->operator () ( n,buf,en,fx,fy,fz,virial, cells, locks , lock_a );
      }

      template<class CellsAccessorT, class Mat3dT,class GridCellLocksT, class ParticleLockT>
      inline void operator ()
        (
        size_t n,
        ComputeBuffer& buf,
        double& en,
        double& fx,
        double& fy,
        double& fz,
        Mat3dT& virial ,
        CellsAccessorT cells,
        GridCellLocksT locks,
        ParticleLockT& lock_a
        ) const
      {
        static constexpr bool compute_virial = std::is_same_v< Mat3dT , Mat3d >;

        size_t tid = omp_get_thread_num();
        assert( tid < m_thread_ctx.size() );
        snapBs& snap_bs = * m_thread_ctx[tid].m_snapbs;

        // energy and force contributions to the particle
        double _en = 0.;
        double _fx = 0.;
        double _fy = 0.;
        double _fz = 0.;

#       pragma omp simd
        for(unsigned int i=0;i<n;++i)
        {
          buf.d2[i] = std::sqrt( buf.d2[i] );
        }

        snap_bs.set_neighbours( buf.drx, buf.dry, buf.drz, buf.d2, m_rcut , n);
        snap_bs.compute_cmm(m_rcut, m_rfac0, m_rmin0);
        snap_bs.compute_bs();
        //snap_bs.compute_bs0();	

        // SNAP energy of the atom
        const double e_tot = snap_bs.en_val();
        double e = 0.;

	      if (m_bzflag == 1)
	      {
	        // SNAP energy zero of the atom
	        const double e_zero = snap_bs.en_zero_val();
	        // SNAP energy of the atom (retrieve e0 = sum b_k B0_k) where B0 is the bispectrum of an isolated atom without neighbors
	        e = (e_tot-e_zero) / n;
	      }
	      else
	      {
	        e = e_tot / n;		
	      }
	
        Mat3dT _vir; // default constructor defines all elements to 0
        //assert( _vir.m11==0 && _vir.m12==0 && _vir.m13==0 && _vir.m21==0 && _vir.m22==0 && _vir.m23==0 && _vir.m31==0 && _vir.m32==0 && _vir.m33==0);

        for(unsigned int i=0;i<n;++i)
        {
          const double3d F = snap_bs.force_val(i);

          auto v_contrib = tensor( Vec3d{F.x,F.y,F.z}, Vec3d{buf.drx[i],buf.dry[i],buf.drz[i]} );

          _fx += F.x;
          _fy += F.y;
          _fz += F.z;
          _en += e;
          _vir += v_contrib * -1.0;

          size_t cell_b=0, p_b=0;
          buf.nbh.get(i, cell_b, p_b);

          auto& lock_b = locks[cell_b][p_b];
          lock_b.lock();
          cells[cell_b][field::fx][p_b] += F.x;
          cells[cell_b][field::fy][p_b] += F.y;
          cells[cell_b][field::fz][p_b] += F.z;
          // cells[cell_b][field::en][p_b] += 0.5 * e // ???
          if constexpr ( compute_virial ) { cells[cell_b][field::virial][p_b] += v_contrib ; }
          lock_b.unlock();
        }

        en += _en; // written only by central atom, no symmetrical contribution
        lock_a.lock();
        fx -= _fx;
        fy -= _fy;
        fz -= _fz;
        virial += _vir;
        lock_a.unlock();
      }
    };

    SnapConfig m_config;
    std::vector<PerThreadContext> m_thread_ctx;
    std::shared_ptr<snapCg> m_cg = nullptr;
    std::vector<double> m_coefs;
    std::vector<double> m_factor;
    double m_rcut = 0.0;
    double m_rfac0 = 0.99363;
    double m_rmin0 = 0.;
    int m_cg_nt = 2;
    bool m_bzflag = true;
  };

  template<class GridT> using SnapComputeForceTmpl = SnapComputeForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(snap)
  {
    OperatorNodeFactory::instance()->register_factory( "snap_force" ,make_grid_variant_operator< SnapComputeForceTmpl > );
  }

}


