#include "hip/hip_runtime.h"
#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>

#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <onika/file_utils.h>

#include <exanb/particle_neighbors/chunk_neighbors.h>
#include <exanb/core/particle_type_id.h>

#include <vector>
#include <memory>
#include <iostream>
//#include <fmt/core.h>

#include <mpi.h>
// #include "cnpy.h"
// #include "wigner/wigner.hpp"
// #include "wigner/wigner_3nj.hpp"


#include "ace-evaluator/ace_c_basis.h"
#include "ace-evaluator/ace_evaluator.h"
#include "ace-evaluator/ace_recursive.h"
#include "ace-evaluator/ace_version.h"
#include "ace/ace_b_basis.h"

#include "pace_params.h"
#include "pace_config.h"
#include "pace_force_op.h"

namespace exaStamp
{

  bool hasExtension(const std::string& filename, const std::string& extension) {
    if (filename.length() >= extension.length()) {
      return std::equal(extension.rbegin(), extension.rend(), filename.rbegin());
    }
    return false;
  }

  static char const *const elements_pace[] = {
    "X",  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si",
    "P",  "S",  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu",
    "Zn", "Ga", "Ge", "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru",
    "Rh", "Pd", "Ag", "Cd", "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr",
    "Nd", "Pm", "Sm", "Eu", "Gd", "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",
    "Re", "Os", "Ir", "Pt", "Au", "Hg", "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac",
    "Th", "Pa", "U",  "Np", "Pu", "Am", "Cm", "Bk", "Cf", "Es", "Fm", "Md", "No", "Lr"};
  static constexpr int elements_num_pace = sizeof(elements_pace) / sizeof(const char *);
  
  static int AtomicNumberByName_pace(const char *elname)
  {
    for (int i = 1; i < elements_num_pace; i++)
      if (strcmp(elname, elements_pace[i]) == 0) return i;
    return -1;
  }
  
  using namespace exanb;
  using onika::memory::DEFAULT_ALIGNMENT;

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz >
    >
  class PaceForce : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( MPI_Comm              , mpi               , INPUT , REQUIRED);
    ADD_SLOT( PaceParams            , parameters        , INPUT , REQUIRED );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( exanb::GridChunkNeighbors , chunk_neighbors   , INPUT , exanb::GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                  , ghost             , INPUT , false );
    ADD_SLOT( bool                  , conv_coef_units   , INPUT , true );
    ADD_SLOT( bool                  , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( GridT                 , grid              , INPUT_OUTPUT );
    ADD_SLOT( Domain                , domain            , INPUT , REQUIRED );
    ADD_SLOT( GridParticleLocks     , particle_locks    , INPUT , OPTIONAL , DocString{"particle spin locks"} );

    ADD_SLOT( long                  , timestep          , INPUT , REQUIRED , DocString{"Iteration number"} );
    //    ADD_SLOT( std::string           , bispectrumchkfile , INPUT , OPTIONAL , DocString{"file with reference values to check bispectrum correctness"} );
    ADD_SLOT( ParticleSpecies       , species           , INPUT        , REQUIRED );
    ADD_SLOT( ParticleTypeMap       , particle_type_map , INPUT        , REQUIRED );    

    ADD_SLOT( PaceContext        , pace_ctx          , PRIVATE );
    ADD_SLOT( bool               , pace_init          , INPUT_OUTPUT, false );

    // shortcut to the Compute buffer used (and passed to functor) by compute_cell_particle_pairs
    static constexpr bool UseWeights = false;
    static constexpr bool UseNeighbors = true;
    //static constexpr bool UseLocks = true;
    //    using ComputeBuffer = ComputePairBuffer2<UseWeights,UseNeighbors>;
    using ComputeBuffer = ComputePairBuffer2<UseWeights,UseNeighbors,PaceComputeBuffer,CopyParticleType>;

    using CellParticles = typename GridT::CellParticles;

    // compile time constant indicating if grid has virial field
    static constexpr bool has_virial_field = GridHasField<GridT,field::_virial>::value;

    // attributes processed during computation
    // using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz >;
    // using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_virial>;
    using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type >;
    using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_virial >;
    using ComputeFields = std::conditional_t< has_virial_field , ComputeFieldsWithVirial , ComputeFieldsWithoutVirial >;
    static constexpr ComputeFields compute_force_field_set{};
    static constexpr FieldSet< field::_type> compute_bispectrum_field_set{};
        
  public:
    
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );
      
      bool recursive = ( (*parameters).pace_algo == "recursive" );
      bool cTildeBasis = false;
      PaceContext PaceCtx = *pace_ctx;
      PaceCtx.aceimpl = new ACEImpl;
      PaceCtx.aceimpl->basis_set = new ACECTildeBasisSet;
      PaceCtx.aceimpl->ace = new ACERecursiveEvaluator();

      ACEBBasisSet bBasisSet;
      ACECTildeBasisSet cTildeBasisSet;

      auto potential_file_name = (*parameters).pace_coef;
      
      if (hasExtension(potential_file_name, ".yaml")) {
        bBasisSet = ACEBBasisSet(potential_file_name);
        cTildeBasisSet = bBasisSet.to_ACECTildeBasisSet();
        *PaceCtx.aceimpl->basis_set = cTildeBasisSet;
        cTildeBasis = true;
      } else {
        cTildeBasis = false;          
        *PaceCtx.aceimpl->basis_set = ACECTildeBasisSet(potential_file_name);
      }
      
      PaceCtx.aceimpl->ace->set_recursive(recursive);
      PaceCtx.aceimpl->ace->element_type_mapping.init((*parameters).nt + 1);
      
      const auto& sp = *species;
      const int n = (*parameters).nt;
      for (int i = 1; i <= n; i++) {
        const char *elemname = sp[i-1].m_name;
        int atomic_number = AtomicNumberByName_pace(elemname);
        if (atomic_number == -1) std::cout << elemname << "is not a valid element" << std::endl;
        SPECIES_TYPE mu = PaceCtx.aceimpl->basis_set->get_species_index_by_name(elemname);
        if (mu != -1) {
          std::cout << "Mapping LAMMPS atom type #"<< i << "("<<elemname<<") -> ACE species type #"<< mu << std::endl;
          PaceCtx.aceimpl->ace->element_type_mapping(i) = mu;
        } else {
          std::cout << "Element "<< elemname << " is not supported by ACE-potential from file " << potential_file_name << std::endl;
        }
      }
      PaceCtx.aceimpl->ace->set_basis(*PaceCtx.aceimpl->basis_set, 1);
      //               *pace_init = true;
      
      double cutoff=0.;
      for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
          *rcut_max = std::max( cutoff, PaceCtx.aceimpl->basis_set->radial_functions->cut(i,j) );
        }
      }

      std::cout << "Total number of basis functions" << std::endl;
      
      for (SPECIES_TYPE mu = 0; mu < PaceCtx.aceimpl->basis_set->nelements; mu++) {
        int n_r1 = PaceCtx.aceimpl->basis_set->total_basis_size_rank1[mu];
        int n = PaceCtx.aceimpl->basis_set->total_basis_size[mu];
        std::cout <<"\t"<< PaceCtx.aceimpl->basis_set->elements_name[mu] << ": "<< n_r1 << " (r=1) " << n << " (r>1)" << std::endl;
      }
      
      size_t n_cells = grid->number_of_cells();
      if( n_cells==0 )
        {
          return ;
        }
      
      if( ! particle_locks.has_value() )
        {
          fatal_error() << "No particle locks" << std::endl;
        }
      
      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
        {
          ldbg << "trigger_thermo_state = " << *trigger_thermo_state << std::endl;
          log_energy = *trigger_thermo_state ;
        }
      else
        {
          ldbg << "trigger_thermo_state missing " << std::endl;
        }
      
      // exanb objects to perform computations with neighbors      
      ComputePairNullWeightIterator cp_weight{};
      exanb::GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();      
      LinearXForm cp_xform { domain->xform() };
      
      auto compute_opt_locks = [&](auto cp_locks)
      {
        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks );
        PaceForceOp force_op { PaceCtx,
                               ! (*conv_coef_units) };
        compute_cell_particle_pairs( *grid, *rcut_max, *ghost, optional, force_buf, force_op , compute_force_field_set , parallel_execution_context() );
      };
      if( omp_get_max_threads() > 1 ) compute_opt_locks( ComputePairOptionalLocks<true>{ particle_locks->data() } );
      else                            compute_opt_locks( ComputePairOptionalLocks<false>{} );
      
    }

  };

  template<class GridT> using PaceForceTmpl = PaceForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(pace)
  {
    OperatorNodeFactory::instance()->register_factory( "pace_force" ,make_grid_variant_operator< PaceForceTmpl > );
  }

}


