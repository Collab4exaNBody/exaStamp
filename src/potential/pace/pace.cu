#include "hip/hip_runtime.h"
#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>

#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <onika/file_utils.h>

#include <exanb/particle_neighbors/chunk_neighbors.h>
#include <exanb/core/particle_type_id.h>

#include <vector>
#include <memory>
#include <iostream>
#include <mpi.h>

#include "ace-evaluator/ace_c_basis.h"
#include "ace-evaluator/ace_evaluator.h"
#include "ace-evaluator/ace_recursive.h"
#include "ace-evaluator/ace_version.h"
#include "ace/ace_b_basis.h"

#include "pace_params.h"
#include "pace_config.h"
#include "pace_force_op.h"

namespace exaStamp
{
  
  using namespace exanb;
  using onika::memory::DEFAULT_ALIGNMENT;
  //  using VecPaceThreadContext = std::vector<PaceThreadContext>;
  
  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz >
    >
  class PaceForce : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( MPI_Comm              , mpi               , INPUT , REQUIRED);
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( exanb::GridChunkNeighbors , chunk_neighbors   , INPUT , exanb::GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                  , ghost             , INPUT , true );
    ADD_SLOT( bool                  , conv_coef_units   , INPUT , true );
    ADD_SLOT( bool                  , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( GridT                 , grid              , INPUT_OUTPUT );
    ADD_SLOT( Domain                , domain            , INPUT , REQUIRED );
    ADD_SLOT( GridParticleLocks     , particle_locks    , INPUT , OPTIONAL , DocString{"particle spin locks"} );

    ADD_SLOT( long                  , timestep          , INPUT , REQUIRED , DocString{"Iteration number"} );
    ADD_SLOT( ParticleSpecies       , species           , INPUT        , REQUIRED );
    ADD_SLOT( ParticleTypeMap       , particle_type_map , INPUT        , REQUIRED );    
    ADD_SLOT( PaceContext           , pace_ctx          , INPUT );
    //    ADD_SLOT( VecPaceThreadContext  , thread_ctx        , PRIVATE );

    // shortcut to the Compute buffer used (and passed to functor) by compute_cell_particle_pairs
    static constexpr bool UseWeights = false;
    static constexpr bool UseNeighbors = true;
    using ComputeBuffer = ComputePairBuffer2<UseWeights,UseNeighbors,PaceComputeBuffer,CopyParticleType>;

    using CellParticles = typename GridT::CellParticles;
    static constexpr bool has_virial_field = GridHasField<GridT,field::_virial>::value;
    
    using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type >;
    using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_virial >;
    using ComputeFields = std::conditional_t< has_virial_field , ComputeFieldsWithVirial , ComputeFieldsWithoutVirial >;
    static constexpr ComputeFields compute_force_field_set{};
        
  public:
    
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );
      size_t nt = omp_get_max_threads();

      // Multi-thread context using std::make_shared<ACEImpl> () and std::shared_ptr<ACEImpl>>
      if( nt > pace_ctx->m_test.size() )
      {
        size_t old_nt = pace_ctx->m_test.size();
        std::cout << "resizing thread context " << std::endl;
        std::cout << "\told size = " << old_nt << ", new size = " << nt << std::endl;
        pace_ctx->m_test.resize( nt );
        for(size_t i=old_nt;i<nt;i++)
          {
            assert( pace_ctx->m_test[i] == nullptr );
            pace_ctx->m_test[i] = std::make_shared<ACEImpl> ();
            pace_ctx->m_test[i]->basis_set = new ACECTildeBasisSet(*(pace_ctx->aceimpl->basis_set));
            pace_ctx->m_test[i]->ace = new ACERecursiveEvaluator();
            pace_ctx->m_test[i]->ace->set_recursive(true);
            pace_ctx->m_test[i]->ace->element_type_mapping.init((*pace_ctx).nspecies + 1);
            for (int j = 1; j <= (*pace_ctx).nspecies; j++) {
              pace_ctx->m_test[i]->ace->element_type_mapping(j) = (*pace_ctx).aceimpl->ace->element_type_mapping(j);
            }
            pace_ctx->m_test[i]->ace->set_basis(*pace_ctx->m_test[i]->basis_set, 1);
          }
      }

      // Multi-thread context using std::vector<PaceThreadContext
      // if( nt > (*thread_ctx).size() )
      // {
      //   size_t old_nt = (*thread_ctx).size();
      //   std::cout << "resizing thread context " << std::endl;
      //   std::cout << "\told size = " << old_nt << ", new size = " << nt << std::endl;
      //   (*thread_ctx).resize( nt );
      //   for(size_t i=old_nt;i<nt;i++)
      //     {
      //       assert( (*thread_ctx)[i].aceimpl == nullptr );
      //       (*thread_ctx)[i].aceimpl = new ACEImpl;
      //       (*thread_ctx)[i].aceimpl->basis_set = new ACECTildeBasisSet(*(pace_ctx->aceimpl->basis_set));
      //       (*thread_ctx)[i].aceimpl->ace = new ACERecursiveEvaluator();
      //       (*thread_ctx)[i].aceimpl->ace->set_recursive(true);
      //       (*thread_ctx)[i].aceimpl->ace->element_type_mapping.init((*pace_ctx).nspecies + 1);
      //       for (int j = 1; j <= (*pace_ctx).nspecies; j++) {
      //         (*thread_ctx)[i].aceimpl->ace->element_type_mapping(j) = (*pace_ctx).aceimpl->ace->element_type_mapping(j);
      //       }
      //       (*thread_ctx)[i].aceimpl->ace->set_basis(*(*thread_ctx)[i].aceimpl->basis_set, 1);
      //     }
      // }
      
      size_t n_cells = grid->number_of_cells();
      if( n_cells==0 )
        {
          return ;
        }
      
      if( ! particle_locks.has_value() )
        {
          fatal_error() << "No particle locks" << std::endl;
        }
      
      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
        {
          ldbg << "trigger_thermo_state = " << *trigger_thermo_state << std::endl;
          log_energy = *trigger_thermo_state ;
        }
      else
        {
          ldbg << "trigger_thermo_state missing " << std::endl;
        }
      bool eflag = log_energy;
      
      ComputePairNullWeightIterator cp_weight{};
      exanb::GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();      
      LinearXForm cp_xform { domain->xform() };
      
      auto compute_opt_locks = [&](auto cp_locks)
      {
        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks );
        PaceForceOp force_op { /* *thread_ctx,*/ pace_ctx->m_test,
                               ! (*conv_coef_units), eflag };
        compute_cell_particle_pairs( *grid, *rcut_max, *ghost, optional, force_buf, force_op , compute_force_field_set , parallel_execution_context() );
      };
      if( omp_get_max_threads() > 1 ) compute_opt_locks( ComputePairOptionalLocks<true>{ particle_locks->data() } );
      else                            compute_opt_locks( ComputePairOptionalLocks<false>{} );
      
    }

  };

  template<class GridT> using PaceForceTmpl = PaceForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(pace)
  {
    OperatorNodeFactory::instance()->register_factory( "pace_force" ,make_grid_variant_operator< PaceForceTmpl > );
  }

}


