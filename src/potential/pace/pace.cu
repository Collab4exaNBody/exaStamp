#include "hip/hip_runtime.h"
#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>

#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <onika/file_utils.h>

#include <exanb/particle_neighbors/chunk_neighbors.h>
#include <exanb/core/particle_type_id.h>

#include <vector>
#include <memory>
#include <iostream>
#include <mpi.h>

#include "ace-evaluator/ace_c_basis.h"
#include "ace-evaluator/ace_evaluator.h"
#include "ace-evaluator/ace_recursive.h"
#include "ace-evaluator/ace_version.h"
#include "ace/ace_b_basis.h"

#include "pace_params.h"
#include "pace_config.h"
#include "pace_force_op.h"

namespace exaStamp
{
  
  using namespace exanb;
  using onika::memory::DEFAULT_ALIGNMENT;

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz >
    >
  class PaceForce : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( MPI_Comm              , mpi               , INPUT , REQUIRED);
    //    ADD_SLOT( PaceParams            , parameters        , INPUT , REQUIRED );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( exanb::GridChunkNeighbors , chunk_neighbors   , INPUT , exanb::GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                  , ghost             , INPUT , true );
    ADD_SLOT( bool                  , conv_coef_units   , INPUT , true );
    ADD_SLOT( bool                  , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( GridT                 , grid              , INPUT_OUTPUT );
    ADD_SLOT( Domain                , domain            , INPUT , REQUIRED );
    ADD_SLOT( GridParticleLocks     , particle_locks    , INPUT , OPTIONAL , DocString{"particle spin locks"} );

    ADD_SLOT( long                  , timestep          , INPUT , REQUIRED , DocString{"Iteration number"} );
    ADD_SLOT( ParticleSpecies       , species           , INPUT        , REQUIRED );
    ADD_SLOT( ParticleTypeMap       , particle_type_map , INPUT        , REQUIRED );    
    ADD_SLOT( PaceContext           , pace_ctx          , INPUT_OUTPUT );

    // shortcut to the Compute buffer used (and passed to functor) by compute_cell_particle_pairs
    static constexpr bool UseWeights = false;
    static constexpr bool UseNeighbors = true;
    using ComputeBuffer = ComputePairBuffer2<UseWeights,UseNeighbors,PaceComputeBuffer,CopyParticleType>;

    using CellParticles = typename GridT::CellParticles;
    static constexpr bool has_virial_field = GridHasField<GridT,field::_virial>::value;
    
    using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type >;
    using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type ,field::_virial >;
    using ComputeFields = std::conditional_t< has_virial_field , ComputeFieldsWithVirial , ComputeFieldsWithoutVirial >;
    static constexpr ComputeFields compute_force_field_set{};
        
  public:
    
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );

      // TODO : multi_thread_context ?
      
      // size_t nt = omp_get_max_threads();
      // if( nt > pace_ctx->m_test.size() )
      // {
      //   size_t old_nt = pace_ctx->m_test.size();
      //   std::cout << "resizing thread context " << std::endl;
      //   std::cout << "\told size = " << old_nt << ", new size = " << nt << std::endl;
      //   pace_ctx->m_test.resize( nt );
      //   for(size_t i=old_nt;i<nt;i++)
      //     {
      //       assert( pace_ctx->m_test[i] == nullptr );
      //       pace_ctx->m_test[i] = std::make_shared<ACEImpl> ();
      //       pace_ctx->m_test[i]->basis_set = pace_ctx->aceimpl->basis_set;
      //       pace_ctx->m_test[i]->ace = pace_ctx->aceimpl->ace;
      //     }
      // }
      
      size_t n_cells = grid->number_of_cells();
      if( n_cells==0 )
        {
          return ;
        }
      
      if( ! particle_locks.has_value() )
        {
          fatal_error() << "No particle locks" << std::endl;
        }
      
      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
        {
          ldbg << "trigger_thermo_state = " << *trigger_thermo_state << std::endl;
          log_energy = *trigger_thermo_state ;
        }
      else
        {
          ldbg << "trigger_thermo_state missing " << std::endl;
        }
      
      ComputePairNullWeightIterator cp_weight{};
      exanb::GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();      
      LinearXForm cp_xform { domain->xform() };
      
      auto compute_opt_locks = [&](auto cp_locks)
      {
        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks );
        PaceForceOp force_op { *pace_ctx, /*pace_ctx->m_test,*/
                               ! (*conv_coef_units) };
        compute_cell_particle_pairs( *grid, *rcut_max, *ghost, optional, force_buf, force_op , compute_force_field_set , parallel_execution_context() );
      };
      if( omp_get_max_threads() > 1 ) compute_opt_locks( ComputePairOptionalLocks<true>{ particle_locks->data() } );
      else                            compute_opt_locks( ComputePairOptionalLocks<false>{} );
      
    }

  };

  template<class GridT> using PaceForceTmpl = PaceForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(pace)
  {
    OperatorNodeFactory::instance()->register_factory( "pace_force" ,make_grid_variant_operator< PaceForceTmpl > );
  }

}


