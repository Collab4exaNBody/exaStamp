#include "hip/hip_runtime.h"
#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>

#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <onika/file_utils.h>
#include <exanb/core/particle_type_id.h>

#include <iostream>

#include "ace-evaluator/ace_c_basis.h"
#include "ace-evaluator/ace_evaluator.h"
#include "ace-evaluator/ace_recursive.h"
#include "ace-evaluator/ace_version.h"
#include "ace/ace_b_basis.h"

#include "pace_params.h"
#include "pace_config.h"

namespace exaStamp
{

  bool hasExtension_bis(const std::string& filename, const std::string& extension) {
    if (filename.length() >= extension.length()) {
      return std::equal(extension.rbegin(), extension.rend(), filename.rbegin());
    }
    return false;
  }

  static char const *const elements_pace_bis[] = {
    "X",  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si",
    "P",  "S",  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu",
    "Zn", "Ga", "Ge", "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru",
    "Rh", "Pd", "Ag", "Cd", "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr",
    "Nd", "Pm", "Sm", "Eu", "Gd", "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",
    "Re", "Os", "Ir", "Pt", "Au", "Hg", "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac",
    "Th", "Pa", "U",  "Np", "Pu", "Am", "Cm", "Bk", "Cf", "Es", "Fm", "Md", "No", "Lr"};
  static constexpr int elements_num_pace = sizeof(elements_pace_bis) / sizeof(const char *);
  
  static int AtomicNumberByName_pace_bis(const char *elname)
  {
    for (int i = 1; i < elements_num_pace; i++)
      if (strcmp(elname, elements_pace_bis[i]) == 0) return i;
    return -1;
  }
  
  using namespace exanb;
  
  class PaceInit : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( PaceParams            , parameters        , INPUT , REQUIRED );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( ParticleSpecies, species, INPUT, REQUIRED );
    ADD_SLOT( PaceContext           , pace_ctx          , OUTPUT );
        
  public:
    
    // Operator execution
    inline void execute () override final
    {

      std::cout << "Entering ACE potential initialization" << std::endl;
      bool recursive = (*parameters).recursive;
      bool cTildeBasis = false;

      (*pace_ctx).aceimpl = new ACEImpl;
      (*pace_ctx).aceimpl->basis_set = new ACECTildeBasisSet;
      (*pace_ctx).aceimpl->ace = new ACERecursiveEvaluator();

      ACEBBasisSet bBasisSet;
      ACECTildeBasisSet cTildeBasisSet;

      auto potential_file_name = (*parameters).pace_coef;
      
      if (hasExtension_bis(potential_file_name, ".yaml")) {
        bBasisSet = ACEBBasisSet(potential_file_name);
        cTildeBasisSet = bBasisSet.to_ACECTildeBasisSet();
        *(*pace_ctx).aceimpl->basis_set = cTildeBasisSet;
        cTildeBasis = true;
      } else {
        cTildeBasis = false;          
        *(*pace_ctx).aceimpl->basis_set = ACECTildeBasisSet(potential_file_name);
      }
      (*pace_ctx).aceimpl->ace->set_recursive(recursive);
      (*pace_ctx).aceimpl->ace->element_type_mapping.init((*parameters).nt + 1);
      
      const auto& sp = *species;
      const int n = (*parameters).nt;
      for (int i = 1; i <= n; i++) {
        const char *elemname = sp[i-1].m_name;
        int atomic_number = AtomicNumberByName_pace_bis(elemname);
        if (atomic_number == -1) std::cout << elemname << "is not a valid element" << std::endl;
        SPECIES_TYPE mu = (*pace_ctx).aceimpl->basis_set->get_species_index_by_name(elemname);
        if (mu != -1) {
          std::cout << "Mapping LAMMPS atom type #"<< i << "("<<elemname<<") -> ACE species type #"<< mu << std::endl;
          (*pace_ctx).aceimpl->ace->element_type_mapping(i) = mu;
        } else {
          std::cout << "Element "<< elemname << " is not supported by ACE-potential from file " << potential_file_name << std::endl;
        }
           }
      (*pace_ctx).aceimpl->ace->set_basis(*(*pace_ctx).aceimpl->basis_set, 1);
      
      double cutoff=0.;
      for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
          *rcut_max = std::max( cutoff, (*pace_ctx).aceimpl->basis_set->radial_functions->cut(i,j) );
        }
      }
      
    }

  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(pace_init)
  {
    OperatorNodeFactory::instance()->register_factory( "pace_init" ,make_simple_operator< PaceInit > );
  }

}


