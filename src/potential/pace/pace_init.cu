#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>

#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <onika/file_utils.h>
#include <exanb/core/particle_type_id.h>

#include <iostream>

#include "ace-evaluator/ace_c_basis.h"
#include "ace-evaluator/ace_evaluator.h"
#include "ace-evaluator/ace_recursive.h"
#include "ace-evaluator/ace_version.h"
#include "ace/ace_b_basis.h"

#include "pace_params.h"
#include "pace_config.h"

namespace exaStamp
{

  bool hasExtension_bis(const std::string& filename, const std::string& extension) {
    if (filename.length() >= extension.length()) {
      return std::equal(extension.rbegin(), extension.rend(), filename.rbegin());
    }
    return false;
  }

  static char const *const elements_pace_bis[] = {
    "X",  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si",
    "P",  "S",  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu",
    "Zn", "Ga", "Ge", "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru",
    "Rh", "Pd", "Ag", "Cd", "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr",
    "Nd", "Pm", "Sm", "Eu", "Gd", "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",
    "Re", "Os", "Ir", "Pt", "Au", "Hg", "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac",
    "Th", "Pa", "U",  "Np", "Pu", "Am", "Cm", "Bk", "Cf", "Es", "Fm", "Md", "No", "Lr"};
  static constexpr int elements_num_pace = sizeof(elements_pace_bis) / sizeof(const char *);
  
  static int AtomicNumberByName_pace_bis(const char *elname)
  {
    for (int i = 1; i < elements_num_pace; i++)
      if (strcmp(elname, elements_pace_bis[i]) == 0) return i;
    return -1;
  }
  
  using namespace exanb;
  
  class PaceInit : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( PaceParams            , parameters        , INPUT , REQUIRED );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0 );
    ADD_SLOT( ParticleSpecies, species, INPUT, REQUIRED );
    ADD_SLOT( PaceContext           , pace_ctx          , OUTPUT );
        
  public:
    
    // Operator execution
    inline void execute () override final
    {

      std::cout << "Entering ACE potential initialization" << std::endl;
      bool cTildeBasis = false;
      (*pace_ctx).recursive = (*parameters).recursive;
      (*pace_ctx).aceimpl = new ACEImpl;
      (*pace_ctx).aceimpl->basis_set = new ACECTildeBasisSet;
      (*pace_ctx).aceimpl->ace = new ACERecursiveEvaluator();

      ACEBBasisSet bBasisSet;
      ACECTildeBasisSet cTildeBasisSet;

      auto potential_file_name = (*parameters).pace_coef;
      const auto& sp = *species;
      const int nspecies = sp.size();
      (*pace_ctx).nspecies = nspecies;
      
      if (hasExtension_bis(potential_file_name, ".yaml")) {
        bBasisSet = ACEBBasisSet(potential_file_name);
        cTildeBasisSet = bBasisSet.to_ACECTildeBasisSet();
        *(*pace_ctx).aceimpl->basis_set = cTildeBasisSet;
        cTildeBasis = true;
      } else {
        cTildeBasis = false;          
        *(*pace_ctx).aceimpl->basis_set = ACECTildeBasisSet(potential_file_name);
      }
      (*pace_ctx).aceimpl->ace->set_recursive((*pace_ctx).recursive);
      (*pace_ctx).aceimpl->ace->element_type_mapping.init((*pace_ctx).nspecies + 1);
      
      for (int i = 1; i <= nspecies; i++) {
        const char *elemname = sp[i-1].m_name;
        int atomic_number = AtomicNumberByName_pace_bis(elemname);
        if (atomic_number == -1) std::cout << elemname << "is not a valid element" << std::endl;
        SPECIES_TYPE mu = (*pace_ctx).aceimpl->basis_set->get_species_index_by_name(elemname);
        if (mu != -1) {
          std::cout << "Mapping LAMMPS atom type #"<< i << "("<<elemname<<") -> ACE species type #"<< mu << std::endl;
          (*pace_ctx).aceimpl->ace->element_type_mapping(i) = mu;
        } else {
          std::cout << "Element "<< elemname << " is not supported by ACE-potential from file " << potential_file_name << std::endl;
        }
           }
      (*pace_ctx).aceimpl->ace->set_basis(*(*pace_ctx).aceimpl->basis_set, 1);
      
      double cutoff=0.;
      for (int i = 0; i < nspecies; i++) {
        for (int j = 0; j < nspecies; j++) {
          *rcut_max = std::max( cutoff, (*pace_ctx).aceimpl->basis_set->radial_functions->cut(i,j) );
        }
      }
      
    }

  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(pace_init)
  {
    OperatorNodeFactory::instance()->register_factory( "pace_init" ,make_simple_operator< PaceInit > );
  }

}


