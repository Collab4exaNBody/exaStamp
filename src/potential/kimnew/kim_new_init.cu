#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <onika/math/basic_types_stream.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/domain.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include "kim_new.h"

namespace exaStamp
{

  using namespace exanb;
  using namespace KIM;
  
  class KIMInitOperator : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( KIMParams,   parameters    , INPUT_OUTPUT , REQUIRED );
    ADD_SLOT( double,      rcut_max      , INPUT_OUTPUT );    
    ADD_SLOT( KIMContext,  kim_ctx       , OUTPUT );
    
  public:
    // Operator execution
    inline void execute () override final
    {
      std::cout << "======= KIM Initialization ======" << std::endl;
      KIM::Log::PushDefaultVerbosity(KIM::LOG_VERBOSITY::silent);

      KIM::Model * kim_local_model;
      int requestedUnitsAccepted;
      int error = KIM::Model::Create(KIM::NUMBERING::zeroBased,
                                     KIM::LENGTH_UNIT::A,
                                     KIM::ENERGY_UNIT::eV,
                                     KIM::CHARGE_UNIT::e,
                                     KIM::TEMPERATURE_UNIT::K,
                                     KIM::TIME_UNIT::ps,
                                     parameters->model,
                                     &requestedUnitsAccepted,
                                     &kim_local_model);
      if (error) { MY_ERROR("KIM::Model::Create()"); }
      
      // Check for compatibility with the model
      if (!requestedUnitsAccepted) { MY_ERROR("Must Adapt to model units"); }

      // Check that we know about all required routines
      int numberOfModelRoutineNames;
      KIM::MODEL_ROUTINE_NAME::GetNumberOfModelRoutineNames(&numberOfModelRoutineNames);
          
      for (int i = 0; i < numberOfModelRoutineNames; ++i)
        {
          KIM::ModelRoutineName modelRoutineName;
          int error
            = KIM::MODEL_ROUTINE_NAME::GetModelRoutineName(i, &modelRoutineName);
          if (error) { MY_ERROR("Unable to get ModelRoutineName."); }
          int present;
          int required;
          error = kim_local_model->IsRoutinePresent(modelRoutineName, &present, &required);
          if (error) { MY_ERROR("Unable to get routine present/required."); }

          std::cout << "Model routine name \"" << modelRoutineName.ToString()
                    << "\" has present = " << present
                    << " and required = " << required << "." << std::endl;

          if ((present == true) && (required == true))
            {
              using namespace KIM::MODEL_ROUTINE_NAME;
              if (!((modelRoutineName == Create)
                    || (modelRoutineName == ComputeArgumentsCreate)
                    || (modelRoutineName == Compute) || (modelRoutineName == Refresh)
                    || (modelRoutineName == ComputeArgumentsDestroy)
                    || (modelRoutineName == Destroy)))
                {
                  MY_ERROR("Unknown Routine \"" + modelRoutineName.ToString()
                           + "\" is required by model.");
                }
            }
        }

      // print model units
      KIM::LengthUnit lengthUnit;
      KIM::EnergyUnit energyUnit;
      KIM::ChargeUnit chargeUnit;
      KIM::TemperatureUnit temperatureUnit;
      KIM::TimeUnit timeUnit;

      kim_local_model->GetUnits(&lengthUnit, &energyUnit, &chargeUnit, &temperatureUnit, &timeUnit);

      std::cout << "\nLengthUnit is \"" << lengthUnit.ToString() << "\"" << std::endl
                << "EnergyUnit is \"" << energyUnit.ToString() << "\"" << std::endl
                << "ChargeUnit is \"" << chargeUnit.ToString() << "\"" << std::endl
                << "TemperatureUnit is \"" << temperatureUnit.ToString() << "\""
                << std::endl
                << "TimeUnit is \"" << timeUnit.ToString() << "\"" << std::endl;

      // check species
      int speciesIsSupported;
      int modelTaCode;
      error = kim_local_model->GetSpeciesSupportAndCode(KIM::SPECIES_NAME::Ta, &speciesIsSupported, &modelTaCode);
      if ((error) || (!speciesIsSupported))
        {
          MY_ERROR("Species Ta not supported");
        }

      KIM::ComputeArguments * computeArguments;
      error = kim_local_model->ComputeArgumentsCreate(&computeArguments);
      if (error) { MY_ERROR("Unable to create a ComputeArguments object."); }

      const double* cutoffs;
      double influencedistance;
      
      const int* modelWillNotRequestNeighborsOfNoncontributingParticles;
      int numberOfNeighborLists;
      kim_local_model->GetNeighborListPointers(&numberOfNeighborLists,
                                               &cutoffs,
                                               &modelWillNotRequestNeighborsOfNoncontributingParticles);
      
      for (int i=0; i<numberOfNeighborLists;i++) {
        std::cout << "cutoff i = " << cutoffs[i] << std::endl;
        kim_ctx->rcut = std::max(kim_ctx->rcut, cutoffs[i]);
        
      }
      
      std::cout << "Model cutoff radius = " << kim_ctx->rcut << std::endl;
      kim_local_model->GetInfluenceDistance( &influencedistance);
      parameters->rcut = influencedistance;
      
      std::cout << "Model influence distance = " << influencedistance << std::endl;
      // Replace rcut_max by influence distance
      KIM::Log::PopDefaultVerbosity();
      std::cout << "=================================\n" << std::endl;
    }
    
    //    (*parameters).rcut = *rcut;    
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(kim_new_init)
  {  
    OperatorNodeFactory::instance()->register_factory( "kim_new_init" , make_simple_operator< KIMInitOperator > );
  }

}
