#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <onika/math/basic_types.h>
#include <onika/math/basic_types_operators.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <onika/cpp_utils.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/file_utils.h>

#include "kim_new.h"
#include "kim_new_force_op.h"

#include <onika/physics/units.h>
#include <onika/physics/constants.h>
#include <onika/cuda/cuda_context.h>

#include <vector>
#include <memory>
#include <iostream>

namespace exaStamp
{
  using onika::memory::DEFAULT_ALIGNMENT;
  
  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type >
    >
  class KIMNewComputeForce : public OperatorNode
  {

    using CellParticles = typename GridT::CellParticles;
    
    // ========= I/O slots =======================
    ADD_SLOT( KIMParams             , parameters        , INPUT        , REQUIRED );
    ADD_SLOT( double                , rcut_max          , INPUT_OUTPUT , 0.0      );
    ADD_SLOT( ParticleSpecies       , species           , INPUT        , REQUIRED );
    ADD_SLOT( GridChunkNeighbors    , chunk_neighbors   , INPUT        , GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                  , ghost             , INPUT        , false    );
    ADD_SLOT( GridT                 , grid              , INPUT_OUTPUT            );
    ADD_SLOT( Domain                , domain            , INPUT        , REQUIRED );
    ADD_SLOT( KIMContext            , kim_ctx           , INPUT );    
    ADD_SLOT( GridParticleLocks     , particle_locks    , INPUT , OPTIONAL , DocString{"particle spin locks"} );
    // shortcut to the Compute buffer used (and passed to functor) by compute_cell_particle_pairs
    //    using ComputeBuffer = ComputePairBuffer2<false,false>;
    static constexpr bool UseWeights = false;
    static constexpr bool UseNeighbors = true;
    using ComputeBuffer = ComputePairBuffer2<UseWeights,UseNeighbors>;
    // compile time constant indicating if grid has virial field
    static constexpr bool has_virial_field = GridHasField<GridT,field::_virial>::value;

    // attributes processed during computation
    using ComputeFieldsWithoutVirial = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type >;
    using ComputeFieldsWithVirial    = FieldSet< field::_ep ,field::_fx ,field::_fy ,field::_fz ,field::_type, field::_virial >;
    using ComputeFields              = std::conditional_t< has_virial_field , ComputeFieldsWithVirial , ComputeFieldsWithoutVirial >;
    static constexpr ComputeFields compute_force_field_set{};
    
  public:
    // Operator execution
    inline void execute () override final
    {
      KIM::Log::PushDefaultVerbosity(KIM::LOG_VERBOSITY::silent);

      ldbg << "KIM model = " << parameters->model << std::endl;
      ldbg << "KIM rcut  = " << parameters->rcut << std::endl;      
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );
      size_t nt = omp_get_max_threads();
      if (nt > kim_ctx->m_thread_ctx.size()) {
        size_t old_nt = kim_ctx->m_thread_ctx.size();
        kim_ctx->m_thread_ctx.resize( nt );
        int requestedUnitsAccepted;
        int error;
        for(size_t j=old_nt;j<nt;j++)
          {
            assert( kim_ctx->m_thread_ctx[j].kim_model == nullptr );
            error = KIM::Model::Create(KIM::NUMBERING::zeroBased,
                                       KIM::LENGTH_UNIT::A,
                                       KIM::ENERGY_UNIT::eV,
                                       KIM::CHARGE_UNIT::e,
                                       KIM::TEMPERATURE_UNIT::K,
                                       KIM::TIME_UNIT::ps,
                                       parameters->model,
                                       &requestedUnitsAccepted,
                                       &kim_ctx->m_thread_ctx[j].kim_model);
            if (error) { MY_ERROR("KIM::Model::Create()"); }
            // Check for compatibility with the model
            if (!requestedUnitsAccepted) { MY_ERROR("Must Adapt to model units"); }


            // Check that we know about all required routines
            int numberOfModelRoutineNames;
            KIM::MODEL_ROUTINE_NAME::GetNumberOfModelRoutineNames(&numberOfModelRoutineNames);
      
            for (int i = 0; i < numberOfModelRoutineNames; ++i)
              {
                KIM::ModelRoutineName modelRoutineName;
                int error = KIM::MODEL_ROUTINE_NAME::GetModelRoutineName(i, &modelRoutineName);
                if (error) { MY_ERROR("Unable to get ModelRoutineName."); }
                int present;
                int required;
                error = kim_ctx->m_thread_ctx[j].kim_model->IsRoutinePresent(modelRoutineName, &present, &required);
                if (error) { MY_ERROR("Unable to get routine present/required."); }

                ldbg << "Model routine name \"" << modelRoutineName.ToString()
                     << "\" has present = " << present
                     << " and required = " << required << "." << std::endl;

                if ((present == true) && (required == true))
                  {
                    using namespace KIM::MODEL_ROUTINE_NAME;
                    if (!((modelRoutineName == Create)
                          || (modelRoutineName == ComputeArgumentsCreate)
                          || (modelRoutineName == Compute) || (modelRoutineName == Refresh)
                          || (modelRoutineName == ComputeArgumentsDestroy)
                          || (modelRoutineName == Destroy)))
                      {
                        MY_ERROR("Unknown Routine \"" + modelRoutineName.ToString()
                                 + "\" is required by model.");
                      }
                  }
              }

            // print model units
            KIM::LengthUnit lengthUnit;
            KIM::EnergyUnit energyUnit;
            KIM::ChargeUnit chargeUnit;
            KIM::TemperatureUnit temperatureUnit;
            KIM::TimeUnit timeUnit;

            kim_ctx->m_thread_ctx[j].kim_model->GetUnits(&lengthUnit, &energyUnit, &chargeUnit, &temperatureUnit, &timeUnit);

            ldbg << "LengthUnit is \"" << lengthUnit.ToString() << "\"" << std::endl
                 << "EnergyUnit is \"" << energyUnit.ToString() << "\"" << std::endl
                 << "ChargeUnit is \"" << chargeUnit.ToString() << "\"" << std::endl
                 << "TemperatureUnit is \"" << temperatureUnit.ToString() << "\""
                 << std::endl
                 << "TimeUnit is \"" << timeUnit.ToString() << "\"" << std::endl;

            // check species
            int speciesIsSupported;
            int modelTaCode;
            error = kim_ctx->m_thread_ctx[j].kim_model->GetSpeciesSupportAndCode(KIM::SPECIES_NAME::Ta, &speciesIsSupported, &modelTaCode);
            if ((error) || (!speciesIsSupported))
              {
                MY_ERROR("Species Ta not supported");
              }

            KIM::ComputeArguments * computeArguments;
            error = kim_ctx->m_thread_ctx[j].kim_model->ComputeArgumentsCreate(&computeArguments);
            if (error) { MY_ERROR("Unable to create a ComputeArguments object."); }
          }
      }
          
      *rcut_max = std::max( *rcut_max , parameters->rcut );

      size_t n_cells = grid->number_of_cells();
      if( n_cells == 0 )
      {
        return ;
      }
		
      if( ! particle_locks.has_value() )
      {
        fatal_error() << "No particle locks" << std::endl;
      }
      
      ComputePairNullWeightIterator cp_weight{};
      GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();
      LinearXForm cp_xform { domain->xform() };

      auto compute_opt_locks = [&](auto cp_locks)
      {
        auto optional = make_compute_pair_optional_args( nbh_it, cp_weight , cp_xform, cp_locks );
        KimNewForceOp force_op { kim_ctx->m_thread_ctx.data() };
        compute_cell_particle_pairs( *grid, parameters->rcut, *ghost, optional, force_buf, force_op , compute_force_field_set , parallel_execution_context() );
      };
      if( omp_get_max_threads() > 1 ) {
        compute_opt_locks( ComputePairOptionalLocks<true>{ particle_locks->data() } );
      } else {
        compute_opt_locks( ComputePairOptionalLocks<false>{} );
      }
      
    }

  };

  template<class GridT> using KIMNewComputeForceTmpl = KIMNewComputeForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(kim_new_force)
  {
    OperatorNodeFactory::instance()->register_factory( "kim_new_force" ,make_grid_variant_operator< KIMNewComputeForceTmpl > );
  }

}


