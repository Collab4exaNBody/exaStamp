#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

//  // DO NOT REMOVE THIS LINE

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>

#include <exaStamp/molecule/bonds_force_functor.h>

#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/math/basic_types_yaml.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/log.h>

#include <string>
#include <iostream>
#include <sstream>

namespace exaStamp
{

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_ep, field::_fx, field::_fy, field::_fz >
    >
  class ComputeForcesBondsNode : public OperatorNode
  {

    // -----------------------------------------------
    // Operator slots
    // -----------------------------------------------
    ADD_SLOT( Domain                  , domain                , INPUT );
    ADD_SLOT( ChemicalBonds           , chemical_bonds        , INPUT, OPTIONAL );
    ADD_SLOT( ParticleSpecies         , species               , INPUT, REQUIRED );
    ADD_SLOT( BondsPotentialParameters, potentials_for_bonds  , INPUT_OUTPUT, REQUIRED );
    ADD_SLOT( GridT                   , grid                  , INPUT_OUTPUT );
    ADD_SLOT( GridParticleLocks       , particle_locks        , INPUT_OUTPUT);

    ADD_SLOT( MoleculeComputeParameterSet  , molecule_compute_parameters , INPUT_OUTPUT, DocString{"Intramolecular functionals' parameters"} );
    ADD_SLOT( IntramolecularParameterIndexLists, intramolecular_parameters , INPUT_OUTPUT, DocString{"Intramolecular functional parmater index lists"} );

    ADD_SLOT( bool                      , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( bool                      , compute_virial      , INPUT , false );

  public:
    inline void execute() override final
    {
      using onika::parallel::parallel_for;

      if( ! grid.has_value() || grid->number_of_cells()==0 )
      {
        return;
      }

      if( ! chemical_bonds.has_value() )
      {
        lerr << "chemical_bonds input missing" << std::endl;
        std::abort();
      }

      // do we need to compute energy and virial
      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
      {
        log_energy = *trigger_thermo_state ;
      }
      else
      {
        ldbg << "trigger_thermo_state missing " << std::endl;
      }
      const bool need_virial = log_energy && *compute_virial;
      using VirialFieldT = decltype( grid->field_accessor( field::virial ) );
      VirialFieldT virial_field = {};
      if( need_virial ) virial_field = grid->field_accessor( field::virial );
      
      const Vec3d size_box {std::abs(domain->extent().x - domain->origin().x),
                      std::abs(domain->extent().y - domain->origin().y),
                      std::abs(domain->extent().z - domain->origin().z)};
      const double half_min_size_box = std::min( std::min(size_box.x,size_box.y) , size_box.z) / 2.0; 

#     ifndef NDEBUG
      Vec3d tmp = domain->xform() * size_box;
      if( fabs(tmp.x) <= 1.5 || fabs(tmp.y) <= 1.5 || fabs(tmp.z) <= 1.5 )
      {
        fatal_error()<<"xform="<<domain->xform()<<", size_box="<<size_box<<", tmp="<<tmp<<std::endl;
      }
#     endif

      ldbg<<"n_bonds = "<<chemical_bonds->size()<<std::endl;

      auto compute_bonds_force_opt_virial = [&]( auto cpvir )
      {
        auto cells = grid->cells_accessor();
        BondForceOp<decltype(cells),VirialFieldT,cpvir.value> bond_force_op =
          { cells
          , particle_locks->data()
          , molecule_compute_parameters->m_func_params.data()
          , intramolecular_parameters->m_bond_param_idx.data()
          , chemical_bonds->data()
          , size_box , half_min_size_box
          , domain->xform() , domain->xform_is_identity()
          , virial_field };

       /* auto parallel_op = */ parallel_for( chemical_bonds->size(), bond_force_op, parallel_execution_context("bond_force") );
        //auto exec_ctrl_obj = parallel_execution_stream() << std::move(parallel_op) ;
      };

      if( need_virial ) compute_bonds_force_opt_virial( onika::TrueType{} );
      else              compute_bonds_force_opt_virial( onika::FalseType{} );

      ldbg<<"compute_force_bond done"<<std::endl;
    }

  };

  template<class GridT> using ComputeForcesBondsNodeTmpl = ComputeForcesBondsNode<GridT>;

  // === register factories ===
  ONIKA_AUTORUN_INIT(compute_forces_molecules_bonds)
  {
    OperatorNodeFactory::instance()->register_factory( "compute_force_bond", make_grid_variant_operator< ComputeForcesBondsNodeTmpl > );
  }

}

