#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

//  // DO NOT REMOVE THIS LINE

#include <yaml-cpp/yaml.h>
#include <memory>
#include <utility>// std::pair

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/math/basic_types_yaml.h>
#include <exaStamp/molecule/mol_connectivity.h>
#include <exanb/core/particle_id_codec.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/log.h>
#include <onika/thread.h>  // GridParticleLocks

#include <exaStamp/molecule/torsions_potentials_parameters.h>
#include <exaStamp/molecule/periodic_r_delta.h>
#include <exaStamp/molecule/torsions_force_functor.h>

namespace exaStamp
{

  ///**
  // * Signum function : give the sign of an expression (+1, -1 or 0)
  // */
  //template <typename T>
  //typename std::enable_if<std::is_unsigned<T>::value, int>::type
  //inline constexpr signum(T x) noexcept {
  //  return T(0) < x;
  //}

  //template <typename T>
  //typename std::enable_if<std::is_signed<T>::value, int>::type
  //inline constexpr signum(T x) noexcept {
  //  return (T(0) < x) - (x < T(0));
  //}


  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_fx, field::_fy, field::_fz, field::_ep >
    >
  class ComputeForcesTorsionsNode : public OperatorNode
  {
    // -----------------------------------------------
    // Operator slots
    // -----------------------------------------------
    ADD_SLOT( GridT                       , grid                    , INPUT_OUTPUT );
    ADD_SLOT( Domain                      , domain                  , INPUT );
    ADD_SLOT( ChemicalTorsions            , chemical_torsions       , INPUT, OPTIONAL );
    ADD_SLOT( TorsionsPotentialParameters , potentials_for_torsions , INPUT, REQUIRED );
    ADD_SLOT( ParticleSpecies             , species                 , INPUT, REQUIRED );
    
    ADD_SLOT( GridParticleLocks        , particle_locks        , INPUT_OUTPUT);

    ADD_SLOT( MoleculeComputeParameterSet  , molecule_compute_parameters , INPUT_OUTPUT, DocString{"Intramolecular functionals' parameters"} );
    ADD_SLOT( IntramolecularParameterIndexLists, intramolecular_parameters , INPUT_OUTPUT, DocString{"Intramolecular functional parmater index lists"} );

    ADD_SLOT( bool                      , trigger_thermo_state, INPUT , OPTIONAL );
    ADD_SLOT( bool                      , compute_virial      , INPUT , false );

    inline void execute ()  override final
    {
      if( ! grid.has_value() || grid->number_of_cells()==0 )
      {
        return;
      }

      if( ! chemical_torsions.has_value() )
      {
        fatal_error() << "chemical_torsions input missing" << std::endl;
      }
      
      // do we need to compute energy and virial
      bool log_energy = false;
      if( trigger_thermo_state.has_value() )
      {
        log_energy = *trigger_thermo_state ;
      }
      else
      {
        ldbg << "trigger_thermo_state missing " << std::endl;
      }
      const bool need_virial = log_energy && *compute_virial;
      using VirialFieldT = decltype( grid->field_accessor( field::virial ) );
      VirialFieldT virial_field = {};
      if( need_virial ) virial_field = grid->field_accessor( field::virial );

      ldbg<<"n_torsions = "<<chemical_torsions->size()<<std::endl;

      const Vec3d size_box {std::abs(domain->extent().x - domain->origin().x),
                      std::abs(domain->extent().y - domain->origin().y),
                      std::abs(domain->extent().z - domain->origin().z)};
      const double half_min_size_box = std::min( std::min(size_box.x,size_box.y) , size_box.z) / 2.0; 

#     ifndef NDEBUG
      if( ! domain->xform_is_identity() )
      {
        Vec3d tmp = domain->xform() * size_box;
        if( fabs(tmp.x) <= 1.5 || fabs(tmp.y) <= 1.5 || fabs(tmp.z) <= 1.5 )
        {
          lerr<<"xform="<<domain->xform()<<", size_box="<<size_box<<", tmp="<<tmp<<std::endl;
          std::abort();
        }
      }
#     endif

      auto compute_torsions_force_opt_virial = [&]( auto cpvir )
      {
        auto cells = grid->cells_accessor();
        TorsionForceOp<decltype(cells),VirialFieldT,cpvir.value> torsion_force_op =
          { cells
          , particle_locks->data()
          , molecule_compute_parameters->m_func_params.data()
          , intramolecular_parameters->m_torsion_param_idx.data()
          , chemical_torsions->data()
          , size_box , half_min_size_box
          , domain->xform() , domain->xform_is_identity()
          , virial_field };

        /* auto parallel_op = */ parallel_for( chemical_torsions->size(), torsion_force_op, parallel_execution_context("torsion_force") );
        // auto exec_ctrl_obj = parallel_execution_stream() << std::move(parallel_op) ;
      };

      if( need_virial ) compute_torsions_force_opt_virial( onika::TrueType{} );
      else              compute_torsions_force_opt_virial( onika::FalseType{} );

      ldbg<<"compute_force_torsion done"<<std::endl;
    }

  };


  template<class GridT> using ComputeForcesTorsionsNodeTmpl = ComputeForcesTorsionsNode<GridT>;

  // === register factories ===
  ONIKA_AUTORUN_INIT(compute_forces_molecules_torsions)
  {
    OperatorNodeFactory::instance()->register_factory( "compute_force_torsion", make_grid_variant_operator< ComputeForcesTorsionsNodeTmpl> );
  }

}


