#include "hip/hip_runtime.h"
//#include <chrono>
#include <memory>

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <exanb/core/grid.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/particle_id_codec.h>
#include <exaStamp/molecule/id_map.h>
#include <exaStamp/molecule/periodic_r_delta.h>

#include <mpi.h>
#include <exanb/mpi/grid_update_ghosts.h>

#include <exanb/core/grid_fields.h>
XNB_DECLARE_FIELD( exanb::Vec3d, molufpos, "molecule unfolded particle position" );

namespace exaStamp
{

  template<typename GridT
    , class = AssertGridHasFields< GridT, field::_id >
    >
  class MoleculeUnfold : public OperatorNode
  {
    using UpdateGhostsScratch = typename UpdateGhostsUtils::UpdateGhostsScratch;

    ADD_SLOT( MPI_Comm                 , mpi               , INPUT , MPI_COMM_WORLD );
    ADD_SLOT( long                     , mpi_tag           , INPUT , 0 );

    ADD_SLOT( Domain                   , domain            , INPUT );
    ADD_SLOT( GridT       , grid       , INPUT_OUTPUT );

    ADD_SLOT( IdMap       , id_map        , INPUT , REQUIRED );
    ADD_SLOT( IdMapGhosts , id_map_ghosts , INPUT , REQUIRED );

    ADD_SLOT( UpdateGhostsScratch      , ghost_comm_buffers, PRIVATE );
    ADD_SLOT( GhostCommunicationScheme , ghost_comm_scheme , INPUT_OUTPUT , REQUIRED );

  public:
    inline void execute ()  override final
    {
      auto cells = grid->cells_accessor();
      size_t n_cells = grid->number_of_cells();
      const auto idmol_field = grid->field_accessor( field::idmol );
      const auto cmol_field = grid->field_accessor( field::cmol );
      static constexpr uint64_t null_id = std::numeric_limits<uint64_t>::max();
      static constexpr uint64_t null_loc = std::numeric_limits<uint64_t>::max();
      static constexpr size_t null_index = std::numeric_limits<size_t>::max();

      // basic assumption
      assert( domain->origin() == grid->origin() );

      // those two lambdas are used to launch parallel kernels within a function exterior to this OperatorNode's implementation
      auto pecfunc = [self=this](auto ... args) { return self->parallel_execution_context(args ...); };
      //      auto pesfunc = [self=this](unsigned int i) { return self->parallel_execution_stream(i); };
      auto peqfunc = [self=this]() -> onika::parallel::ParallelExecutionQueue& { return self->parallel_execution_queue(); };

      // map of molecule id (identical to owner's particle id) to owner particle location ( encoded cell / position in cell )
      std::unordered_map<uint64_t , uint64_t> molecule_owner;
      
      // maximum molecule bond distance found in dataset
      double bond_max_dist = 0.0;

      // preamble : assign particle's molid particles' ids, so that we start with individual molecules
      // of one atom each, and we will then merge those bound by intramolecular bond
      for(size_t cell_i=0;cell_i<n_cells;cell_i++)
      {
        size_t n = cells[cell_i].size();
        for(size_t p_i=0;p_i<n;p_i++) cells[cell_i][idmol_field][p_i] = cells[cell_i][field::id][p_i];
      }
      ldbg << "molecule_unfold: id_map @"<<id_map.get_pointer()<<" size = "<<id_map->size()<<" , id_map_ghosts @ "<<id_map_ghosts.get_pointer() <<" size = "<<id_map_ghosts->size() <<std::endl;

      // first multi pass algorithm : propagate molecule ids through bonds, keeping only the minimum id a the molecule id
      long update_count = 0;
      int pass = 0;
      do
      {
        const auto ghost_update_fields = onika::make_flat_tuple( idmol_field ); 
        grid_update_ghosts( exanb::ldbg, *mpi, *ghost_comm_scheme, *grid, *domain, nullptr,
                            *ghost_comm_buffers, pecfunc, peqfunc, ghost_update_fields,
                            *mpi_tag, true, true, true, true, false, std::false_type{} );
        update_count = 0;
        for(size_t cell_i=0;cell_i<n_cells;cell_i++) if( ! grid->is_ghost_cell(cell_i) )
        {
#         ifndef NDEBUG
          const auto * __restrict__ ids = cells[cell_i][field::id];
#         endif
          const auto *  __restrict__ cmol = cells[cell_i][cmol_field];
          auto * __restrict__ mol_ids   = cells[cell_i][idmol_field];
          size_t n = cells[cell_i].size();
          for(size_t p_i=0;p_i<n;p_i++)
          {
            assert( mol_ids[p_i] <= ids[p_i] );
            uint64_t molidmin = mol_ids[p_i];
            for(size_t j=0;j<cmol[p_i].size();j++)
            {
              const auto conloc = atom_from_idmap_if_found( cmol[p_i][j] , *id_map , *id_map_ghosts , null_loc );
              if( conloc != null_loc )
              {
                size_t cell=null_index, pos=null_index;
                decode_cell_particle( conloc , cell, pos );
                assert( cell!=null_index && pos!=null_index );
                const auto con_mol_id = cells[cell][idmol_field][pos];
                molidmin = std::min( molidmin , con_mol_id );
                if( molidmin < con_mol_id )
                {
                  cells[cell][idmol_field][pos] = molidmin;
                  ++ update_count;
                }
              }
            }
            if( molidmin < mol_ids[p_i] )
            {
              mol_ids[p_i] = molidmin;
              ++ update_count;
              // ldbg << "cell #"<<cell_i<<" particle #"<<p_i<<" molid "<<mol_ids[p_i]<<" -> "<<molidmin<<std::endl;
            }
          }
        }
        MPI_Allreduce( MPI_IN_PLACE, &update_count , 1 , MPI_LONG , MPI_SUM , *mpi );
        ++ pass;
        ldbg << "molecule_unfold: connect molecules, pass="<<pass<<" , updates="<<update_count<<std::endl;
      } while( update_count > 0 );

      // this temporary position field holds particle's unfolded position with respect
      // to molecule owner particle.
      const auto ufpos_field = grid->field_accessor( field::molufpos );

      // a first pre-process pass initialize molufpos to unmodified particle's postiotion
      for(size_t cell_i=0;cell_i<n_cells;cell_i++) if( ! grid->is_ghost_cell(cell_i) )
      {
        const auto * __restrict__ ids = cells[cell_i][field::id];
        const auto * __restrict__ rx = cells[cell_i][field::rx];
        const auto * __restrict__ ry = cells[cell_i][field::ry];
        const auto * __restrict__ rz = cells[cell_i][field::rz];
        auto * __restrict__ mol_ids = cells[cell_i][idmol_field];
        auto * __restrict__ ufpos = cells[cell_i][ufpos_field];
        const size_t n = cells[cell_i].size();
        for(size_t p_i=0;p_i<n;p_i++)
        {
          ufpos[p_i] = Vec3d{ rx[p_i] , ry[p_i] , rz[p_i] };
          const auto mid = mol_ids[p_i];
          if( ids[p_i] == mid )
          {
            const auto ploc = encode_cell_particle(cell_i,p_i,0);
            if( molecule_owner.find(mid) != molecule_owner.end() )
            {
              fatal_error() << "duplicate owner particle for molecule id #"<<mol_ids[p_i]<<std::endl;
            }
            molecule_owner.insert( { mid , ploc } );
            // ldbg << "cell #"<<cell_i<<" particle #"<<p_i<<" owner of molid #"<<mid<<std::endl;
          }
          else
          {
            mol_ids[p_i] = null_id;
          }
        }
      }

      const Vec3d size_box {std::abs(domain->extent().x - domain->origin().x),
                      std::abs(domain->extent().y - domain->origin().y),
                      std::abs(domain->extent().z - domain->origin().z)};
      const double half_min_size_box = std::min( std::min(size_box.x,size_box.y) , size_box.z) / 2.0; 
      ldbg << "Number of molecules = "<< molecule_owner.size()<<" , size_box = "<<size_box<<" , half_min_size_box = "<<half_min_size_box<<std::endl;

      // debug map to keep track of duplicated positions
      std::unordered_map<uint64_t,Vec3d> particle_pos_map;

      // second multipass algorithm : unfold positions by applying boundary conditions shift
      // whenever a bound particle has an excessive distance, taking the one with the
      pass = 0;
      do
      {                  
        int sub_pass = 0;        
        do
        {
          update_count = 0;
          for(size_t cell_i=0;cell_i<n_cells;cell_i++) if( ! grid->is_ghost_cell(cell_i) )
          {
            const auto *  __restrict__ cmol = cells[cell_i][cmol_field];
            auto * __restrict__ mol_ids = cells[cell_i][idmol_field];
            auto * __restrict__ ufpos = cells[cell_i][ufpos_field];
            size_t n = cells[cell_i].size();
            for(size_t p_i=0;p_i<n;p_i++)
            {
              if( mol_ids[p_i] != null_id ) // unfolded central particle
              {
                const Vec3d ri = ufpos[p_i];
                for(size_t j=0;j<cmol[p_i].size();j++)
                {
                  const auto conloc = atom_from_idmap_if_found( cmol[p_i][j] , *id_map , *id_map_ghosts , null_loc );
                  if( conloc != null_loc )
                  {
                    size_t cell=null_index, pos=null_index;
                    decode_cell_particle( conloc , cell, pos );
                    assert( cell!=null_index && pos!=null_index );
                    assert( cell!=cell_i || pos!=p_i );
                    const Vec3d rj = cells[cell][ufpos_field][pos] ;
                    if( cells[cell][idmol_field][pos] == null_id ) // not unfolded bond neighbor
                    {
                      const Vec3d rij = periodic_r_delta_loop( ri , rj , size_box , half_min_size_box );
                      const double norm_rij = norm(rij);
                      if( norm_rij > half_min_size_box || norm_rij == 0.0 )
                      {
                        fatal_error() << "pre-check: in C#"<<cell_i<<"P#"<<p_i<<" (id="<<cells[cell_i][field::id][p_i]<<") bond#"<<j
                        <<" with C#"<<cell<<"P#"<<pos<<" (id="<<cells[cell][field::id][pos]<<",ghost="<<grid->is_ghost_cell(cell)<<") bad dist. "<<norm_rij<<" not in ] 0 ; "<<half_min_size_box<<" ]"
                        <<" , ri="<<ri<<" , rj="<<rj<< std::endl;
                      }
                      bond_max_dist = std::max( bond_max_dist , norm_rij );   
                      const Vec3d rj_unfolded = ri + rij;
                      auto it = particle_pos_map.find( cells[cell][field::id][pos] );
                      if( it != particle_pos_map.end() )
                      {
                        if( rj_unfolded != it->second )
                        {
                          fatal_error() << "different ghost pos : rj_unfolded="<<rj_unfolded<<" , stored="<<it->second<<" dist="<<norm(rj_unfolded-it->second)<<std::endl;
                        }
                      }
                      else
                      {
                        particle_pos_map.insert( { cells[cell][field::id][pos] , rj_unfolded } );
                      }
                      cells[cell][ufpos_field][pos] = rj_unfolded;
                      cells[cell][idmol_field][pos] = mol_ids[p_i];
                      ++ update_count;
                    } // if bond neighbor nor unfolded yet
                  } // bond neighbor is present
                } // for each bond neighbor
              } // if central is unfolded
            } // for each cell's particle
          } // for each non ghost cell

          ++ sub_pass;
          ldbg << "molecule_unfold: unfold central positions, sub_pass="<<sub_pass<<" , updates="<<update_count <<std::endl;
        } while( update_count > 0 );

        // propagate ghost updates
        const auto ghost_update_fields = onika::make_flat_tuple( idmol_field, ufpos_field );
        grid_update_ghosts( exanb::ldbg, *mpi, *ghost_comm_scheme, *grid, *domain, nullptr,
                            *ghost_comm_buffers, pecfunc,peqfunc, ghost_update_fields,
                            *mpi_tag, true, true, true, true, false, std::false_type{} );        

        // gather updates from ghosts (from neighbor sub-domains)
        update_count = 0;
        for(size_t cell_i=0;cell_i<n_cells;cell_i++) if( ! grid->is_ghost_cell(cell_i) )
        {
          const auto *  __restrict__ cmol = cells[cell_i][cmol_field];
          auto * __restrict__ mol_ids = cells[cell_i][idmol_field];
          auto * __restrict__ ufpos = cells[cell_i][ufpos_field];
          size_t n = cells[cell_i].size();
          for(size_t p_i=0;p_i<n;p_i++)
          {
            if( mol_ids[p_i] == null_id ) // central particle not unfolded yet
            {
              const Vec3d ri = ufpos[p_i];
              int ri_updates = 0;
              Vec3d last_ri_unfolded = { 0, 0, 0 };
              for(size_t j=0;j<cmol[p_i].size();j++)
              {
                const auto conloc = atom_from_idmap_if_found( cmol[p_i][j] , *id_map , *id_map_ghosts , null_loc );
                if( conloc != null_loc )
                {
                  size_t cell=null_index, pos=null_index;
                  decode_cell_particle( conloc , cell, pos );
                  assert( cell!=null_index && pos!=null_index );
                  assert( cell!=cell_i || pos!=p_i );
                  const Vec3d rj = cells[cell][ufpos_field][pos] ;
                  if( cells[cell][idmol_field][pos] != null_id ) // bond neighbor already unfolded
                  {
                    const Vec3d rij = periodic_r_delta_loop( ri , rj , size_box , half_min_size_box );
                    const Vec3d ri_unfolded = ri - rij;
                    if( ri_updates > 0 )
                    {
                      if( ri_unfolded != last_ri_unfolded )
                      {
                        fatal_error() << "different update of central particle : last_ri_unfolded="<<last_ri_unfolded<<" ri_unfolded="<<ri_unfolded<<" dist="<<norm(ri_unfolded-last_ri_unfolded)<<" ri_updates="<<ri_updates<<" j="<<j<<std::endl;
                      }
                    }
                    else
                    {
                      last_ri_unfolded = ri_unfolded;
                    }
                    ufpos[p_i] = ri_unfolded;
                    mol_ids[p_i] = cells[cell][idmol_field][pos];
                    ++ ri_updates;
                  } // if bond neighbor nor unfolded yet
                } // bond neighbor is present
              } // for each bond neighbor
              update_count += ri_updates;
            } // if central is unfolded
          } // for each cell's particle
        } // for each non ghost cell

        MPI_Allreduce( MPI_IN_PLACE, &update_count , 1 , MPI_LONG , MPI_SUM , *mpi );
        ++ pass;
        ldbg << "molecule_unfold: unfold from bond neighbor positions, pass="<<pass<<" , updates="<<update_count <<std::endl;
        
      } while( update_count > 0 );

      for(size_t cell_i=0;cell_i<n_cells;cell_i++) if( ! grid->is_ghost_cell(cell_i) )
      {
        const auto * __restrict__ ufpos = cells[cell_i][ufpos_field];
        const auto * __restrict__ mol_ids = cells[cell_i][idmol_field];
        auto * __restrict__ rx   = cells[cell_i][field::rx];
        auto * __restrict__ ry   = cells[cell_i][field::ry];
        auto * __restrict__ rz   = cells[cell_i][field::rz];
        const size_t n = cells[cell_i].size();
        for(size_t p_i=0;p_i<n;p_i++)
        {
          if( mol_ids[p_i] != null_id )
          {
            rx[p_i] = ufpos[p_i].x;
            ry[p_i] = ufpos[p_i].y;
            rz[p_i] = ufpos[p_i].z;
          }
          else
          {
            fatal_error() << "Cell #"<<cell_i<<" , particle #"<<p_i<<" not unfolded"<<std::endl;
          }
        }
      }

      grid->remove_flat_array( "molufpos" );
      
      MPI_Allreduce( MPI_IN_PLACE, &bond_max_dist , 1 , MPI_DOUBLE , MPI_MAX , *mpi );
      ldbg << "bond_max_dist = " << bond_max_dist << std::endl;
    }

  };

  template<class GridT> using MoleculeUnfoldTmpl = MoleculeUnfold<GridT>;

  // === register factories ===
  ONIKA_AUTORUN_INIT(molecule_unfold)
  {
   OperatorNodeFactory::instance()->register_factory( "molecule_unfold", make_grid_variant_operator<MoleculeUnfoldTmpl> );
  }

}
