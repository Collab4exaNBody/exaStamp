#include "hip/hip_runtime.h"


#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <exanb/core/grid.h>
#include <onika/memory/allocator.h>
#include <exaStamp/parrinellorahman/parrinellorahman.h>
#include <exaStamp/parrinellorahman/parrinellorahman_yaml.h>
#include <exaStamp/parrinellorahman/parrinellorahman_stream.h>
#include <exanb/core/domain.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <exanb/core/unityConverterHelper.h>
#include <exanb/core/quantity.h>
#include <exanb/core/physics_constants.h>
#include <exanb/core/unityConverterHelper.h>

#include <onika/soatl/field_pointer_tuple.h>
#include <memory>
#include <iostream>
#include <iomanip>
#include <mpi.h>
#include <onika/math/basic_types_stream.h>


namespace exaStamp
{

  // get particle virial tensor. assume the virial is null if particle hasn't virial field
  template<bool has_virial>
  static inline Mat3d get_particle_virial(const Mat3d* virials, size_t p_i, std::integral_constant<bool,has_virial> )
  {
    if constexpr (has_virial) { return virials[p_i]; }
    return Mat3d();
  }
  
  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_vx, field::_vy, field::_vz, field::_ax, field::_ay, field::_az, field::_id >
    >
  struct FirstPushParrinelloRahman : public OperatorNode
  {
    using PointerTuple = onika::soatl::FieldPointerTuple<
                          GridT::CellParticles::Alignment , GridT::CellParticles::ChunkSize , 
                          field::_rx, field::_ry, field::_rz,
                          field::_vx, field::_vy, field::_vz,
                          field::_ax, field::_ay, field::_az >;

    // compile time constant indicating if grid has type virial
    using has_virial_field_t = typename GridT::CellParticles::template HasField < field::_virial > ;
    static constexpr std::integral_constant<bool,has_virial_field_t::value> has_virial_field{};
    
    ADD_SLOT( MPI_Comm                , mpi        , INPUT , REQUIRED );    
    ADD_SLOT( GridT                   , grid       , INPUT_OUTPUT );
    ADD_SLOT( double                  , dt         , INPUT , REQUIRED );
    ADD_SLOT( double                  , dt_scale   , INPUT , REQUIRED );
    ADD_SLOT( ParrinelloRahmanContext , parrinello_rahman_ctx , INPUT_OUTPUT );
    ADD_SLOT( Domain                  , domain     , INPUT );    
    ADD_SLOT( ParticleSpecies         , species    , INPUT , REQUIRED );
    
    inline void execute () override final
    {
      static const double boltzmann_internal = UnityConverterHelper::convert(legacy_constant::boltzmann, "m^2*kg/s^2/K");
      static const double conv_gnvtv = UnityConverterHelper::convert(1.0,"m^2/s^2");
      static const double conv_time = UnityConverterHelper::convert(1.0,"1/s^2");
      ParrinelloRahmanContext& data = *parrinello_rahman_ctx;

      //ldbg << "ParrinelloRahman::firstPush begin" << std::endl;

      // compute extension
      Mat3d mat = domain->xform();
      //Vec3d mat_ext = { norm(column1(mat)) , norm(column2(mat)) , norm(column3(mat)) };
      //Vec3d mat_exti = reciprocal( mat_ext );
      Vec3d dom_ext = domain->bounds_size();
      Vec3d dom_exti = reciprocal( dom_ext );
      
      //Vec3d ext = mat_ext * dom_ext;
      //const Vec3d exti = reciprocal( ext );
      const Mat3d mcr = multiply( mat , diag_matrix(dom_ext) );
      const Mat3d mcri = multiply( diag_matrix(dom_exti), inverse(mat) );

      data.h = mcr; // multiply( mat ,  diag_matrix( ext ) ); // == mcr
      data.updateMembers();
      
      data.m_gammaNVTp = 0.;
      data.hp = make_zero_matrix(); 
      data.hpp = make_zero_matrix();
      
      const double raw_dt = *dt;
      const double scale = *dt_scale;

      auto cells = grid->cells();
      IJK dims = grid->dimension();
      ssize_t gl = grid->ghost_layers();
      // size_t nb_particles = grid->number_of_particles() - grid->number_of_ghost_particles() ;

      const double dt = raw_dt * scale;
      const double dt2 = dt*dt;

      //data.print( ldbg );

      size_t total_particles = 0;          
      Mat3d hpp_cell = {0.};
      double gammaNVTp_cell = 0.;

/*
      ldbg << "conv_gnvtv = " << conv_gnvtv << std::endl;
      ldbg << "conv_time = " << conv_time << std::endl;
      ldbg << "dt   = " << dt << std::endl;
      ldbg << "dt2  = " << dt2 << std::endl;
      ldbg << "mat      = " << mat << std::endl;
      ldbg << "mat_ext  = " << mat_ext << std::endl;
      ldbg << "mat_ext*dom_ext  = " << dom_ext*mat_ext << std::endl;
      ldbg << "dom_ext  = " << dom_ext << std::endl;
      ldbg << "ext  = " << ext << std::endl;
      ldbg << "exti = " << exti << std::endl;
      ldbg << "mcr  = " << mcr << std::endl ;
      ldbg << "mcri = " << mcri << std::endl ;
*/

#     pragma omp parallel
      {
        PointerTuple ptrs;   
        GRID_OMP_FOR_BEGIN(dims-2*gl,_,loc, schedule(dynamic) reduction(+:hpp_cell,gammaNVTp_cell,total_particles) )
        {
          size_t i = grid_ijk_to_index( dims , loc + gl );
          int n = cells[i].size();
          cells[i].capture_pointers( ptrs );
          
          auto* __restrict__ rx = ptrs[ field::rx ]; ONIKA_ASSUME_ALIGNED(rx);
          auto* __restrict__ ry = ptrs[ field::ry ]; ONIKA_ASSUME_ALIGNED(ry);
          auto* __restrict__ rz = ptrs[ field::rz ]; ONIKA_ASSUME_ALIGNED(rz);

          auto* __restrict__ vx = ptrs[ field::vx ]; ONIKA_ASSUME_ALIGNED(vx);
          auto* __restrict__ vy = ptrs[ field::vy ]; ONIKA_ASSUME_ALIGNED(vy);
          auto* __restrict__ vz = ptrs[ field::vz ]; ONIKA_ASSUME_ALIGNED(vz);
          
          const auto* __restrict__ ax = ptrs[ field::ax ]; ONIKA_ASSUME_ALIGNED(ax);
          const auto* __restrict__ ay = ptrs[ field::ay ]; ONIKA_ASSUME_ALIGNED(ay);
          const auto* __restrict__ az = ptrs[ field::az ]; ONIKA_ASSUME_ALIGNED(az);
          
          const uint8_t* __restrict__ types = cells[i].field_pointer_or_null(field::type); ONIKA_ASSUME_ALIGNED(types);
          const Mat3d* __restrict__ vir = cells[i].field_pointer_or_null(field::virial); ONIKA_ASSUME_ALIGNED(vir);

          const auto* __restrict__ ids = cells[i].field_pointer_or_null(field::id); ONIKA_ASSUME_ALIGNED(ids);

          Mat3d virk;
          double mk;          
          Mat3d hpp_k = {0.};
          double gammaNVTp_k = 0.;

#         pragma omp simd reduction(+:hpp_k,gammaNVTp_k)
          for(int k=0;k<n;k++)
          {
            mk = species->at(types[k]).m_mass; // FIXME: warning, cases with no type field will crash, replace with get_type method similar to get_virial
            virk = get_particle_virial( vir, k, has_virial_field);

            Vec3d r = dom_exti * Vec3d{ rx[k], ry[k], rz[k] };
            Vec3d v = mcri * Vec3d{ vx[k], vy[k], vz[k] };
            Vec3d f = Vec3d{ ax[k], ay[k], az[k] };

            const auto cnv_1 = ( data.Giht * f ) / conv_time;
            const auto cnv_2 = data.GiGp * v;
            const auto cnv_3 = ( data.m_gammaNVT * v ) / conv_gnvtv;
            const auto a = ( cnv_1 - cnv_2 - cnv_3 ) * conv_time;
            
            // const auto tmp1 = ext * (r + dt * v + 0.5 * dt2 * a);
            const auto tmp1_v2 = dom_ext * (r + dt * v + 0.5 * dt2 * a);
            const auto tmp2 = mcr * (v + 0.5 * dt * a);

            rx[k] = tmp1_v2.x;
            ry[k] = tmp1_v2.y;
            rz[k] = tmp1_v2.z;

            vx[k] = tmp2.x;
            vy[k] = tmp2.y;
            vz[k] = tmp2.z;

            hpp_k += (data.h * tensor(v, v)) * mk + transpose(data.hi * virk);
            gammaNVTp_k += dot(v, data.G * v) * mk;
          }

          hpp_cell += hpp_k;
          gammaNVTp_cell += gammaNVTp_k;
          total_particles += n;
        }
        GRID_OMP_FOR_END
      }

      // sum hpp and gammaNVTp over all processors
      
      //ldbg << "syncDataReduce" << std::endl;
      {
        double tmp[11] = { hpp_cell.m11, hpp_cell.m12, hpp_cell.m13, hpp_cell.m21, hpp_cell.m22, hpp_cell.m23, hpp_cell.m31, hpp_cell.m32, hpp_cell.m33,
                           gammaNVTp_cell, static_cast<double>(total_particles) };
        MPI_Allreduce(MPI_IN_PLACE,tmp,11,MPI_DOUBLE,MPI_SUM,*mpi);
        hpp_cell.m11 = tmp[0];
        hpp_cell.m12 = tmp[1];
        hpp_cell.m13 = tmp[2];
        hpp_cell.m21 = tmp[3];
        hpp_cell.m22 = tmp[4];
        hpp_cell.m23 = tmp[5];
        hpp_cell.m31 = tmp[6];
        hpp_cell.m32 = tmp[7];
        hpp_cell.m33 = tmp[8];
        gammaNVTp_cell = tmp[9];
        total_particles = static_cast<size_t>( tmp[10] );
      }

      //ldbg << "total particles = "<< total_particles<<std::endl;
      data.m_gammaNVTp = gammaNVTp_cell;
      data.hpp = hpp_cell;
      //data.print( ldbg );

      // WARNING: try to move it afterward for test
      // data.apply_mask();

      //std::cout << "after hp/hpp update" << std::endl;
      data.hpp = (data.hpp - data.hp * (data.m_gammaNVT * data.m_config.m_masseB) - comatrix(data.h) * data.m_config.m_Pext ) / data.m_config.m_masseB;  
      data.apply_mask(); // WARNING moved here for test // seems to work for NPT_iso_xy test case
      
      data.h = data.h + data.hp * dt + data.hpp * 0.5 * dt2;
      data.hp = data.hp + data.hpp * (0.5 * dt);
      data.m_gammaNVT += 0.5 * dt * data.m_gammaNVTp;

      //data.print( ldbg );

      const double oldtrace = trace_matrix(data.hpthp);
      //ldbg << "oldtrace = "<<oldtrace << std::endl;
      
      //ldbg << "updateMembers" << std::endl;
      // update scheme parameters
      data.updateMembers();
      //data.print( ldbg );

      // update gamma
				   
//      ldbg << "boltzmann =" << boltzmann_internal << std::endl;
//      ldbg << "Text =" << data.m_config.m_Text << std::endl;

      double tmp = data.m_config.m_masseB * oldtrace - 3. * total_particles * boltzmann_internal * data.m_config.m_Text;
//      ldbg << "tmp = " << tmp << std::endl;
      data.m_gammaNVTp = (data.m_gammaNVTp + tmp) / data.m_config.m_masseNVT;
      data.m_gammaNVT = data.m_gammaNVT + 0.5 * dt * data.m_gammaNVTp;

      // ldbg << "after gamma update" << std::endl;
      //data.print( ldbg );
      //ldbg << "--- end first push ---" << std::endl;
    }

  };

  template<class GridT> using FirstPushParrinelloRahmanTmpl = FirstPushParrinelloRahman<GridT>;
  
 // === register factories ===  
  ONIKA_AUTORUN_INIT(first_push_parrinellorahman)
  {
   OperatorNodeFactory::instance()->register_factory( "first_push_parrinellorahman", make_grid_variant_operator< FirstPushParrinelloRahmanTmpl > );
  }

}

