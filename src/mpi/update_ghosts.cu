#include "hip/hip_runtime.h"
//  // DO NOT REMOVE THIS LINE !!

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/math/basic_types_stream.h>
#include <exanb/core/grid.h>
#include <exanb/grid_cell_particles/grid_cell_values.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/particle_id_codec.h>
#include <exanb/core/grid_fields.h>
#include <exanb/core/check_particles_inside_cell.h>

#include <onika/soatl/field_tuple.h>

#include <vector>
#include <string>
#include <list>
#include <algorithm>
#include <tuple>

#include <mpi.h>
#include <exanb/mpi/update_ghost_utils.h>
#include <exanb/mpi/ghosts_comm_scheme.h>
#include <exanb/mpi/update_ghosts.h>
#include <onika/mpi/data_types.h>

namespace exaStamp
{
  using namespace exanb;
  using namespace UpdateGhostsUtils;

  // === register factory ===
  template<typename GridT> using UpdateGhostsAllFieldsNoFV = UpdateGhostsNode< GridT , RemoveFields< typename GridT::Fields , FieldSet<field::_fx,field::_fy,field::_fz,field::_ep, field::_vx, field::_vy, field::_vz > > , true >;
  template<typename GridT> using UpdateGhostsRandVandVir = UpdateGhostsNode< GridT , FieldSet<field::_rx, field::_ry, field::_rz,  field::_vx, field::_vy, field::_vz, field::_virial > , false >;
  template<typename GridT> using UpdateGhostsRandV = UpdateGhostsNode< GridT , FieldSet<field::_rx, field::_ry, field::_rz,  field::_vx, field::_vy, field::_vz > , false >;

  template<typename GridT> using UpdateGhostsRandRf = UpdateGhostsNode< GridT , FieldSet<field::_rx, field::_ry, field::_rz, field::_rxf, field::_ryf, field::_rzf > , false >;  
  template<typename GridT> using UpdateGhostsRandRfandV = UpdateGhostsNode< GridT , FieldSet<field::_rx, field::_ry, field::_rz, field::_rxf, field::_ryf, field::_rzf, field::_vx, field::_vy, field::_vz > , false >;
  
  template<typename GridT> using UpdateGhostsRQ = UpdateGhostsNode< GridT , FieldSet<field::_rx, field::_ry, field::_rz , field::_orient > , false >;
  template<typename GridT> using UpdateGhostsIdMol = UpdateGhostsNode< GridT , FieldSet<field::_idmol> , false >;

  ONIKA_AUTORUN_INIT(update_ghosts)
  {
    OperatorNodeFactory::instance()->register_factory( "ghost_update_r_v_vir",   make_grid_variant_operator<UpdateGhostsRandVandVir> );
    OperatorNodeFactory::instance()->register_factory( "ghost_update_r_v",       make_grid_variant_operator<UpdateGhostsRandV> );
    OperatorNodeFactory::instance()->register_factory( "ghost_update_r_rf",      make_grid_variant_operator<UpdateGhostsRandRf> );    
    OperatorNodeFactory::instance()->register_factory( "ghost_update_r_rf_v",    make_grid_variant_operator<UpdateGhostsRandRfandV> );    
    OperatorNodeFactory::instance()->register_factory( "ghost_update_rq",        make_grid_variant_operator<UpdateGhostsRQ> );
    OperatorNodeFactory::instance()->register_factory( "ghost_update_idmol",     make_grid_variant_operator<UpdateGhostsIdMol> );
  }

}

