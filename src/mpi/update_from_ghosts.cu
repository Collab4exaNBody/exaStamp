#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/

//  // DO NOT REMOVE THIS LINE !!

#include <exanb/core/grid_fields.h>
#include <exanb/core/grid_fields.h>

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/math/basic_types_stream.h>
#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/particle_id_codec.h>
#include <exanb/core/check_particles_inside_cell.h>

#include <onika/mpi/data_types.h>
#include <exanb/mpi/ghosts_comm_scheme.h>

#include <mpi.h>
#include <vector>
#include <string>
#include <list>
#include <algorithm>

#include <exanb/mpi/update_from_ghost_utils.h>
#include <exanb/mpi/ghosts_comm_scheme.h>
#include <exanb/mpi/update_from_ghosts.h>
#include <exanb/grid_cell_particles/cell_particle_update_functor.h>

namespace exaStamp
{
  using namespace exanb;
    
  // === register factory ===
  template<typename GridT> using UpdateForceEnergyFromGhosts = UpdateFromGhosts< GridT , FieldSet<field::_fx,field::_fy,field::_fz, field::_ep>, UpdateValueAdd >;
  template<typename GridT> using UpdateFlatForceEnergyFromGhosts = UpdateFromGhosts< GridT , FieldSet<field::_flat_fx,field::_flat_fy,field::_flat_fz, field::_flat_ep>, UpdateValueAdd >;
  template<typename GridT> using UpdateVirialForceEnergyFromGhosts = UpdateFromGhosts< GridT , FieldSet<field::_fx,field::_fy,field::_fz, field::_ep, field::_virial>, UpdateValueAdd >;
  template<typename GridT> using UpdateFromGhostsTestId = UpdateFromGhosts< GridT , FieldSet<field::_id>, UpdateValueAssertEqual >;

  ONIKA_AUTORUN_INIT(update_from_ghosts)
  {
    OperatorNodeFactory::instance()->register_factory( "update_force_energy_from_ghost", make_grid_variant_operator<UpdateForceEnergyFromGhosts> );
    OperatorNodeFactory::instance()->register_factory( "flat_force_energy_from_ghost", make_grid_variant_operator<UpdateFlatForceEnergyFromGhosts> );
    OperatorNodeFactory::instance()->register_factory( "update_virial_force_energy_from_ghost", make_grid_variant_operator<UpdateVirialForceEnergyFromGhosts> );
    OperatorNodeFactory::instance()->register_factory( "update_from_ghost_check_id", make_grid_variant_operator<UpdateFromGhostsTestId> );
  }

}

