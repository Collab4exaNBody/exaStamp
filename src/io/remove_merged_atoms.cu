#include "hip/hip_runtime.h"
//_enable_cuda // DO NOT REMOVE THIS LINE

#include <exanb/core/grid.h>
#include <onika/math/basic_types.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <onika/log.h>
#include <exanb/core/domain.h>

#include <exanb/particle_neighbors/chunk_neighbors.h>
#include <onika/cuda/cuda_math.h>

// this allows for parallel compilation of templated operator for each available field set

namespace exanb
{

  struct RemoveZeroDistParticlesFunctor
  {
    const double epsilon2 = 0.0;
    // ComputeBuffer less computation without virial
    template<class CellParticlesT, class NbhDataT>
    ONIKA_HOST_DEVICE_FUNC inline void operator () (const Vec3d& dr, double d2, uint64_t& id, CellParticlesT, size_t, size_t, const NbhDataT&) const
    {
      if( d2 <= epsilon2 ) id = onika::cuda::numeric_limits<uint64_t>::max;
    }
  };

  template<> struct ComputePairTraits<RemoveZeroDistParticlesFunctor>
  {
    static inline constexpr bool RequiresBlockSynchronousCall = false;
    static inline constexpr bool ComputeBufferCompatible = false;
    static inline constexpr bool BufferLessCompatible = true;
    static inline constexpr bool CudaCompatible = true;
  };

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_id >
    >
  class RemoveMergedAtoms : public OperatorNode
  {      
    ADD_SLOT( GridChunkNeighbors , chunk_neighbors , INPUT, REQUIRED, DocString{"neighbor list"} );
    ADD_SLOT( Domain             , domain          , INPUT );
    ADD_SLOT( double             , distance        , 1.e-3 );
    ADD_SLOT( GridT              , grid            , INPUT_OUTPUT );

  public:
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );      
      size_t n_cells = grid->number_of_cells();
      if( n_cells==0 )
      {
        return ;
      }
      
      const double mdist = *distance;
      
      ComputePairOptionalLocks<false> cp_locks {};
      // true means symmetric: we don't want to delete the pair of merged atoms, just one of the two
      exanb::GridChunkNeighborsLightWeightIt<true> nbh_it{ *chunk_neighbors };
      ComputePairBufferFactory< ComputePairBuffer2<> > force_buf = {};
      RemoveZeroDistParticlesFunctor force_op = { mdist*mdist };
      LinearXForm cp_xform { domain->xform() };
      auto optional = make_compute_pair_optional_args( nbh_it, ComputePairNullWeightIterator{} , cp_xform, cp_locks );
      compute_cell_particle_pairs(*grid, mdist, false, optional, force_buf, force_op, FieldSet<field::_id>{}, parallel_execution_context() );

      auto cells = grid->cells();
      size_t number_of_removed_particles = 0;
      
#     pragma omp parallel for schedule(dynamic) reduction(+:number_of_removed_particles)
      for(size_t cell_i=0;cell_i<n_cells;cell_i++) if( ! grid->is_ghost_cell(cell_i) )
      {
        unsigned int n_particles = cells[cell_i].size();
        for(unsigned int p=0;p<n_particles;)
        {
          if( cells[cell_i][field::id][p] == onika::cuda::numeric_limits<uint64_t>::max )
          {
            -- n_particles;
            cells[cell_i].write_tuple( p , cells[cell_i][n_particles] );
            ++ number_of_removed_particles;
          }
          else
          {
            ++ p;
          }
        }
        cells[cell_i].resize( n_particles , grid->cell_allocator() );
      }
      
      grid->rebuild_particle_offsets();
      
      ldbg << "number_of_removed_particles = " << number_of_removed_particles << std::endl;
    }

  };

  template<class GridT> using RemoveMergedAtomsTmpl = RemoveMergedAtoms<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(remove_merged_atoms)
  {  
    OperatorNodeFactory::instance()->register_factory( "remove_merged_atoms" , make_grid_variant_operator<RemoveMergedAtomsTmpl> );
  }

}


