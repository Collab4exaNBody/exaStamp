#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements. See the NOTICE file
distributed with this work for additional information
regarding copyright ownership. The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License. You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied. See the License for the
specific language governing permissions and limitations
under the License.
*/





#include <string>
#include <numeric>

#include <onika/math/basic_types_yaml.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/grid.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <onika/log.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/parallel/random.h>
#include <onika/physics/units.h>
#include <onika/physics/constants.h>
#include <onika/physics/units.h>

#include <onika/cuda/hip/hip_runtime.h>
#include <exanb/compute/compute_cell_particles.h>

//#include "quaternion_rotation.h"
#include <onika/math/quaternion_operators.h>

namespace exaStamp
{
  using namespace exanb;

  struct PushTorqueAngmomComputeFunc
  {
    const double dt;
    ONIKA_HOST_DEVICE_FUNC inline void operator () ( const Vec3d& couple, Vec3d& angmom ) const
    {
      angmom += couple * dt / 2.0;
    }
  };
}

namespace exanb
{
  template<> struct ComputeCellParticlesTraits< exaStamp::PushTorqueAngmomComputeFunc >
  {
    static inline constexpr bool RequiresBlockSynchronousCall = false;
    static inline constexpr bool CudaCompatible = true;
  };
}

namespace exaStamp
{
  using namespace exanb;

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_angmom, field::_couple >
    >
  class PushTorqueAngmomRigidMol : public OperatorNode
  {
    //ADD_SLOT( MPI_Comm        , mpi          , INPUT , MPI_COMM_WORLD  );
    ADD_SLOT( GridT           , grid         , INPUT_OUTPUT );
    ADD_SLOT( double          , dt           , INPUT, REQUIRED );

    static constexpr FieldSet< field::_couple , field::_angmom > compute_field_set{};

  public:
    inline void execute () override final
    {
      compute_cell_particles( *grid , false , PushTorqueAngmomComputeFunc{*dt} , compute_field_set , parallel_execution_context() );
    }

  };

  // === register factories ===
  template<class GridT> using PushTorqueAngmomRigidMolTmpl = PushTorqueAngmomRigidMol<GridT>;

  ONIKA_AUTORUN_INIT(push_torque_angmom)
  {
    OperatorNodeFactory::instance()->register_factory("push_torque_angmom", make_grid_variant_operator< PushTorqueAngmomRigidMolTmpl >);
  }

}
