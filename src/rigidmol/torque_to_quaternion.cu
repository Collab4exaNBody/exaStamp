#include "hip/hip_runtime.h"




#include <chrono>
#include <ctime>
#include <mpi.h>
#include <string>
#include <numeric>

#include <onika/math/basic_types_yaml.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/grid.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <onika/log.h>
#include <exaStamp/particle_species/particle_specie.h>
#include <onika/parallel/random.h>
#include <onika/physics/units.h>
#include <onika/physics/constants.h>
#include <onika/physics/units.h>

#include <onika/cuda/hip/hip_runtime.h>
#include <exanb/compute/compute_cell_particles.h>

//#include "quaternion_rotation.h"
#include <exanb/core/quaternion_operators.h>
#include <exanb/core/quaternion_to_matrix.h>

namespace exaStamp
{
  using namespace exanb;

  struct TorqueToQuaternionComputeFunc
  {
    const double dt = 0.0;
    const ParticleSpecie * __restrict__ species = nullptr;
    
    ONIKA_HOST_DEVICE_FUNC inline void operator () ( const Vec3d& couple, const int t, Quaternion& orient, Vec3d& angmom ) const
    {
      Mat3d mat_lab_bf;
      mat_lab_bf.m11 = orient.w*orient.w + orient.x*orient.x - orient.y*orient.y - orient.z*orient.z;
      mat_lab_bf.m22 = orient.w*orient.w - orient.x*orient.x + orient.y*orient.y - orient.z*orient.z;
      mat_lab_bf.m33 = orient.w*orient.w - orient.x*orient.x - orient.y*orient.y + orient.z*orient.z;
      mat_lab_bf.m12 = 2.0 * (orient.x*orient.y + orient.w*orient.z ); 
      mat_lab_bf.m21 = 2.0 * (orient.x*orient.y - orient.w*orient.z );
      mat_lab_bf.m13 = 2.0 * (orient.x*orient.z - orient.w*orient.y );
      mat_lab_bf.m31 = 2.0 * (orient.x*orient.z + orient.w*orient.y );
      mat_lab_bf.m23 = 2.0 * (orient.y*orient.z + orient.w*orient.x );
      mat_lab_bf.m32 = 2.0 * (orient.y*orient.z - orient.w*orient.x );
      
      //recuperation moment inertiel
      Vec3d minert = species[t].m_minert;
      //calcul du moment angulaire dans le repere mobile
      Vec3d angmom_m = mat_lab_bf * angmom;
      //calcul du couple dans le repere mobile
      Vec3d couple_m = mat_lab_bf * couple;
      //calcul du moment angulaire dans repere fixe a t+dt/2
      angmom += couple*dt/2.0;
      //calcul du moment angulaire dans repere mobile a t+dt/2
      Vec3d omega_m={0.,0.,0.};
      if (minert.x>0.) {omega_m.x=angmom_m.x/minert.x;}
      if (minert.y>0.) {omega_m.y=angmom_m.y/minert.y;}
      if (minert.z>0.) {omega_m.z=angmom_m.z/minert.z;}
      Vec3d angmom_m_ddt = angmom_m + ( couple_m - cross( omega_m, angmom_m) )*dt/2.0;
      //calcul de la derivee du quaternion a t+dt/2
      Quaternion dquat_dt;
      Vec3d omega_m_ddt={0.,0.,0.};
      if (minert.x>0.) {omega_m_ddt.x=angmom_m_ddt.x/minert.x;}
      if (minert.y>0.) {omega_m_ddt.y=angmom_m_ddt.y/minert.y;}
      if (minert.z>0.) {omega_m_ddt.z=angmom_m_ddt.z/minert.z;}
      dquat_dt.w = 0.5 * (-orient.x*omega_m_ddt.x - orient.y*omega_m_ddt.y - orient.z*omega_m_ddt.z );
      dquat_dt.x = 0.5 * (orient.w*omega_m_ddt.x - orient.z*omega_m_ddt.y + orient.y*omega_m_ddt.z );
      dquat_dt.y = 0.5 * (orient.z*omega_m_ddt.x + orient.w*omega_m_ddt.y - orient.x*omega_m_ddt.z );
      dquat_dt.z = 0.5 * (-orient.y*omega_m_ddt.x + orient.x*omega_m_ddt.y + orient.w*omega_m_ddt.z );
      //calcul du quaternion a t+dt/2 
      Quaternion quat_ddt0 = normalize(orient + dquat_dt*dt/2.0);
      double conv = 1.0;
      static constexpr double conv_crit = 1.e-15;
      int niter = 0;
      static constexpr int niter_max = 30;
      while ( conv>conv_crit && niter<niter_max)
      {
        niter += 1;
        //calcul du moment angulaire dans le repere mobile
        mat_lab_bf.m11 = quat_ddt0.w*quat_ddt0.w + quat_ddt0.x*quat_ddt0.x - quat_ddt0.y*quat_ddt0.y - quat_ddt0.z*quat_ddt0.z;
        mat_lab_bf.m22 = quat_ddt0.w*quat_ddt0.w - quat_ddt0.x*quat_ddt0.x + quat_ddt0.y*quat_ddt0.y - quat_ddt0.z*quat_ddt0.z;
        mat_lab_bf.m33 = quat_ddt0.w*quat_ddt0.w - quat_ddt0.x*quat_ddt0.x - quat_ddt0.y*quat_ddt0.y + quat_ddt0.z*quat_ddt0.z;
        mat_lab_bf.m12 = 2.0 * (quat_ddt0.x*quat_ddt0.y + quat_ddt0.w*quat_ddt0.z ); 
        mat_lab_bf.m21 = 2.0 * (quat_ddt0.x*quat_ddt0.y - quat_ddt0.w*quat_ddt0.z );
        mat_lab_bf.m13 = 2.0 * (quat_ddt0.x*quat_ddt0.z - quat_ddt0.w*quat_ddt0.y );
        mat_lab_bf.m31 = 2.0 * (quat_ddt0.x*quat_ddt0.z + quat_ddt0.w*quat_ddt0.y );
        mat_lab_bf.m23 = 2.0 * (quat_ddt0.y*quat_ddt0.z + quat_ddt0.w*quat_ddt0.x );
        mat_lab_bf.m32 = 2.0 * (quat_ddt0.y*quat_ddt0.z - quat_ddt0.w*quat_ddt0.x );
        
        //calcul du moment angulaire dans le repere mobile a t+dt/2
        angmom_m_ddt = mat_lab_bf * angmom;
        //calcul de la vitesse angulaire dans le repere mobile a t+dt/2
        if (minert.x>0.) {omega_m_ddt.x=angmom_m_ddt.x/minert.x;}
        if (minert.y>0.) {omega_m_ddt.y=angmom_m_ddt.y/minert.y;}
        if (minert.z>0.) {omega_m_ddt.z=angmom_m_ddt.z/minert.z;}
        //calcul de la derivee du quaternion a t+dt/2
        dquat_dt.w = 0.5 * (-quat_ddt0.x*omega_m_ddt.x - quat_ddt0.y*omega_m_ddt.y - quat_ddt0.z*omega_m_ddt.z );
        dquat_dt.x = 0.5 * (quat_ddt0.w*omega_m_ddt.x - quat_ddt0.z*omega_m_ddt.y + quat_ddt0.y*omega_m_ddt.z );
        dquat_dt.y = 0.5 * (quat_ddt0.z*omega_m_ddt.x + quat_ddt0.w*omega_m_ddt.y - quat_ddt0.x*omega_m_ddt.z );
        dquat_dt.z = 0.5 * (-quat_ddt0.y*omega_m_ddt.x + quat_ddt0.x*omega_m_ddt.y + quat_ddt0.w*omega_m_ddt.z );
        //calcul du nouveau quaternion
        Quaternion quat_ddt1 = normalize(orient + dquat_dt*dt/2.0);
        //convergence
        conv = norm(quat_ddt1-quat_ddt0); //en realite on pourrait fusionner les deux lignes precedentes et n'avoir qu'un quaternion temporaire
        //stockage du nouveau quaternion obtenu
        quat_ddt0=quat_ddt1;
      }

      if( niter_max == niter && conv>conv_crit )
      {
        /* ABORT */
      }

      //calcul du quaternion a l'instant t+dt
      orient = normalize(orient + dquat_dt*dt);
    }
  };
}

namespace exanb
{
  template<> struct ComputeCellParticlesTraits< exaStamp::TorqueToQuaternionComputeFunc >
  {
    static inline constexpr bool RequiresBlockSynchronousCall = false;
    static inline constexpr bool CudaCompatible = true;
  };
}

namespace exaStamp
{
  using namespace exanb;

  inline std::ostream& operator << (  std::ostream& out , const exanb::Quaternion& q )
  {
    return out <<"("<< q.w<<","<<q.x<<","<<q.y<<","<<q.z<<")";
  }

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_angmom, field::_couple, field::_orient, field::_type >
    >
  class TorqueToQuaternionRigidMol : public OperatorNode
  {
    //ADD_SLOT( MPI_Comm        , mpi          , INPUT , MPI_COMM_WORLD  );
    ADD_SLOT( GridT           , grid         , INPUT_OUTPUT );
    ADD_SLOT( ParticleSpecies , species      , INPUT_OUTPUT );
    ADD_SLOT( double          , dt           , INPUT , REQUIRED );
    
    static constexpr FieldSet< field::_couple, field::_type, field::_orient, field::_angmom > compute_field_set{};
    
  public:
    inline void execute () override final
    {
      compute_cell_particles( *grid , false , TorqueToQuaternionComputeFunc{*dt,species->data()} , compute_field_set , parallel_execution_context() );

#if 0
      auto cells = grid->cells();
      IJK dims = grid->dimension();
      size_t ghost_layers = grid->ghost_layers();
      IJK dims_no_ghost = dims - (2*ghost_layers);
      const double dt           = *(this->dt);

      // partie 1
#     pragma omp parallel
      {
        GRID_OMP_FOR_BEGIN(dims_no_ghost,_,loc_no_ghosts, schedule(dynamic) )
        {
          IJK loc = loc_no_ghosts + ghost_layers;
          size_t cell_i = grid_ijk_to_index(dims,loc);

          auto* __restrict__ angmom = cells[cell_i][field::angmom];
          auto* __restrict__ orient = cells[cell_i][field::orient];
          const auto* __restrict__ couple = cells[cell_i][field::couple];
          const auto* __restrict__ type_atom = cells[cell_i][field::type];

          size_t n = cells[cell_i].size();

          for(size_t j=0;j<n;j++)
          {
            int t = type_atom[j];
            Mat3d mat_lab_bf;
            mat_lab_bf.m11 = orient[j].w*orient[j].w + orient[j].x*orient[j].x - orient[j].y*orient[j].y - orient[j].z*orient[j].z;
            mat_lab_bf.m22 = orient[j].w*orient[j].w - orient[j].x*orient[j].x + orient[j].y*orient[j].y - orient[j].z*orient[j].z;
            mat_lab_bf.m33 = orient[j].w*orient[j].w - orient[j].x*orient[j].x - orient[j].y*orient[j].y + orient[j].z*orient[j].z;
            mat_lab_bf.m12 = 2.0 * (orient[j].x*orient[j].y + orient[j].w*orient[j].z ); 
            mat_lab_bf.m21 = 2.0 * (orient[j].x*orient[j].y - orient[j].w*orient[j].z );
            mat_lab_bf.m13 = 2.0 * (orient[j].x*orient[j].z - orient[j].w*orient[j].y );
            mat_lab_bf.m31 = 2.0 * (orient[j].x*orient[j].z + orient[j].w*orient[j].y );
            mat_lab_bf.m23 = 2.0 * (orient[j].y*orient[j].z + orient[j].w*orient[j].x );
            mat_lab_bf.m32 = 2.0 * (orient[j].y*orient[j].z - orient[j].w*orient[j].x );
            
            //recuperation moment inertiel
            Vec3d minert = species->at(t).m_minert;
            //calcul du moment angulaire dans le repere mobile
            Vec3d angmom_m = mat_lab_bf * angmom[j];
            //calcul du couple dans le repere mobile
            Vec3d couple_m = mat_lab_bf * couple[j];
            //calcul du moment angulaire dans repere fixe a t+dt/2
            angmom[j] += couple[j]*dt/2.0;
            //calcul du moment angulaire dans repere mobile a t+dt/2
            Vec3d omega_m={0.,0.,0.};
            if (minert.x>0.) {omega_m.x=angmom_m.x/minert.x;}
            if (minert.y>0.) {omega_m.y=angmom_m.y/minert.y;}
            if (minert.z>0.) {omega_m.z=angmom_m.z/minert.z;}
            Vec3d angmom_m_ddt = angmom_m + ( couple_m - cross( omega_m, angmom_m) )*dt/2.0;
            //calcul de la derivee du quaternion a t+dt/2
            Quaternion dquat_dt;
            Vec3d omega_m_ddt={0.,0.,0.};
            if (minert.x>0.) {omega_m_ddt.x=angmom_m_ddt.x/minert.x;}
            if (minert.y>0.) {omega_m_ddt.y=angmom_m_ddt.y/minert.y;}
            if (minert.z>0.) {omega_m_ddt.z=angmom_m_ddt.z/minert.z;}
            dquat_dt.w = 0.5 * (-orient[j].x*omega_m_ddt.x - orient[j].y*omega_m_ddt.y - orient[j].z*omega_m_ddt.z );
            dquat_dt.x = 0.5 * (orient[j].w*omega_m_ddt.x - orient[j].z*omega_m_ddt.y + orient[j].y*omega_m_ddt.z );
            dquat_dt.y = 0.5 * (orient[j].z*omega_m_ddt.x + orient[j].w*omega_m_ddt.y - orient[j].x*omega_m_ddt.z );
            dquat_dt.z = 0.5 * (-orient[j].y*omega_m_ddt.x + orient[j].x*omega_m_ddt.y + orient[j].w*omega_m_ddt.z );
            //calcul du quaternion a t+dt/2 
            Quaternion quat_ddt0 = normalize(orient[j] + dquat_dt*dt/2.0);
            double conv = 1.0;
            static constexpr double conv_crit = 1.e-15;
            int niter = 0;
            static constexpr int niter_max = 30;
            while ( conv>conv_crit && niter<niter_max)
            {
              niter += 1;
              //calcul du moment angulaire dans le repere mobile
              mat_lab_bf.m11 = quat_ddt0.w*quat_ddt0.w + quat_ddt0.x*quat_ddt0.x - quat_ddt0.y*quat_ddt0.y - quat_ddt0.z*quat_ddt0.z;
              mat_lab_bf.m22 = quat_ddt0.w*quat_ddt0.w - quat_ddt0.x*quat_ddt0.x + quat_ddt0.y*quat_ddt0.y - quat_ddt0.z*quat_ddt0.z;
              mat_lab_bf.m33 = quat_ddt0.w*quat_ddt0.w - quat_ddt0.x*quat_ddt0.x - quat_ddt0.y*quat_ddt0.y + quat_ddt0.z*quat_ddt0.z;
              mat_lab_bf.m12 = 2.0 * (quat_ddt0.x*quat_ddt0.y + quat_ddt0.w*quat_ddt0.z ); 
              mat_lab_bf.m21 = 2.0 * (quat_ddt0.x*quat_ddt0.y - quat_ddt0.w*quat_ddt0.z );
              mat_lab_bf.m13 = 2.0 * (quat_ddt0.x*quat_ddt0.z - quat_ddt0.w*quat_ddt0.y );
              mat_lab_bf.m31 = 2.0 * (quat_ddt0.x*quat_ddt0.z + quat_ddt0.w*quat_ddt0.y );
              mat_lab_bf.m23 = 2.0 * (quat_ddt0.y*quat_ddt0.z + quat_ddt0.w*quat_ddt0.x );
              mat_lab_bf.m32 = 2.0 * (quat_ddt0.y*quat_ddt0.z - quat_ddt0.w*quat_ddt0.x );
              //calcul du moment angulaire dans le repere mobile a t+dt/2
              angmom_m_ddt = mat_lab_bf * angmom[j];
              //calcul de la vitesse angulaire dans le repere mobile a t+dt/2
              if (minert.x>0.) {omega_m_ddt.x=angmom_m_ddt.x/minert.x;}
              if (minert.y>0.) {omega_m_ddt.y=angmom_m_ddt.y/minert.y;}
              if (minert.z>0.) {omega_m_ddt.z=angmom_m_ddt.z/minert.z;}
              //calcul de la derivee du quaternion a t+dt/2
              dquat_dt.w = 0.5 * (-quat_ddt0.x*omega_m_ddt.x - quat_ddt0.y*omega_m_ddt.y - quat_ddt0.z*omega_m_ddt.z );
              dquat_dt.x = 0.5 * (quat_ddt0.w*omega_m_ddt.x - quat_ddt0.z*omega_m_ddt.y + quat_ddt0.y*omega_m_ddt.z );
              dquat_dt.y = 0.5 * (quat_ddt0.z*omega_m_ddt.x + quat_ddt0.w*omega_m_ddt.y - quat_ddt0.x*omega_m_ddt.z );
              dquat_dt.z = 0.5 * (-quat_ddt0.y*omega_m_ddt.x + quat_ddt0.x*omega_m_ddt.y + quat_ddt0.w*omega_m_ddt.z );
              //calcul du nouveau quaternion
              Quaternion quat_ddt1 = normalize(orient[j] + dquat_dt*dt/2.0);
              //convergence
              conv = norm(quat_ddt1-quat_ddt0); //en realite on pourrait fusionner les deux lignes precedentes et n'avoir qu'un quaternion temporaire
              //stockage du nouveau quaternion obtenu
              quat_ddt0=quat_ddt1;
            }

	          if( niter_max == niter && conv>conv_crit )
            {
              std::cerr << "Convergence Quaternion "<<conv<<">"<<conv_crit<<" , quat["<<j<<"]="<<orient[j]<<" , couple="<<couple_m<<" , dquat_dt="<<dquat_dt<<std::endl;
              std::cerr << "Cell @"<<loc<<" , part #"<<j<<" , dims="<<dims<<", gl="<<ghost_layers<<" , offset="<<grid->offset()<<std::endl << std::flush;
              for(size_t k=0;k<j;k++)
              {
                std::cerr << "\t"<<k<<" : orient="<<orient[k]<<" , couple="<<couple[k]<<std::endl;                
              }
              std::cerr <<std::endl << std::flush;
              std::abort();
	          }

            //calcul du quaternion a l'instant t+dt
            orient[j] = normalize(orient[j] + dquat_dt*dt);
          }
        }
        GRID_OMP_FOR_END
      }
#endif

    }
    
  };

  // === register factories ===
  template<class GridT> using TorqueToQuaternionRigidMolTmpl = TorqueToQuaternionRigidMol<GridT>;

  ONIKA_AUTORUN_INIT(torque_to_quaternion)
  {
    OperatorNodeFactory::instance()->register_factory("torque_to_quaternion", make_grid_variant_operator< TorqueToQuaternionRigidMolTmpl >);
  }

}
